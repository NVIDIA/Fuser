#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
// [Z,Y,X]_THREADS is the number of participating threads in the z, y, x
// dimension of the block. If set to false the dimension doesn't
// participate in the reduction. We could start with warp reductions, then
// reduce the warps, this could save some shared memory, but could be slower in
// some instances.
//
//  EXAMPLE USAGE:
//  blockReduceSum<X_THREADS, Y_THREADS, Z_THREADS>
//    (output[output_index], inputs[input_index],
//      [] __device__ (T& a, const T b) { a += b; });
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    bool Aligned,
    typename T,
    typename Func>
__device__ void blockReduce(
    T& out,
    const T& inp_val,
    Func reduction_op,
    T* shared_mem,
    bool read_pred,
    bool write_pred,
    T init_val) {
  // If this thread will output a final result
  bool should_write =
      index_utils::maskedIsZero<X_REDUCE, Y_REDUCE, Z_REDUCE>(threadIdx);

  // Size of the reduction segments
  unsigned int reduction_size =
      index_utils::maskedSize<X_REDUCE, Y_REDUCE, Z_REDUCE>(blockDim);

  // Index into the reduction segment
  unsigned int reduction_tid =
      index_utils::maskedOffset<X_REDUCE, Y_REDUCE, Z_REDUCE>(
          threadIdx, blockDim);

  // Index of the reduction segment
  unsigned int reduction_idx =
      index_utils::maskedOffset<!X_REDUCE, !Y_REDUCE, !Z_REDUCE>(
          threadIdx, blockDim);

  // Offset into smem for the current thread
  unsigned int smem_offset = reduction_idx * reduction_size + reduction_tid;

  // Initialize shared memory
  if (read_pred) {
    shared_mem[smem_offset] = inp_val;
  } else {
    shared_mem[smem_offset] = init_val;
  }

  block_sync::sync<Aligned>();
  // Reduce down to nearest power of 2 for the tree reduction:
  int np2 = 1 << (31 - __clz(reduction_size));

  if (reduction_tid < np2 && reduction_tid + np2 < reduction_size) {
    reduction_op(shared_mem[smem_offset], shared_mem[smem_offset + np2]);
  }
  block_sync::sync<Aligned>();

  // loop peel the final iteration to save one syncthread for the end
  for (int factor = np2 / 2; factor > 1; factor >>= 1) {
    if (reduction_tid < factor) {
      reduction_op(shared_mem[smem_offset], shared_mem[smem_offset + factor]);
    }
    block_sync::sync<Aligned>();
  }

  if (should_write && write_pred) {
    T result = out;
    reduction_op(result, shared_mem[smem_offset]);
    if (reduction_size > 1) {
      reduction_op(result, shared_mem[smem_offset + 1]);
    }
    out = result;
  }
  block_sync::sync<Aligned>();
}

// Use the same pred for both reads and writes
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    bool Aligned,
    typename T,

    typename Func>
__device__ void blockReduce(
    T& out,
    const T& inp_val,
    Func reduction_op,
    T* shared_mem,
    bool read_write_pred,
    T init_val) {
  blockReduce<X_REDUCE, Y_REDUCE, Z_REDUCE, Aligned, T, Func>(
      out,
      inp_val,
      reduction_op,
      shared_mem,
      read_write_pred,
      read_write_pred,
      init_val);
}

// Each thread in the iteration dimension processes N elements
// Typical usage is in outer reduction where the iteration dimension
// is parallelized by vectorized loads, bidmx. The reduction dimension
// is parallelized by bdimy. This function works as follows:
// (1) Each thread vectorized loads N elements from input register array to
// smem. (2) do N * bdimx parallel reductions in smem.

// TODO: merge `blockIterGroupedReduce` with `blockReduce`
// (1) for-loops are fully unrolled should not cause overhead for `blockReduce`
// (2) used in gridReduce, needs to change correspodning gridReduce function
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    bool Aligned,
    int N, // Number of elements per input array
    typename T,
    typename Func>
__device__ void blockIterGroupedReduce(
    T out[N],
    const T inp_val[N],
    Func reduction_op,
    T* shared_mem,
    bool read_pred,
    bool write_pred,
    T init_val) {
  // N should be a valid vectorization factor
  static_assert(
      N == 2 || N == 4 || N == 8 || N == 16,
      "N should be a valid vectorization factor, one of (2, 4, 8, 16)!");

  bool should_write =
      index_utils::maskedIsZero<X_REDUCE, Y_REDUCE, Z_REDUCE>(threadIdx);

  unsigned int reduction_size =
      index_utils::maskedSize<X_REDUCE, Y_REDUCE, Z_REDUCE>(blockDim);

  unsigned int reduction_tid =
      index_utils::maskedOffset<X_REDUCE, Y_REDUCE, Z_REDUCE>(
          threadIdx, blockDim);

  unsigned int reduction_idx =
      index_utils::maskedOffset<!X_REDUCE, !Y_REDUCE, !Z_REDUCE>(
          threadIdx, blockDim);

  // Adjust shared memory offset for array processing
  unsigned int smem_offset =
      (reduction_idx * reduction_size + reduction_tid) * N;
  if (read_pred) {
    // This section calculates the number of vectorized load operations required
    // to fetch all elements of an array into shared memory, assuming each load
    // can transfer up to 16 bytes. For example, with fusion input vectorized by
    // 8 (N = 8) and computations in fp32 (sizeof(T) = 4 bytes), the total data
    // size is 4 * 8 = 32 bytes, necessitating 32 / 16 = 2 load transactions.
    // Each transaction loads 16 / 4 (bytes per element) = 4 elements.
    if constexpr (sizeof(T) * N <= 16) {
      loadGeneric<T, N>(shared_mem + smem_offset, const_cast<T*>(inp_val));
    } else {
      constexpr unsigned int total_loads = sizeof(T) * N / 16;
      constexpr unsigned int elements_per_load = 16 / sizeof(T);
      static_assert(
          sizeof(T) * N == 16 * total_loads,
          "This combination of vectorization factor and data type is not supported!");
      static_assert(
          sizeof(T) * elements_per_load == 16,
          "This data type is not supported!");
#pragma unroll
      for (unsigned int i = 0; i < total_loads; ++i) {
        loadGeneric<T, elements_per_load>(
            shared_mem + smem_offset + i * elements_per_load,
            const_cast<T*>(inp_val) + i * elements_per_load);
      }
    }
  } else {
#pragma unroll
    for (int i = 0; i < N; ++i) {
      shared_mem[smem_offset + i] = init_val;
    }
  }

  block_sync::sync<Aligned>();

  // Reduce down to nearest power of 2 for the tree reduction:
  int np2 = 1 << (31 - __clz(reduction_size));

  // Perform parallel reduction for each element in the array
  if (reduction_tid < np2 && reduction_tid + np2 < reduction_size) {
#pragma unroll
    for (int i = 0; i < N; ++i) {
      reduction_op(
          shared_mem[smem_offset + i], shared_mem[smem_offset + np2 * N + i]);
    }
  }

  block_sync::sync<Aligned>();

  for (int factor = np2 / 2; factor > 1; factor >>= 1) {
    if (reduction_tid < factor) {
#pragma unroll
      for (int i = 0; i < N; ++i) {
        reduction_op(
            shared_mem[smem_offset + i],
            shared_mem[smem_offset + factor * N + i]);
      }
    }
    block_sync::sync<Aligned>();
  }

  if (should_write && write_pred) {
#pragma unroll
    for (int i = 0; i < N; ++i) {
      T result = out[i];
      reduction_op(result, shared_mem[smem_offset + i]);
      if (reduction_size > 1) {
        reduction_op(
            result,
            shared_mem[smem_offset + N + i]); // Handle the last element if
                                              // reduction size is odd
      }
      out[i] = result;
    }
  }
  block_sync::sync<Aligned>();
}

// Use the same pred for both reads and writes
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    bool Aligned,
    int N, // Number of elements per input array
    typename T,
    typename Func>
__device__ void blockIterGroupedReduce(
    T out[N],
    const T inp_val[N],
    Func reduction_op,
    T* shared_mem,
    bool read_write_pred,
    T init_val) {
  blockIterGroupedReduce<X_REDUCE, Y_REDUCE, Z_REDUCE, Aligned, N, T, Func>(
      out,
      inp_val,
      reduction_op,
      shared_mem,
      read_write_pred,
      read_write_pred,
      init_val);
}
