#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
// [Z,Y,X]_THREADS is the number of participating threads in the z, y, x
// dimension of the block. If set to false the dimension doesn't
// participate in the reduction. We could start with warp reductions, then
// reduce the warps, this could save some shared memory, but could be slower in
// some instances.
//
//  EXAMPLE USAGE:
//  blockReduceSum<X_THREADS, Y_THREADS, Z_THREADS>
//    (output[output_index], inputs[input_index],
//      [] __device__ (T& a, const T b) { a += b; });
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    bool Aligned,
    typename T,
    typename Func>
__device__ void blockReduce(
    T& out,
    const T& inp_val,
    Func reduction_op,
    T* shared_mem,
    bool read_pred,
    bool write_pred,
    T init_val) {
  // If this thread will output a final result
  bool should_write =
      index_utils::maskedIsZero<X_REDUCE, Y_REDUCE, Z_REDUCE>(threadIdx);

  // Size of the reduction segments
  unsigned int reduction_size =
      index_utils::maskedSize<X_REDUCE, Y_REDUCE, Z_REDUCE>(blockDim);

  // Index into the reduction segment
  unsigned int reduction_tid =
      index_utils::maskedOffset<X_REDUCE, Y_REDUCE, Z_REDUCE>(
          threadIdx, blockDim);

  // Index of the reduction segment
  unsigned int reduction_idx =
      index_utils::maskedOffset<!X_REDUCE, !Y_REDUCE, !Z_REDUCE>(
          threadIdx, blockDim);

  // Offset into smem for the current thread
  unsigned int smem_offset = reduction_idx * reduction_size + reduction_tid;

  // Initialize shared memory
  if (read_pred) {
    shared_mem[smem_offset] = inp_val;
  } else {
    shared_mem[smem_offset] = init_val;
  }

  block_sync::sync<Aligned>();
  // Reduce down to nearest power of 2 for the tree reduction:
  int np2 = 1 << (31 - __clz(reduction_size));

  if (reduction_tid < np2 && reduction_tid + np2 < reduction_size) {
    reduction_op(shared_mem[smem_offset], shared_mem[smem_offset + np2]);
  }
  block_sync::sync<Aligned>();

  // loop peel the final iteration to save one syncthread for the end
  for (int factor = np2 / 2; factor > 1; factor >>= 1) {
    if (reduction_tid < factor) {
      reduction_op(shared_mem[smem_offset], shared_mem[smem_offset + factor]);
    }
    block_sync::sync<Aligned>();
  }

  if (should_write && write_pred) {
    T result = out;
    reduction_op(result, shared_mem[smem_offset]);
    if (reduction_size > 1) {
      reduction_op(result, shared_mem[smem_offset + 1]);
    }
    out = result;
  }
  block_sync::sync<Aligned>();
}

// Use the same pred for both reads and writes
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    bool Aligned,
    typename T,

    typename Func>
__device__ void blockReduce(
    T& out,
    const T& inp_val,
    Func reduction_op,
    T* shared_mem,
    bool read_write_pred,
    T init_val) {
  blockReduce<X_REDUCE, Y_REDUCE, Z_REDUCE, Aligned, T, Func>(
      out,
      inp_val,
      reduction_op,
      shared_mem,
      read_write_pred,
      read_write_pred,
      init_val);
}

// Each thread in the iteration dimension processes N elements
// Typical usage is in outer reduction where the iteration dimension
// is parallelized by vectorized loads, bidmx. The reduction dimension
// is parallelized by bdimy. This function works as follows:
// (1) Each thread vectorized loads N elements from input register array to
// smem. (2) do N * bdimx parallel reductions in smem.
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    bool Aligned,
    int N, // Number of elements per input array
    typename T,
    typename Func>
__device__ void iterGroupedBlockReduce(
    T out[N],
    const T inp_val[N],
    Func reduction_op,
    T* shared_mem,
    bool read_pred,
    bool write_pred,
    T init_val) {
  bool should_write =
      index_utils::maskedIsZero<X_REDUCE, Y_REDUCE, Z_REDUCE>(threadIdx);

  unsigned int reduction_size =
      index_utils::maskedSize<X_REDUCE, Y_REDUCE, Z_REDUCE>(blockDim);

  unsigned int reduction_tid =
      index_utils::maskedOffset<X_REDUCE, Y_REDUCE, Z_REDUCE>(
          threadIdx, blockDim);

  unsigned int reduction_idx =
      index_utils::maskedOffset<!X_REDUCE, !Y_REDUCE, !Z_REDUCE>(
          threadIdx, blockDim);

  // Adjust shared memory offset for array processing
  unsigned int smem_offset =
      (reduction_idx * reduction_size + reduction_tid) * N;
  if (read_pred) {
    if constexpr (sizeof(T) * N <= 16) {
      loadGeneric<T, N>(shared_mem + smem_offset, const_cast<T*>(inp_val));
    } else {
      // may larger than 16 bytes, e.g. input fp16 vectorized by 8
      // but calculation is fp32/fp64 vectorized by 8
      constexpr unsigned int total_loads = sizeof(T) * N / 16;
      constexpr unsigned int elements_per_load = 16 / sizeof(T);
#pragma unroll
      for (unsigned int i = 0; i < total_loads; ++i) {
        loadGeneric<T, elements_per_load>(
            shared_mem + smem_offset + i * elements_per_load,
            const_cast<T*>(inp_val) + i * elements_per_load);
      }
    }
  } else {
#pragma unroll
    for (int i = 0; i < N; ++i) {
      shared_mem[smem_offset + i] = init_val;
    }
  }

  block_sync::sync<Aligned>();

  int np2 = 1 << (31 - __clz(reduction_size));

  // Perform parallel reduction for each element in the array
  if (reduction_tid < np2 && reduction_tid + np2 < reduction_size) {
#pragma unroll
    for (int i = 0; i < N; ++i) {
      reduction_op(
          shared_mem[smem_offset + i], shared_mem[smem_offset + np2 * N + i]);
    }
  }

  block_sync::sync<Aligned>();

  for (int factor = np2 / 2; factor > 1; factor >>= 1) {
    if (reduction_tid < factor) {
#pragma unroll
      for (int i = 0; i < N; ++i) {
        reduction_op(
            shared_mem[smem_offset + i],
            shared_mem[smem_offset + factor * N + i]);
      }
    }
    block_sync::sync<Aligned>();
  }

  if (should_write && write_pred) {
#pragma unroll
    for (int i = 0; i < N; ++i) {
      T result = out[i];
      reduction_op(result, shared_mem[smem_offset + i]);
      if (reduction_size > 1) {
        reduction_op(
            result,
            shared_mem[smem_offset + N + i]); // Handle the last element if
                                              // reduction size is odd
      }
      out[i] = result;
    }
  }
  block_sync::sync<Aligned>();
}

// Use the same pred for both reads and writes
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    bool Aligned,
    int N, // Number of elements per input array
    typename T,
    typename Func>
__device__ void iterGroupedBlockReduce(
    T out[N],
    const T inp_val[N],
    Func reduction_op,
    T* shared_mem,
    bool read_write_pred,
    T init_val) {
  iterGroupedBlockReduce<X_REDUCE, Y_REDUCE, Z_REDUCE, Aligned, N, T, Func>(
      out,
      inp_val,
      reduction_op,
      shared_mem,
      read_write_pred,
      read_write_pred,
      init_val);
}
