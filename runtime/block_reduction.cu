#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
// [Z,Y,X]_THREADS is the number of participating threads in the z, y, x
// dimension of the block. If set to false the dimension doesn't
// participate in the reduction. We could start with warp reductions, then
// reduce the warps, this could save some shared memory, but could be slower in
// some instances.
//
//  EXAMPLE USAGE:
//  blockReduceSum<X_THREADS, Y_THREADS, Z_THREADS>
//    (output[output_index], inputs[input_index],
//      [] __device__ (T& a, const T b) { a += b; });
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    bool Aligned,
    typename T,
    typename Func>
__device__ void blockReduce(
    T& out,
    const T& inp_val,
    Func reduction_op,
    T* shared_mem,
    bool read_pred,
    bool write_pred,
    T init_val) {
  // If this thread will output a final result
  bool should_write =
      index_utils::maskedIsZero<X_REDUCE, Y_REDUCE, Z_REDUCE>(threadIdx);

  // Size of the reduction segments
  unsigned int reduction_size =
      index_utils::maskedSize<X_REDUCE, Y_REDUCE, Z_REDUCE>(blockDim);

  // Index into the reduction segment
  unsigned int reduction_tid =
      index_utils::maskedOffset<X_REDUCE, Y_REDUCE, Z_REDUCE>(
          threadIdx, blockDim);

  // Index of the reduction segment
  unsigned int reduction_idx =
      index_utils::maskedOffset<!X_REDUCE, !Y_REDUCE, !Z_REDUCE>(
          threadIdx, blockDim);

  // smem_offset is the offset into shared memory for the current thread.
  // To ensure coalesced access to shared memory, we need to ensure
  // each transaction is accessing a contiguous block of 128 bytes.
  // For outer reduction where TIDy is in the reduction dimension and TIDx
  // is in the iteration dimension and TIDz is not used. We have
  // reduction_tid = TIDy and reduction_idx = TIDx. If we directly use the
  // offset based on reduction_tid and reduction_idx, we will have stride
  // access to shared memory. For example:
  // offset = reduction_idx * reduction_size + reduction_tid
  //        = TIDx * blockDim.y + TIDy
  // To avoid this, we should always use the offset based on the indexing of
  // threads within a block.
  // Offset into smem for the current thread
  unsigned int smem_offset = threadIdx.x + threadIdx.y * blockDim.x +
      threadIdx.z * blockDim.x * blockDim.y;

  // Initialize shared memory
  if (read_pred) {
    shared_mem[smem_offset] = inp_val;
  } else {
    shared_mem[smem_offset] = init_val;
  }

  block_sync::sync<Aligned>();
  // Reduce down to nearest power of 2 for the tree reduction:
  int np2 = 1 << (31 - __clz(reduction_size));

  if (reduction_tid < np2 && reduction_tid + np2 < reduction_size) {
    reduction_op(shared_mem[smem_offset], shared_mem[smem_offset + np2]);
  }
  block_sync::sync<Aligned>();

  // loop peel the final iteration to save one syncthread for the end
  for (int factor = np2 / 2; factor > 1; factor >>= 1) {
    if (reduction_tid < factor) {
      reduction_op(shared_mem[smem_offset], shared_mem[smem_offset + factor]);
    }
    block_sync::sync<Aligned>();
  }

  if (should_write && write_pred) {
    T result = out;
    reduction_op(result, shared_mem[smem_offset]);
    if (reduction_size > 1) {
      reduction_op(result, shared_mem[smem_offset + 1]);
    }
    out = result;
  }
  block_sync::sync<Aligned>();
}

// Use the same pred for both reads and writes
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    bool Aligned,
    typename T,
    typename Func>
__device__ void blockReduce(
    T& out,
    const T& inp_val,
    Func reduction_op,
    T* shared_mem,
    bool read_write_pred,
    T init_val) {
  blockReduce<X_REDUCE, Y_REDUCE, Z_REDUCE, Aligned, T, Func>(
      out,
      inp_val,
      reduction_op,
      shared_mem,
      read_write_pred,
      read_write_pred,
      init_val);
}

// Each thread in the iteration dimension processes N elements
// Typical usage is in outer reduction where the iteration dimension
// is parallelized by vectorized loads, bidmx. The reduction dimension
// is parallelized by bdimy. This function works as follows:
// (1) Each thread vectorized loads N elements from input register array to
// smem. (2) do N * bdimx parallel reductions in smem.

// TODO: merge `blockIterGroupedReduce` with `blockReduce`
// (1) for-loops are fully unrolled should not cause overhead for `blockReduce`
// (2) used in gridReduce, needs to change correspodning gridReduce function
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    bool Aligned,
    int N, // Number of elements per input array
    typename T,
    typename Func>
__device__ void blockIterGroupedReduce(
    T out[N],
    const T inp_val[N],
    Func reduction_op,
    T* shared_mem,
    bool read_pred,
    bool write_pred,
    T init_val) {
  // N should be a valid vectorization factor
  static_assert(
      N == 2 || N == 4 || N == 8 || N == 16,
      "N should be a valid vectorization factor, one of (2, 4, 8, 16)!");

  bool should_write =
      index_utils::maskedIsZero<X_REDUCE, Y_REDUCE, Z_REDUCE>(threadIdx);

  unsigned int reduction_size =
      index_utils::maskedSize<X_REDUCE, Y_REDUCE, Z_REDUCE>(blockDim);

  unsigned int reduction_tid =
      index_utils::maskedOffset<X_REDUCE, Y_REDUCE, Z_REDUCE>(
          threadIdx, blockDim);

  unsigned int reduction_idx =
      index_utils::maskedOffset<!X_REDUCE, !Y_REDUCE, !Z_REDUCE>(
          threadIdx, blockDim);

  // In shared memory, each row has 128 bytes, if sizeof(T) * N = 32 bytes, each row has 128 / 32 = 4 threads.
  // Each transaction can only load data from one row, with a max of 16 bytes per thread.
  // So the total bytes per transaction is 4 x 16 = 64 bytes which is only half of the
  // maximum 128 bytes per transaction. we should change the layout from [TIDy, TIDx, N] to 
  // [N/4, TIDy, TIDx, 4]
  constexpr unsigned int total_loads = sizeof(T) * N / 16 > 1 ? sizeof(T) * N / 16 : 1;
  constexpr unsigned int elements_per_load = 16 / sizeof(T) > N ? N : 16 / sizeof(T);

  if(true){
    unsigned int smem_offset_inter = blockDim.x * blockDim.y * blockDim.z * elements_per_load;
    unsigned int smem_offset_intra = (threadIdx.z * blockDim.x  * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x) * elements_per_load;
    
    // load to [total_loads] sections of shared memory
    #pragma unroll
    for (unsigned int i = 0; i < total_loads; ++i) {
      loadGeneric<T, elements_per_load>(
          shared_mem + smem_offset_inter * i + smem_offset_intra,
          const_cast<T*>(inp_val) + i * elements_per_load);
    }
    block_sync::sync<Aligned>();

    // Reduce down to nearest power of 2 for the tree reduction:
    // Perform parallel reduction for each element in the array
    int np2 = 1 << (31 - __clz(reduction_size));
    if (reduction_tid < np2 && reduction_tid + np2 < reduction_size) {
      // vectorized load from smem to regs
      T self[N];
      T peer[N];
      #pragma unroll
      for (unsigned int i = 0; i < total_loads; ++i) {
        int self_offset = smem_offset_inter * i + smem_offset_intra;
        int peer_offset = self_offset + np2 * elements_per_load * blockDim.x;
        loadGeneric<T, elements_per_load>(self + i * elements_per_load, shared_mem + self_offset);
        loadGeneric<T, elements_per_load>(peer + i * elements_per_load, shared_mem + peer_offset);          
      }
      // reduction
      #pragma unroll
      for (int i = 0; i < N; ++i) {
        reduction_op(self[i], peer[i]);
      }
      // write self back to smem
      #pragma unroll
      for (unsigned int i = 0; i < total_loads; ++i) {
        int self_offset = smem_offset_inter * i + smem_offset_intra;
        loadGeneric<T, elements_per_load>(shared_mem + self_offset, self + i * elements_per_load);
      }
    }
    block_sync::sync<Aligned>();

    // Tree reduction
    for (int factor = np2 / 2; factor > 1; factor >>= 1) {
      if (reduction_tid < factor) {
        // vectorized load from smem to regs
        T self[N];
        T peer[N];
        #pragma unroll
        for (unsigned int i = 0; i < total_loads; ++i) {
          int self_offset = smem_offset_inter * i + smem_offset_intra;
          int peer_offset = self_offset + factor * elements_per_load * blockDim.x;
          loadGeneric<T, elements_per_load>(self + i * elements_per_load, shared_mem + self_offset);
          loadGeneric<T, elements_per_load>(peer + i * elements_per_load, shared_mem + peer_offset);             
        }
        // reduction
        #pragma unroll
        for (int i = 0; i < N; ++i) {
          reduction_op(self[i], peer[i]);
        }
        // write self back to smem
        #pragma unroll
        for (unsigned int i = 0; i < total_loads; ++i) {
          int self_offset = smem_offset_inter * i + smem_offset_intra;
          loadGeneric<T, elements_per_load>(shared_mem + self_offset, self + i * elements_per_load);
        }
      }
      block_sync::sync<Aligned>();
    }

    // last reduction
    if (should_write && write_pred) {
      // init result
      T result[N];
      #pragma unroll
      for (int i = 0; i < N; ++i) {
        result[i] = out[i];
      }
  
      // copy first element to result
      T self[N];
      #pragma unroll
      for (unsigned int i = 0; i < total_loads; ++i) {
        int self_offset = smem_offset_inter * i + smem_offset_intra;
        loadGeneric<T, elements_per_load>( self + i * elements_per_load, shared_mem + self_offset);
      }
      #pragma unroll
      for (int i = 0; i < N; ++i) {
        reduction_op(result[i], self[i]);
      }
  
      // reduction of the 2nd last element
      if(reduction_size > 1){
        T peer[N];
        #pragma unroll
        for (unsigned int i = 0; i < total_loads; ++i) {
          int peer_offset = smem_offset_inter * i + smem_offset_intra + elements_per_load * blockDim.x;
          loadGeneric<T, elements_per_load>( peer + i * elements_per_load,  shared_mem + peer_offset);
        }
        #pragma unroll
        for (int i = 0; i < N; ++i) {
          reduction_op(result[i], peer[i]);
        }
      }
      #pragma unroll
      for (int i = 0; i < N; ++i) {
        out[i] = result[i];
      }
    }
    block_sync::sync<Aligned>();
  }
}

// Use the same pred for both reads and writes
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    bool Aligned,
    int N, // Number of elements per input array
    typename T,
    typename Func>
__device__ void blockIterGroupedReduce(
    T out[N],
    const T inp_val[N],
    Func reduction_op,
    T* shared_mem,
    bool read_write_pred,
    T init_val) {
  blockIterGroupedReduce<X_REDUCE, Y_REDUCE, Z_REDUCE, Aligned, N, T, Func>(
      out,
      inp_val,
      reduction_op,
      shared_mem,
      read_write_pred,
      read_write_pred,
      init_val);
}
