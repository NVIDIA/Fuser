#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
// [Z,Y,X]_THREADS is the number of participating threads in the z, y, x
// dimension of the block. If set to false the dimension doesn't
// participate in the reduction. We could start with warp reductions, then
// reduce the warps, this could save some shared memory, but could be slower in
// some instances.
//
//  EXAMPLE USAGE:
//  blockReduceSum<X_THREADS, Y_THREADS, Z_THREADS>
//    (output[output_index], inputs[input_index],
//      [] __device__ (T& a, const T b) { a += b; });
//
// Note: We agressively template functions taking dim3 in the functions below
//       because ROCM uses different types for the various dim3 and maps them
//       directly to intrinsics, but they're dim3 when used after modification.
//
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    typename T,
    typename Func,
    typename _dim3,
    typename _dim3_2>
__device__ void blockReduce(
    T& out,
    const T& inp_val,
    Func reduction_op,
    const _dim3& thread_idx,
    const _dim3_2& block_dim,
    T* shared_mem,
    bool read_pred,
    bool write_pred,
    T init_val) {
  // If this thread will output a final result
  bool should_write =
      index_utils::maskedIsZero<X_REDUCE, Y_REDUCE, Z_REDUCE>(thread_idx);

  // Size of the reduction segments
  unsigned int reduction_size =
      index_utils::maskedSize<X_REDUCE, Y_REDUCE, Z_REDUCE>(block_dim);

  // Index into the reduction segment
  unsigned int reduction_tid =
      index_utils::maskedOffset<X_REDUCE, Y_REDUCE, Z_REDUCE>(
          thread_idx, block_dim);

  // Index of the reduction segment
  unsigned int reduction_idx =
      index_utils::maskedOffset<!X_REDUCE, !Y_REDUCE, !Z_REDUCE>(
          thread_idx, block_dim);

  // Offset into smem for the current thread
  unsigned int smem_offset = reduction_idx * reduction_size + reduction_tid;

  // Initialize shared memory
  if (read_pred) {
    shared_mem[smem_offset] = inp_val;
  } else {
    shared_mem[smem_offset] = init_val;
  }

  block_sync::sync();
  // Reduce down to nearest power of 2 for the tree reduction:
  int np2 = 1 << (31 - __clz(reduction_size));

  if (reduction_tid < np2 && reduction_tid + np2 < reduction_size) {
    reduction_op(shared_mem[smem_offset], shared_mem[smem_offset + np2]);
  }
  block_sync::sync();

  // loop peel the final iteration to save one syncthread for the end
  for (int factor = np2 / 2; factor > 1; factor >>= 1) {
    if (reduction_tid < factor) {
      reduction_op(shared_mem[smem_offset], shared_mem[smem_offset + factor]);
    }
    block_sync::sync();
  }

  if (should_write && write_pred) {
    T result = out;
    reduction_op(result, shared_mem[smem_offset]);
    if (reduction_size > 1) {
      reduction_op(result, shared_mem[smem_offset + 1]);
    }
    out = result;
  }
  block_sync::sync();
}

// Use the same pred for both reads and writes
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    typename T,
    typename Func,
    typename _dim3,
    typename _dim3_2>
__device__ void blockReduce(
    T& out,
    const T& inp_val,
    Func reduction_op,
    const _dim3& thread_idx,
    const _dim3_2& block_dim,
    T* shared_mem,
    bool read_write_pred,
    T init_val) {
  blockReduce<X_REDUCE, Y_REDUCE, Z_REDUCE, T, Func, _dim3, _dim3_2>(
      out,
      inp_val,
      reduction_op,
      thread_idx,
      block_dim,
      shared_mem,
      read_write_pred,
      read_write_pred,
      init_val);
}
