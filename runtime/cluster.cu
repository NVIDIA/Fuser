// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
#pragma once
#include <hip/hip_runtime.h>


#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))

// The optional .relaxed qualifier on barrier.cluster.arrive specifies that
// there are no memory ordering and visibility guarantees provided for the
// memory accesses performed prior to barrier.cluster.arrive.
void clusterArriveRelaxed() {
  asm volatile("barrier.cluster.arrive.relaxed.aligned;" : :);
}

// A thread arrives at barrier but it does not have to wait for threads in other
// participating warps.
void clusterArrive() {
  asm volatile("barrier.cluster.arrive.aligned;" : :);
}

// A thread waits for all non-exited threads of the cluster to perform
// cluster_arrive.
void clusterWait() {
  asm volatile("barrier.cluster.wait.aligned;" : :);
}

// Synchronize threads in cluster
void clusterSync() {
  clusterArrive();
  clusterWait();
}

// Returns the dim3 grid size in terms of number of clusters.
dim3 clusterGridDims() {
  uint32_t x, y, z;
  asm volatile("mov.u32 %0, %%nclusterid.x;" : "=r"(x) :);
  asm volatile("mov.u32 %0, %%nclusterid.y;" : "=r"(y) :);
  asm volatile("mov.u32 %0, %%nclusterid.z;" : "=r"(z) :);
  return {x, y, z};
}

// Returns the dim3 cluster rank in the grid.
dim3 clusterIdInGrid() {
  uint32_t x, y, z;
  asm volatile("mov.u32 %0, %%clusterid.x;" : "=r"(x) :);
  asm volatile("mov.u32 %0, %%clusterid.y;" : "=r"(y) :);
  asm volatile("mov.u32 %0, %%clusterid.z;" : "=r"(z) :);
  return {x, y, z};
}

// Returns the relative dim3 block rank local to the cluster.
dim3 blockIdInCluster() {
  uint32_t x, y, z;
  asm volatile("mov.u32 %0, %%cluster_ctaid.x;" : "=r"(x) :);
  asm volatile("mov.u32 %0, %%cluster_ctaid.y;" : "=r"(y) :);
  asm volatile("mov.u32 %0, %%cluster_ctaid.z;" : "=r"(z) :);
  return {x, y, z};
}

// Returns the dim3 cluster shape.
dim3 clusterShape() {
  uint32_t x, y, z;
  asm volatile("mov.u32 %0, %%cluster_nctaid.x;" : "=r"(x) :);
  asm volatile("mov.u32 %0, %%cluster_nctaid.y;" : "=r"(y) :);
  asm volatile("mov.u32 %0, %%cluster_nctaid.z;" : "=r"(z) :);
  return {x, y, z};
}

// Get 1D ctaid in a cluster.
uint32_t blockRankInCluster() {
  uint32_t rank;
  asm volatile("mov.u32 %0, %%cluster_ctarank;" : "=r"(rank) :);
  return rank;
}

// Set the destination block-ID in cluster for a given SMEM Address
uint32_t mapSharedRank(uint32_t smemAddr, uint32_t rank) {
  uint32_t result;
  asm volatile("mapa.shared::cluster.u32  %0, %1, %2;"
               : "=r"(result)
               : "r"(smemAddr), "r"(rank));
  return result;
}

#endif // Arch 90
