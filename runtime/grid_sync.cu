#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
namespace grid_sync {

// Get the first bit in a 64 bit integer
#define FIRST_UINT64_BIT ((uint64_t)1 << (sizeof(uint64_t) * 8 - 1))

template <typename T>
__device__ T globalAsVolatile(volatile T& global_val) {
  return global_val;
}

// A grid synchronization that can be called multiple times in a kernel assuming
// all the blocks fit on device at once. The semaphore is an integer semaphore
// assumed to be initialized to 0 before launching the kernel. The persistent
// option should be envoked if this sync will be called multiple times in one
// kernel (i.e. having a grid reduce within a loop). Having multiple grid syncs
// called once in the same kernel does not require persistent mode. Segment size
// is the number of blocks participating in the sync in the dimensions marked by
// [X,Y,Z]_BLOCK. The granularity of this sync are those dimensions. I.E.
// Marking X and Y but not Z means there should be Z semaphores of size X*Y.
template <
    bool X_BLOCK,
    bool Y_BLOCK,
    bool Z_BLOCK,
    bool PERSISTENT,
    bool Aligned>
__device__ void sync(
    int64_t& semaphore,
    const uint64_t& segment_size,
    const bool last_block) {
  // Finish all global memory transactions before synchronizing
  __threadfence();

  // Synchronize all threads in a block before synchronizing blocks
  block_sync::sync<Aligned>();

  // Only allow linear_tid == 0 to participate in the synchronization
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    // Get increment value, only want a single block to have the large
    // increment, doesn't really matter which one, the goal is to flip/flop the
    // first bit of a uint64_t value, since our semaphores are actualy int64_t
    // we will just reinterpret_cast it to act as a uint64_t
    uint64_t semaphore_increment = 1;

    // Makes the assumption that blocks are in increasing order, this is not
    // guaranteed by CUDA but this is the current behavior, and unlikely to
    // change.
    if (last_block) {
      semaphore_increment = FIRST_UINT64_BIT - (segment_size - 1);
    }

    uint64_t oldArrive =
        atomicAdd(reinterpret_cast<uint64_t*>(&semaphore), semaphore_increment);

    // If for persistent kernels, lock all blocks until the semaphore has been
    // reached. Make sure we access semaphore as a volatile address so we get
    // the global memory updates.
    unsigned int ns = 8;
    while ((PERSISTENT || last_block) &&
           ((oldArrive ^ globalAsVolatile(semaphore)) & FIRST_UINT64_BIT) ==
               0) {
      // Put a sleep here so we have some breaks in probing the global
      // semaphore, giving a better chance for other warps/blocks to catch up.
#if __CUDA_ARCH__ >= 700
      // __nanosleep only available on compute capability 7.0 or higher
      __nanosleep(ns); // avoids busy waiting
      if (ns < 256) {
        ns *= 2;
      }
#endif
    }
  }

  // Sync block to make sure all other threads are waiting on the sync
  block_sync::sync<Aligned>();
}

template <
    bool X_BLOCK,
    bool Y_BLOCK,
    bool Z_BLOCK,
    bool PERSISTENT,
    bool Aligned>
__device__ void sync(int64_t& semaphore, const uint64_t& segment_size) {
  sync<X_BLOCK, Y_BLOCK, Z_BLOCK, PERSISTENT, Aligned>(
      semaphore,
      segment_size,
      index_utils::maskedIsLast<X_BLOCK, Y_BLOCK, Z_BLOCK>(blockIdx, gridDim));
}

// Grid sync that can be called multiple times in the same kernel without all
// blocks being resident on device. This allows grid sync to be called multiple
// times as long as it's not broadcasted on the parallel axis it was reduced on.
//
// n_entrances is how many times every block is expected to enter into this
// function. All blocks must enter n_entrances times. The last block is only
// allowed to proceed once all other blocks have entered n_entrance
// times.
//
// Note that this is not currently used by grid and welford reduction
// as they use a separate sync flag for each each grid sync call.
template <bool X_BLOCK, bool Y_BLOCK, bool Z_BLOCK, bool Aligned>
__device__ void sync(
    int64_t& semaphore,
    const uint64_t& segment_size,
    const nvfuser_index_t n_entrances) {
  // Finish all global memory transactions before synchronizing
  __threadfence();

  // Synchronize all threads in a block before synchronizing blocks
  block_sync::sync<Aligned>();

  // Only allow linear_tid == 0 to participate in the synchronization
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    // Makes the assumption that blocks are in increasing order, this is not
    // guaranteed by CUDA but this is the current behavior, and unlikely to
    // change.
    bool last_block =
        index_utils::maskedIsLast<X_BLOCK, Y_BLOCK, Z_BLOCK>(blockIdx, gridDim);
    if (last_block) {
      int64_t finished_val =
          ((int64_t)(index_utils::maskedSize<X_BLOCK, Y_BLOCK, Z_BLOCK>(gridDim) - 1)) *
          ((int64_t)n_entrances);

      unsigned int ns = 8;
      // Last block needs to wait for all other blocks to finish
      while (globalAsVolatile(semaphore) < finished_val) {
#if __CUDA_ARCH__ >= 700
        // __nanosleep only available on compute capability 7.0 or higher
        __nanosleep(ns); // avoids busy waiting
        if (ns < 256) {
          ns *= 2;
        }
#endif
      }
    } else {
      auto old = atomicAdd(reinterpret_cast<uint64_t*>(&semaphore), 1);
    }
  }

  // Sync block to make sure all other threads are waiting on the sync
  block_sync::sync<Aligned>();
}

} // namespace grid_sync
