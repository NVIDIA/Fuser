#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
// aligned register array for vectorized load/store
template <typename scalar_t, int size, int align_size = 1>
struct alignas(sizeof(scalar_t) * align_size) Array {
  scalar_t array[size];

  __device__ void set(scalar_t v) {
#pragma unroll
    for (int i = 0; i < size; ++i) {
      array[i] = v;
    }
  }

  __device__ scalar_t& operator[](const unsigned int i) {
    return array[i];
  }

  __device__ const scalar_t& operator[](const unsigned int i) const {
    return array[i];
  }

  Array& operator=(const Array& a) {
#pragma unroll
    for (int i = 0; i < size; ++i) {
      array[i] = a[i];
    }
    return *this;
  }
};

template <int size, int align_size>
struct alignas(align_size / 2) Array<__e2m1, size, align_size> {
  static_assert(size % 2 == 0, "There must be an even number of fp4 elements");
  __e2m1 array[size / 2];

  __device__ __e2m1& operator[](const unsigned int i) {
    // For performance reason, we do not check the index is even, but we assume
    // it. assert(i % 2 == 0);
    return array[i / 2];
  }

  __device__ const __e2m1& operator[](const unsigned int i) const {
    // For performance reason, we do not check the index is even, but we assume
    // it. assert(i % 2 == 0);
    return array[i / 2];
  }

  Array& operator=(const Array& a) {
#pragma unroll
    for (int i = 0; i < size / 2; ++i) {
      array[i] = a.array[i];
    }
    return *this;
  }
};

static_assert(
    sizeof(Array<__e2m1, 2, 2>) == 1,
    "sizeof(Array<__e2m1, 2, 2>) must be 1");
static_assert(
    sizeof(Array<__e2m1, 4, 2>) == 2,
    "sizeof(Array<__e2m1, 4, 2>) must be 2");
static_assert(
    sizeof(Array<__e2m1, 4, 4>) == 2,
    "sizeof(Array<__e2m1, 4, 4>) must be 2");
static_assert(
    sizeof(Array<__e2m1, 8, 2>) == 4,
    "sizeof(Array<__e2m1, 8, 4>) must be 4");
static_assert(
    sizeof(Array<__e2m1, 8, 4>) == 4,
    "sizeof(Array<__e2m1, 8, 4>) must be 4");
static_assert(
    sizeof(Array<__e2m1, 8, 8>) == 4,
    "sizeof(Array<__e2m1, 8, 8>) must be 4");
static_assert(
    sizeof(Array<__e2m1, 16, 2>) == 8,
    "sizeof(Array<__e2m1, 16, 2>) must be 8");
static_assert(
    sizeof(Array<__e2m1, 16, 4>) == 8,
    "sizeof(Array<__e2m1, 16, 4>) must be 8");
static_assert(
    sizeof(Array<__e2m1, 16, 8>) == 8,
    "sizeof(Array<__e2m1, 16, 8>) must be 8");
static_assert(
    sizeof(Array<__e2m1, 16, 16>) == 8,
    "sizeof(Array<__e2m1, 16, 16>) must be 8");

// Used for vectorized allocations that are not in registers
template <typename scalar_t, int vec_size>
__device__ void arraySet(scalar_t* buff, scalar_t val) {
#pragma unroll
  for (int i = 0; i < vec_size; ++i) {
    buff[i] = val;
  }
}

template <typename scalar_t>
constexpr int64_t vecSizeBit(int64_t vec_size) {
  return vec_size * sizeof(scalar_t) * 8;
}

template <>
constexpr int64_t vecSizeBit<__e2m1>(int64_t vec_size) {
  return vec_size * 4;
}

template <typename scalar_t, int vec_size>
__device__ void loadGeneric(scalar_t* to, scalar_t* from) {
  // It would be really nice to use memcpy here, but one example was failing
  // with:
  //
  //  memcpy(to, from, vec_size * sizeof(scalar_t));
  //
  // Yet passing with:
  //
  // for(int i = 0; i < vec_size; i++){
  //   to[i] = from[i];
  // }

  constexpr int64_t vec_size_bit = vecSizeBit<scalar_t>(vec_size);
  static_assert(vec_size_bit % 8 == 0, "vec_size_bit must be a multiple of 8");
  switch (vec_size_bit) {
    case 8:
      *reinterpret_cast<uchar1*>(to) = *reinterpret_cast<uchar1*>(from);
      break;
    case 16:
      *reinterpret_cast<uchar2*>(to) = *reinterpret_cast<uchar2*>(from);
      break;
    case 32:
      *reinterpret_cast<uint1*>(to) = *reinterpret_cast<uint1*>(from);
      break;
    case 64:
      *reinterpret_cast<uint2*>(to) = *reinterpret_cast<uint2*>(from);
      break;
    case 96:
      *reinterpret_cast<uint3*>(to) = *reinterpret_cast<uint3*>(from);
      break;
    case 128:
      *reinterpret_cast<uint4*>(to) = *reinterpret_cast<uint4*>(from);
      break;
  }
}

// Volatile version only works with c++ fundamnetal types
template <
    typename scalar_t,
    int vec_size,
    bool is_volatile_to,
    bool is_volatile_from>
__device__ void loadGenericVolatile(
    typename MaybeVolatile<scalar_t, is_volatile_to>::type* to,
    typename MaybeVolatile<scalar_t, is_volatile_from>::type* from) {
  constexpr int64_t vec_size_bit = vecSizeBit<scalar_t>(vec_size);
  static_assert(vec_size_bit % 8 == 0, "vec_size_bit must be a multiple of 8");
  switch (vec_size_bit) {
    // Reinterpret cast like this with volatile types only works for C++
    // fundamental types otherwise the = operator is not defined
    case 8:
      *reinterpret_cast<
          typename MaybeVolatile<unsigned char, is_volatile_to>::type*>(to) =
          *reinterpret_cast<
              typename MaybeVolatile<unsigned char, is_volatile_from>::type*>(
              from);
      break;
    case 16:
      *reinterpret_cast<typename MaybeVolatile<short, is_volatile_to>::type*>(
          to) =
          *reinterpret_cast<
              typename MaybeVolatile<short, is_volatile_from>::type*>(from);
      break;
    case 32:
      *reinterpret_cast<
          typename MaybeVolatile<unsigned int, is_volatile_to>::type*>(to) =
          *reinterpret_cast<
              typename MaybeVolatile<unsigned int, is_volatile_from>::type*>(
              from);
      break;
    case 64:
      *reinterpret_cast<typename MaybeVolatile<double, is_volatile_to>::type*>(
          to) =
          *reinterpret_cast<
              typename MaybeVolatile<double, is_volatile_from>::type*>(from);
      break;
  }
}

template <typename scalar_t, int vec_size, bool is_volatile>
__device__ void loadLocalToGlobal(
    typename MaybeVolatile<scalar_t, is_volatile>::type* to,
    scalar_t* from) {
  constexpr int64_t vec_size_bit = vecSizeBit<scalar_t>(vec_size);
  static_assert(vec_size_bit % 8 == 0, "vec_size_bit must be a multiple of 8");
  switch (vec_size_bit) {
    case 8:
    case 16:
    case 32:
      loadGenericVolatile<scalar_t, vec_size, is_volatile, false>(to, from);
      break;
    case 64: {
      uint2 const& data = *reinterpret_cast<uint2*>(from);
      if (is_volatile) {
        asm volatile(
            "st.volatile.global.v2.s32 [%0], {%1,%2};" ::"l"(
                (typename MaybeVolatile<uint2, is_volatile>::type*)to),
            "r"(data.x),
            "r"(data.y));
      } else {
        asm volatile(
            "st.global.cs.v2.s32 [%0], {%1,%2};" ::"l"(
                (typename MaybeVolatile<uint2, is_volatile>::type*)to),
            "r"(data.x),
            "r"(data.y));
      }
      break;
    }
    case 128: {
      uint4 const& data = *reinterpret_cast<uint4*>(from);
      if (is_volatile) {
        asm volatile(
            "st.volatile.global.v4.s32 [%0], {%1,%2,%3,%4};" ::"l"(
                (typename MaybeVolatile<uint4, is_volatile>::type*)to),
            "r"(data.x),
            "r"(data.y),
            "r"(data.z),
            "r"(data.w));
      } else {
        asm volatile(
            "st.global.cs.v4.s32 [%0], {%1,%2,%3,%4};" ::"l"(
                (typename MaybeVolatile<uint4, is_volatile>::type*)to),
            "r"(data.x),
            "r"(data.y),
            "r"(data.z),
            "r"(data.w));
      }
      break;
    }
  }
}

// This is copied from csrc/type.h and should be kept consistent.
enum class CacheOp {
  AllLevels,
  Streaming,
  Global,
};

template <typename T, CacheOp cache_op>
__device__ void loadGlobalToLocalCached(void* to, void* from) {
  T* typed_to = reinterpret_cast<T*>(to);
  T* typed_from = reinterpret_cast<T*>(from);
  switch (cache_op) {
    case CacheOp::AllLevels:
      *typed_to = __ldca(typed_from);
      break;
    case CacheOp::Streaming:
      *typed_to = __ldcs(typed_from);
      break;
    case CacheOp::Global:
      *typed_to = __ldcg(typed_from);
      break;
  }
}

// For simplicity, cache_op is only used for non-volatile loads written in
// inline assembly. Other loads are done with the default cache operator --
// cache all levels. ld.volatile doesn't accept cache operator anyway.
template <typename scalar_t, int vec_size, bool is_volatile, CacheOp cache_op>
__device__ void loadGlobalToLocal(
    scalar_t* to,
    typename MaybeVolatile<scalar_t, is_volatile>::type* from) {
  constexpr int64_t vec_size_bit = vecSizeBit<scalar_t>(vec_size);
  static_assert(vec_size_bit % 8 == 0, "vec_size_bit must be a multiple of 8");
  switch (vec_size_bit) {
    case 8:
    case 16:
    case 32:
      loadGenericVolatile<scalar_t, vec_size, false, is_volatile>(to, from);
      break;
    case 64: {
      if (is_volatile) {
        uint2& data = *reinterpret_cast<uint2*>(to);
        asm volatile("ld.volatile.global.v2.s32 {%0,%1}, [%2];"
                     : "=r"(data.x), "=r"(data.y)
                     : "l"((uint2*)from));
      } else {
        loadGlobalToLocalCached<uint2, cache_op>(
            to, const_cast<scalar_t*>(from));
      }
      break;
    }
    case 128: {
      if (is_volatile) {
        uint4& data = *reinterpret_cast<uint4*>(to);
        asm volatile("ld.volatile.global.v4.s32 {%0,%1,%2,%3}, [%4];"
                     : "=r"(data.x), "=r"(data.y), "=r"(data.z), "=r"(data.w)
                     : "l"((uint4*)from));
      } else {
        loadGlobalToLocalCached<uint4, cache_op>(
            to, const_cast<scalar_t*>(from));
      }
      break;
    }
  }
}

template <
    typename scalar_t,
    int vec_size,
    bool is_volatile_to,
    bool is_volatile_from>
__device__ void loadGlobalToGlobal(
    typename MaybeVolatile<scalar_t, is_volatile_to>::type* to,
    typename MaybeVolatile<scalar_t, is_volatile_from>::type* from) {
  constexpr int64_t vec_size_bit = vecSizeBit<scalar_t>(vec_size);
  static_assert(vec_size_bit % 8 == 0, "vec_size_bit must be a multiple of 8");
  switch (vec_size_bit) {
    // Reinterpret cast like this with volatile types only works for C++
    // fundamental types otherwise the = operator is not defined
    case 8:
    case 16:
    case 32:
    case 64:
      loadGenericVolatile<scalar_t, vec_size, is_volatile_to, is_volatile_from>(
          to, from);
      break;
    case 96: {
      uint3 local_intermediate;
      loadGlobalToLocal<
          scalar_t,
          vec_size,
          is_volatile_from,
          CacheOp::Streaming>(
          reinterpret_cast<scalar_t*>(&local_intermediate), from);
      loadLocalToGlobal<scalar_t, vec_size, is_volatile_to>(
          to, reinterpret_cast<scalar_t*>(&local_intermediate));
      break;
    }
    case 128: {
      uint4 local_intermediate;
      loadGlobalToLocal<
          scalar_t,
          vec_size,
          is_volatile_from,
          CacheOp::Streaming>(
          reinterpret_cast<scalar_t*>(&local_intermediate), from);
      loadLocalToGlobal<scalar_t, vec_size, is_volatile_to>(
          to, reinterpret_cast<scalar_t*>(&local_intermediate));
      break;
    }
  }
}
