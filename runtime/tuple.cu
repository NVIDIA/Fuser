// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
// std::tuple-like type
template <typename... Types>
struct Tuple;

#define TUPLE_INCREMENT_PTR(idx)                                        \
  do {                                                                  \
    static_assert(                                                      \
        IsPointerType<T##idx>::value, "Invalid for non-pointer types"); \
    val##idx += offset;                                                 \
  } while (0)

template <typename T0>
struct Tuple<T0> {
  T0 val0;

  Tuple() = default;

  __device__ Tuple(T0 _val0) : val0(_val0) {}

  // Only valid when instantiated for pointer types
  __device__ void operator+=(nvfuser_index_t offset) {
    TUPLE_INCREMENT_PTR(0);
  }
};

template <typename T0, typename T1>
struct Tuple<T0, T1> {
  T0 val0;
  T1 val1;

  Tuple() = default;

  __device__ Tuple(T0 _val0, T1 _val1) : val0(_val0), val1(_val1) {}

  // Only valid when instantiated for pointer types
  __device__ void operator+=(nvfuser_index_t offset) {
    TUPLE_INCREMENT_PTR(0);
    TUPLE_INCREMENT_PTR(1);
  }
};

template <typename T0, typename T1, typename T2>
struct Tuple<T0, T1, T2> {
  T0 val0;
  T1 val1;
  T2 val2;

  Tuple() = default;

  __device__ Tuple(T0 _val0, T1 _val1, T2 _val2)
      : val0(_val0), val1(_val1), val2(_val2) {}

  // Only valid when instantiated for pointer types
  __device__ void operator+=(nvfuser_index_t offset) {
    TUPLE_INCREMENT_PTR(0);
    TUPLE_INCREMENT_PTR(1);
    TUPLE_INCREMENT_PTR(2);
  }
};

template <typename T0, typename T1, typename T2, typename T3>
struct Tuple<T0, T1, T2, T3> {
  T0 val0;
  T1 val1;
  T2 val2;
  T3 val3;

  Tuple() = default;

  __device__ Tuple(T0 _val0, T1 _val1, T2 _val2, T3 _val3)
      : val0(_val0), val1(_val1), val2(_val2), val3(_val3) {}

  // Only valid when instantiated for pointer types
  __device__ void operator+=(nvfuser_index_t offset) {
    TUPLE_INCREMENT_PTR(0);
    TUPLE_INCREMENT_PTR(1);
    TUPLE_INCREMENT_PTR(2);
    TUPLE_INCREMENT_PTR(3);
  }
};

template <typename T0, typename T1, typename T2, typename T3, typename T4>
struct Tuple<T0, T1, T2, T3, T4> {
  T0 val0;
  T1 val1;
  T2 val2;
  T3 val3;
  T4 val4;

  Tuple() = default;

  __device__ Tuple(T0 _val0, T1 _val1, T2 _val2, T3 _val3, T4 _val4)
      : val0(_val0), val1(_val1), val2(_val2), val3(_val3), val4(_val4) {}

  // Only valid when instantiated for pointer types
  __device__ void operator+=(nvfuser_index_t offset) {
    TUPLE_INCREMENT_PTR(0);
    TUPLE_INCREMENT_PTR(1);
    TUPLE_INCREMENT_PTR(2);
    TUPLE_INCREMENT_PTR(3);
    TUPLE_INCREMENT_PTR(4);
  }
};

template <
    typename T0,
    typename T1,
    typename T2,
    typename T3,
    typename T4,
    typename T5>
struct Tuple<T0, T1, T2, T3, T4, T5> {
  T0 val0;
  T1 val1;
  T2 val2;
  T3 val3;
  T4 val4;
  T5 val5;

  Tuple() = default;

  __device__ Tuple(T0 _val0, T1 _val1, T2 _val2, T3 _val3, T4 _val4, T5 _val5)
      : val0(_val0),
        val1(_val1),
        val2(_val2),
        val3(_val3),
        val4(_val4),
        val5(_val5) {}

  // Only valid when instantiated for pointer types
  __device__ void operator+=(nvfuser_index_t offset) {
    TUPLE_INCREMENT_PTR(0);
    TUPLE_INCREMENT_PTR(1);
    TUPLE_INCREMENT_PTR(2);
    TUPLE_INCREMENT_PTR(3);
    TUPLE_INCREMENT_PTR(4);
    TUPLE_INCREMENT_PTR(5);
  }
};

template <
    typename T0,
    typename T1,
    typename T2,
    typename T3,
    typename T4,
    typename T5,
    typename T6>
struct Tuple<T0, T1, T2, T3, T4, T5, T6> {
  T0 val0;
  T1 val1;
  T2 val2;
  T3 val3;
  T4 val4;
  T5 val5;
  T6 val6;

  Tuple() = default;

  __device__ Tuple(
      T0 _val0,
      T1 _val1,
      T2 _val2,
      T3 _val3,
      T4 _val4,
      T5 _val5,
      T6 _val6)
      : val0(_val0),
        val1(_val1),
        val2(_val2),
        val3(_val3),
        val4(_val4),
        val5(_val5),
        val6(_val6) {}

  // Only valid when instantiated for pointer types
  __device__ void operator+=(nvfuser_index_t offset) {
    TUPLE_INCREMENT_PTR(0);
    TUPLE_INCREMENT_PTR(1);
    TUPLE_INCREMENT_PTR(2);
    TUPLE_INCREMENT_PTR(3);
    TUPLE_INCREMENT_PTR(4);
    TUPLE_INCREMENT_PTR(5);
    TUPLE_INCREMENT_PTR(6);
  }
};

template <
    typename T0,
    typename T1,
    typename T2,
    typename T3,
    typename T4,
    typename T5,
    typename T6,
    typename T7>
struct Tuple<T0, T1, T2, T3, T4, T5, T6, T7> {
  T0 val0;
  T1 val1;
  T2 val2;
  T3 val3;
  T4 val4;
  T5 val5;
  T6 val6;
  T7 val7;

  Tuple() = default;

  __device__ Tuple(
      T0 _val0,
      T1 _val1,
      T2 _val2,
      T3 _val3,
      T4 _val4,
      T5 _val5,
      T6 _val6,
      T7 _val7)
      : val0(_val0),
        val1(_val1),
        val2(_val2),
        val3(_val3),
        val4(_val4),
        val5(_val5),
        val6(_val6),
        val7(_val7) {}

  // Only valid when instantiated for pointer types
  __device__ void operator+=(nvfuser_index_t offset) {
    TUPLE_INCREMENT_PTR(0);
    TUPLE_INCREMENT_PTR(1);
    TUPLE_INCREMENT_PTR(2);
    TUPLE_INCREMENT_PTR(3);
    TUPLE_INCREMENT_PTR(4);
    TUPLE_INCREMENT_PTR(5);
    TUPLE_INCREMENT_PTR(6);
    TUPLE_INCREMENT_PTR(7);
  }
};

template <
    typename T0,
    typename T1,
    typename T2,
    typename T3,
    typename T4,
    typename T5,
    typename T6,
    typename T7,
    typename T8,
    typename T9,
    typename T10,
    typename T11,
    typename T12,
    typename T13,
    typename T14,
    typename T15>
struct Tuple<
    T0,
    T1,
    T2,
    T3,
    T4,
    T5,
    T6,
    T7,
    T8,
    T9,
    T10,
    T11,
    T12,
    T13,
    T14,
    T15> {
  T0 val0;
  T1 val1;
  T2 val2;
  T3 val3;
  T4 val4;
  T5 val5;
  T6 val6;
  T7 val7;
  T8 val8;
  T9 val9;
  T10 val10;
  T11 val11;
  T12 val12;
  T13 val13;
  T14 val14;
  T15 val15;

  Tuple() = default;

  __device__ Tuple(
      T0 _val0,
      T1 _val1,
      T2 _val2,
      T3 _val3,
      T4 _val4,
      T5 _val5,
      T6 _val6,
      T7 _val7,
      T8 _val8,
      T9 _val9,
      T10 _val10,
      T11 _val11,
      T12 _val12,
      T13 _val13,
      T14 _val14,
      T15 _val15)
      : val0(_val0),
        val1(_val1),
        val2(_val2),
        val3(_val3),
        val4(_val4),
        val5(_val5),
        val6(_val6),
        val7(_val7),
        val8(_val8),
        val9(_val9),
        val10(_val10),
        val11(_val11),
        val12(_val12),
        val13(_val13),
        val14(_val14),
        val15(_val15) {}

  // Only valid when instantiated for pointer types
  __device__ void operator+=(nvfuser_index_t offset) {
    TUPLE_INCREMENT_PTR(0);
    TUPLE_INCREMENT_PTR(1);
    TUPLE_INCREMENT_PTR(2);
    TUPLE_INCREMENT_PTR(3);
    TUPLE_INCREMENT_PTR(4);
    TUPLE_INCREMENT_PTR(5);
    TUPLE_INCREMENT_PTR(6);
    TUPLE_INCREMENT_PTR(7);
    TUPLE_INCREMENT_PTR(8);
    TUPLE_INCREMENT_PTR(9);
    TUPLE_INCREMENT_PTR(10);
    TUPLE_INCREMENT_PTR(11);
    TUPLE_INCREMENT_PTR(12);
    TUPLE_INCREMENT_PTR(13);
    TUPLE_INCREMENT_PTR(14);
    TUPLE_INCREMENT_PTR(15);
  }
};

#undef TUPLE_INCREMENT_PTR

// Accessor for Tuple
template <int idx>
struct get;

#define DEFINE_TUPLE_GET(idx)                              \
  template <>                                              \
  struct get<idx> {                                        \
    template <typename Tuple>                              \
    __device__ auto& operator()(Tuple& vals) {             \
      return vals.val##idx;                                \
    }                                                      \
    template <typename Tuple>                              \
    __device__ const auto& operator()(const Tuple& vals) { \
      return vals.val##idx;                                \
    }                                                      \
  };

DEFINE_TUPLE_GET(0);
DEFINE_TUPLE_GET(1);
DEFINE_TUPLE_GET(2);
DEFINE_TUPLE_GET(3);
DEFINE_TUPLE_GET(4);
DEFINE_TUPLE_GET(5);
DEFINE_TUPLE_GET(6);
DEFINE_TUPLE_GET(7);
DEFINE_TUPLE_GET(8);
DEFINE_TUPLE_GET(9);
DEFINE_TUPLE_GET(10);
DEFINE_TUPLE_GET(11);
DEFINE_TUPLE_GET(12);
DEFINE_TUPLE_GET(13);
DEFINE_TUPLE_GET(14);
DEFINE_TUPLE_GET(15);
#undef DEFINE_TUPLE_GET

template <typename DstType, typename SrcType>
__inline__ __device__ static void copyTuple(
    DstType& dst,
    nvfuser_index_t dst_offset,
    const SrcType& src,
    nvfuser_index_t src_offset = 0);

template <typename DstType, typename SrcType>
__inline__ __device__ static void copyTuple(
    DstType& dst,
    const SrcType& src,
    nvfuser_index_t src_offset = 0);

template <typename DstType>
__inline__ __device__ static void setTuple(
    DstType& dst,
    typename DstType::template ValType<0> src);

template <typename... Types>
class LocalTuple {
 public:
  static constexpr int num_vals = sizeof...(Types);
  using ValTypes = TypeList<Types...>;

  template <int idx>
  using ValType = typename TypeSelector<idx, Types...>::type;

  LocalTuple() = default;

  __device__ explicit LocalTuple(Types... args) : vals_(args...) {}

  __device__ LocalTuple(const LocalTuple& other) : vals_(other.vals_) {}

  template <template <typename...> typename TupleType>
  __device__ LocalTuple(const TupleType<Types...>& other) {
    copyTuple(*this, other);
  }

  __device__ LocalTuple& operator=(const LocalTuple<Types...>& other) {
    copyTuple(*this, other);
    return *this;
  }

  template <template <typename...> typename TupleType>
  __device__ LocalTuple& operator=(const TupleType<Types...>& other) {
    copyTuple(*this, other);
    return *this;
  }

  template <int val_idx>
  __device__ auto& val(nvfuser_index_t ptr_offset = 0) {
    static_assert(val_idx < num_vals, "Out-of-range value index");
    return get<val_idx>()(vals_);
  }

  template <int val_idx>
  __device__ const auto& val(nvfuser_index_t ptr_offset = 0) const {
    static_assert(val_idx < num_vals, "Out-of-range value index");
    return get<val_idx>()(vals_);
  }

 private:
  Tuple<Types...> vals_;
};

template <bool is_volatile, typename... Types>
class PtrTupleBase {
 public:
  static constexpr int num_vals = sizeof...(Types);
  using ValTypes = TypeList<Types...>;
  template <int idx>
  using ValType = typename TypeSelector<idx, Types...>::type;
  template <int val_idx>
  using TypeIMaybeVolatile = typename MaybeVolatile<
      typename TypeSelector<val_idx, Types...>::type,
      is_volatile>::type;

  __device__ PtrTupleBase(Types*... args) : vals_(args...) {}

  __device__ PtrTupleBase(const PtrTupleBase& other) : vals_(other.vals_) {}

  // Note: this is a deep copy
  __device__ PtrTupleBase& operator=(
      const PtrTupleBase<is_volatile, Types...>& other) {
    copyTuple(*this, other);
    return *this;
  }

  template <template <typename...> typename TupleType>
  __device__ PtrTupleBase& operator=(const TupleType<Types...>& other) {
    copyTuple(*this, other);
    return *this;
  }

  template <int val_idx>
  __device__ TypeIMaybeVolatile<val_idx>& val(nvfuser_index_t ptr_offset = 0) {
    static_assert(val_idx < num_vals, "Out-of-range value index");
    return ((TypeIMaybeVolatile<val_idx>*)get<val_idx>()(vals_))[ptr_offset];
  }

  template <int val_idx>
  __device__ const TypeIMaybeVolatile<val_idx>& val(
      nvfuser_index_t ptr_offset = 0) const {
    static_assert(val_idx < num_vals, "Out-of-range value index");
    return ((TypeIMaybeVolatile<val_idx>*)get<val_idx>()(vals_))[ptr_offset];
  }

  __device__ void operator+=(nvfuser_index_t ptr_offset) {
    vals_ += ptr_offset;
  }

 private:
  Tuple<Types*...> vals_;
};

template <typename... Types>
class RefTuple {
 public:
  static constexpr int num_vals = sizeof...(Types);
  using ValTypes = TypeList<Types...>;
  template <int idx>
  using ValType = typename TypeSelector<idx, Types...>::type;

  __device__ RefTuple(Types&... args) : vals_(args...) {}

  __device__ RefTuple(const RefTuple& other) : vals_(other.vals_) {}

  template <template <typename...> typename TupleType>
  __device__ RefTuple(const TupleType<Types...>& other) {
    copyTuple(*this, other);
  }

  __device__ RefTuple& operator=(const RefTuple<Types...>& other) {
    copyTuple(*this, other);
    return *this;
  }

  template <template <typename...> typename TupleType>
  __device__ RefTuple& operator=(const TupleType<Types...>& other) {
    copyTuple(*this, other);
    return *this;
  }

  template <int val_idx>
  __device__ auto& val(nvfuser_index_t ptr_offset = 0) {
    static_assert(val_idx < num_vals, "Out-of-range value index");
    return get<val_idx>()(vals_);
  }

  template <int val_idx>
  __device__ const auto& val(nvfuser_index_t ptr_offset = 0) const {
    static_assert(val_idx < num_vals, "Out-of-range value index");
    return get<val_idx>()(vals_);
  }

 private:
  Tuple<Types&...> vals_;
};

template <typename DstType, typename SrcType, int num_vals>
struct TupleCopy {
  __inline__ __device__ static void copy(
      DstType& dst,
      nvfuser_index_t dst_offset,
      const SrcType& src,
      nvfuser_index_t src_offset) {
    static_assert(
        IsSameType<typename DstType::ValTypes, typename SrcType::ValTypes>::
            value,
        "Invalid value types");
    TupleCopy<DstType, SrcType, num_vals - 1>::copy(
        dst, dst_offset, src, src_offset);
    dst.val<num_vals - 1>(dst_offset) = src.val<num_vals - 1>(src_offset);
  }
};

template <typename DstType, typename SrcType>
struct TupleCopy<DstType, SrcType, 0> {
  __inline__ __device__ static void copy(
      DstType& dst,
      nvfuser_index_t dst_offset,
      const SrcType& src,
      nvfuser_index_t src_offset) {}
};

template <typename DstType, typename SrcType>
__inline__ __device__ static void copyTuple(
    DstType& dst,
    nvfuser_index_t dst_offset,
    const SrcType& src,
    nvfuser_index_t src_offset) {
  static_assert(
      IsSameType<typename DstType::ValTypes, typename SrcType::ValTypes>::value,
      "Invalid value types");
  TupleCopy<DstType, SrcType, DstType::num_vals>::copy(
      dst, dst_offset, src, src_offset);
};

template <typename DstType, typename SrcType>
__inline__ __device__ static void copyTuple(
    DstType& dst,
    const SrcType& src,
    nvfuser_index_t src_offset) {
  copyTuple<DstType, SrcType>(dst, 0, src, src_offset);
};

template <typename DstType, int num_vals>
struct TupleSet {
  __inline__ __device__ static void set(
      DstType& dst,
      nvfuser_index_t dst_offset,
      typename DstType::template ValType<0> src) {
    static_assert(
        IsSameType<
            typename DstType::template ValType<num_vals - 1>,
            typename DstType::template ValType<0>>::value,
        "Invalid value types");
    TupleSet<DstType, num_vals - 1>::set(dst, dst_offset, src);
    dst.val<num_vals - 1>(dst_offset) = src;
  }
};

template <typename DstType>
struct TupleSet<DstType, 0> {
  __inline__ __device__ static void set(
      DstType& dst,
      nvfuser_index_t dst_offset,
      typename DstType::template ValType<0> src) {}
};

template <typename DstType>
__inline__ __device__ static void setTuple(
    DstType& dst,
    nvfuser_index_t dst_offset,
    typename DstType::template ValType<0> src) {
  TupleSet<DstType, DstType::num_vals>::set(dst, dst_offset, src);
};

template <typename DstType>
__inline__ __device__ static void setTuple(
    DstType& dst,
    typename DstType::template ValType<0> src) {
  setTuple(dst, 0, src);
};

template <typename DstType, typename SrcType, typename PredType, int num_vals>
struct PredicatedTupleCopy {
  __inline__ __device__ static void copy(
      DstType& dst,
      nvfuser_index_t dst_offset,
      const SrcType& src,
      nvfuser_index_t src_offset,
      const PredType& pred) {
    static_assert(
        IsSameType<typename PredType::template ValType<num_vals - 1>, bool>::
            value,
        "Invalid predicate type");
    PredicatedTupleCopy<DstType, SrcType, PredType, num_vals - 1>::copy(
        dst, dst_offset, src, src_offset, pred);
    if (pred.val<num_vals - 1>(0)) {
      dst.val<num_vals - 1>(dst_offset) = src.val<num_vals - 1>(src_offset);
    }
  }
};

template <typename DstType, typename SrcType, typename PredType>
struct PredicatedTupleCopy<DstType, SrcType, PredType, 0> {
  __inline__ __device__ static void copy(
      DstType& dst,
      nvfuser_index_t dst_offset,
      const SrcType& src,
      nvfuser_index_t src_offset,
      const PredType& pred) {}
};

template <typename DstType, typename SrcType, typename PredType>
__inline__ __device__ static void copyTupleIf(
    DstType& dst,
    nvfuser_index_t dst_offset,
    const SrcType& src,
    nvfuser_index_t src_offset,
    const PredType& pred) {
  static_assert(
      IsSameType<typename DstType::ValTypes, typename SrcType::ValTypes>::value,
      "Invalid value types");
  static_assert(
      PredType::num_vals == DstType::num_vals, "Invalid predicate type");
  PredicatedTupleCopy<DstType, SrcType, PredType, DstType::num_vals>::copy(
      dst, dst_offset, src, src_offset, pred);
};

template <typename DstType, typename SrcType, typename PredType>
__inline__ __device__ static void copyTupleIf(
    DstType& dst,
    const SrcType& src,
    nvfuser_index_t src_offset,
    const PredType& pred) {
  copyTupleIf(dst, 0, src, src_offset, pred);
};

template <typename DstType, typename SrcType, typename PredType>
__inline__ __device__ static void copyTupleIf(
    DstType& dst,
    const SrcType& src,
    const PredType& pred) {
  copyTupleIf(dst, 0, src, 0, pred);
};

// Can a generic const and non-const RefTupe be defined?
template <typename... Types>
class ConstRefTuple {
 public:
  static constexpr int num_vals = sizeof...(Types);
  using ValTypes = TypeList<Types...>;

  __device__ ConstRefTuple(const Types&... args) : vals_(args...) {}

  __device__ ConstRefTuple(const ConstRefTuple& other) : vals_(other.vals_) {}

  template <template <typename...> typename TupleType>
  __device__ ConstRefTuple(const TupleType<Types...>& other) {
    copyTuple(*this, other);
  }

  template <int val_idx>
  __device__ const auto& val(nvfuser_index_t ptr_offset = 0) const {
    static_assert(val_idx < num_vals, "Out-of-range value index");
    return get<val_idx>()(vals_);
  }

 private:
  Tuple<const Types&...> vals_;
};

template <typename... Types>
using PtrTuple = PtrTupleBase<false, Types...>;

template <typename... Types>
using VolatilePtrTuple = PtrTupleBase<true, Types...>;

// Define a LocalTuple of NumVals values of type Type
template <int NumVals, typename Type>
struct MakeLocalTuple;

template <typename Type>
struct MakeLocalTuple<1, Type> {
  using type = LocalTuple<Type>;
};

template <typename Type>
struct MakeLocalTuple<2, Type> {
  using type = LocalTuple<Type, Type>;
};

template <typename Type>
struct MakeLocalTuple<3, Type> {
  using type = LocalTuple<Type, Type, Type>;
};

template <typename Type>
struct MakeLocalTuple<4, Type> {
  using type = LocalTuple<Type, Type, Type, Type>;
};

template <typename Type>
struct MakeLocalTuple<5, Type> {
  using type = LocalTuple<Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeLocalTuple<6, Type> {
  using type = LocalTuple<Type, Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeLocalTuple<7, Type> {
  using type = LocalTuple<Type, Type, Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeLocalTuple<8, Type> {
  using type = LocalTuple<Type, Type, Type, Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeLocalTuple<16, Type> {
  using type = LocalTuple<
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type>;
};

template <int NumVals, typename Type>
struct MakeRefTuple;

template <typename Type>
struct MakeRefTuple<1, Type> {
  using type = RefTuple<Type>;
};

template <typename Type>
struct MakeRefTuple<2, Type> {
  using type = RefTuple<Type, Type>;
};

template <typename Type>
struct MakeRefTuple<3, Type> {
  using type = RefTuple<Type, Type, Type>;
};

template <typename Type>
struct MakeRefTuple<4, Type> {
  using type = RefTuple<Type, Type, Type, Type>;
};

template <typename Type>
struct MakeRefTuple<5, Type> {
  using type = RefTuple<Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeRefTuple<6, Type> {
  using type = RefTuple<Type, Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeRefTuple<7, Type> {
  using type = RefTuple<Type, Type, Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeRefTuple<8, Type> {
  using type = RefTuple<Type, Type, Type, Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeRefTuple<16, Type> {
  using type = RefTuple<
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type>;
};

template <int NumVals, typename Type>
struct MakeConstRefTuple;

template <typename Type>
struct MakeConstRefTuple<1, Type> {
  using type = ConstRefTuple<Type>;
};

template <typename Type>
struct MakeConstRefTuple<2, Type> {
  using type = ConstRefTuple<Type, Type>;
};

template <typename Type>
struct MakeConstRefTuple<3, Type> {
  using type = ConstRefTuple<Type, Type, Type>;
};

template <typename Type>
struct MakeConstRefTuple<4, Type> {
  using type = ConstRefTuple<Type, Type, Type, Type>;
};

template <typename Type>
struct MakeConstRefTuple<5, Type> {
  using type = ConstRefTuple<Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeConstRefTuple<6, Type> {
  using type = ConstRefTuple<Type, Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeConstRefTuple<7, Type> {
  using type = ConstRefTuple<Type, Type, Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeConstRefTuple<8, Type> {
  using type = ConstRefTuple<Type, Type, Type, Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeConstRefTuple<16, Type> {
  using type = ConstRefTuple<
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type>;
};

template <int NumVals, typename Type>
struct MakeVolatilePtrTuple;

template <typename Type>
struct MakeVolatilePtrTuple<1, Type> {
  using type = VolatilePtrTuple<Type>;
};

template <typename Type>
struct MakeVolatilePtrTuple<2, Type> {
  using type = VolatilePtrTuple<Type, Type>;
};

template <typename Type>
struct MakeVolatilePtrTuple<3, Type> {
  using type = VolatilePtrTuple<Type, Type, Type>;
};

template <typename Type>
struct MakeVolatilePtrTuple<4, Type> {
  using type = VolatilePtrTuple<Type, Type, Type, Type>;
};

template <typename Type>
struct MakeVolatilePtrTuple<5, Type> {
  using type = VolatilePtrTuple<Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeVolatilePtrTuple<6, Type> {
  using type = VolatilePtrTuple<Type, Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeVolatilePtrTuple<7, Type> {
  using type = VolatilePtrTuple<Type, Type, Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeVolatilePtrTuple<8, Type> {
  using type = VolatilePtrTuple<Type, Type, Type, Type, Type, Type, Type, Type>;
};

template <typename Type>
struct MakeVolatilePtrTuple<16, Type> {
  using type = VolatilePtrTuple<
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type,
      Type>;
};

// Utility definitions. Currently only used with LocalTuple

template <int idx, typename BinaryFunc, typename... DataTypes>
struct TupleBinaryOp {
  static __inline__ __device__ void apply(
      BinaryFunc func,
      const LocalTuple<DataTypes...>& lhs,
      const LocalTuple<DataTypes...>& rhs,
      LocalTuple<DataTypes...>& result) {
    TupleBinaryOp<idx - 1, BinaryFunc, DataTypes...>::apply(
        func, lhs, rhs, result);
    result.val<idx - 1>(0) = func(lhs.val<idx - 1>(0), rhs.val<idx - 1>(0));
  }
};

template <typename BinaryFunc, typename... DataTypes>
struct TupleBinaryOp<0, BinaryFunc, DataTypes...> {
  static __inline__ __device__ void apply(
      BinaryFunc func,
      const LocalTuple<DataTypes...>& lhs,
      const LocalTuple<DataTypes...>& rhs,
      LocalTuple<DataTypes...>& result) {}
};

template <typename BinaryFunc, typename... DataTypes>
__inline__ __device__ LocalTuple<DataTypes...> apply(
    BinaryFunc func,
    const LocalTuple<DataTypes...>& lhs,
    const LocalTuple<DataTypes...>& rhs) {
  LocalTuple<DataTypes...> result = lhs;
  TupleBinaryOp<sizeof...(DataTypes), BinaryFunc, DataTypes...>::apply(
      func, result, rhs, result);
  return result;
}

template <typename... BoolTypes>
__inline__ __device__ LocalTuple<BoolTypes...> operator&&(
    const LocalTuple<BoolTypes...>& lhs,
    const LocalTuple<BoolTypes...>& rhs) {
  return apply([](bool x, bool y) { return x && y; }, lhs, rhs);
}

template <typename... BoolTypes>
__inline__ __device__ LocalTuple<BoolTypes...> operator&&(
    bool lhs,
    const LocalTuple<BoolTypes...>& rhs) {
  LocalTuple<BoolTypes...> lhs_tuple;
  setTuple(lhs_tuple, lhs);
  return lhs_tuple && rhs;
}

template <typename... BoolTypes>
__inline__ __device__ LocalTuple<BoolTypes...> operator&&(
    const LocalTuple<BoolTypes...>& lhs,
    bool rhs) {
  LocalTuple<BoolTypes...> rhs_tuple;
  setTuple(rhs_tuple, rhs);
  return lhs && rhs_tuple;
}
