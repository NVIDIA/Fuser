
#include <hip/hip_runtime.h>
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
#ifdef __NVCC__
#include <bit>
#else

namespace std {

template <class To, class From>
std::enable_if_t<sizeof(To) == sizeof(From), To> bit_cast(
    const From& src) noexcept {
  return *reinterpret_cast<const To*>(&src);
}

} // namespace std

#endif
