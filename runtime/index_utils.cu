// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
namespace index_utils {

// Utility functions

// Total size of provided dimension
template <typename _dim3>
__device__ __forceinline__ nvfuser_index_t size(const _dim3& d) {
  return (nvfuser_index_t)d.x * (nvfuser_index_t)d.y * (nvfuser_index_t)d.z;
}

// Linearized indexing of idx based on dim, if bool==false that dimension does
// not participate
template <bool X, bool Y, bool Z, typename _dim3, typename _dim3_2>
__device__ nvfuser_index_t maskedOffset(const _dim3& idx, const _dim3_2& dim) {
  nvfuser_index_t offset = 0;
  if (Z)
    offset += idx.z;
  if (Y)
    offset = offset * dim.y + idx.y;
  if (X)
    offset = offset * dim.x + idx.x;
  return offset;
}

// Linearized indexing of idx based on dim. All dimensions participate.
template <typename _dim3, typename _dim3_2>
__device__ nvfuser_index_t offset(const _dim3& idx, const _dim3_2& dim) {
  nvfuser_index_t offset = idx.z;
  offset = offset * dim.y + idx.y;
  offset = offset * dim.x + idx.x;
  return offset;
}

// Masks the provided dim3, those == false get truncated to 1
template <bool X, bool Y, bool Z, typename _dim3>
__device__ dim3 maskedDims(const _dim3& dim) {
  return dim3{
      X ? (unsigned)dim.x : 1U,
      Y ? (unsigned)dim.y : 1U,
      Z ? (unsigned)dim.z : 1U};
}

// Provides total size of dim with masking, those dims == false do not
// participate in the size calculation
template <bool X_BLOCK, bool Y_BLOCK, bool Z_BLOCK, typename _dim3>
__device__ nvfuser_index_t maskedSize(const _dim3& dim) {
  return size(maskedDims<X_BLOCK, Y_BLOCK, Z_BLOCK>(dim));
}

// Checks if provided idx is zero on those dims == true
template <bool X, bool Y, bool Z, typename _dim3>
__device__ bool maskedIsZero(const _dim3& idx) {
  bool isZero = true;
  if (X)
    isZero = isZero && idx.x == 0;
  if (Y)
    isZero = isZero && idx.y == 0;
  if (Z)
    isZero = isZero && idx.z == 0;
  return isZero;
}

// Checks if provided idx is zero on those dims == true
template <bool X, bool Y, bool Z, typename _dim3, typename _dim3_2>
__device__ bool maskedIsLast(const _dim3& idx, const _dim3_2& dim) {
  bool isZero = true;
  if (X)
    isZero = isZero && idx.x == dim.x - 1;
  if (Y)
    isZero = isZero && idx.y == dim.y - 1;
  if (Z)
    isZero = isZero && idx.z == dim.z - 1;
  return isZero;
}

} // namespace index_utils
