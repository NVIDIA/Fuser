#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
#define NVFUSER_DEFINE_MAGIC_ZERO          \
  __shared__ int nvfuser_zero_s;           \
  if (threadIdx.x == 0)                    \
    nvfuser_zero_s = 0;                    \
  __syncthreads();                         \
  atomicMin(&nvfuser_zero_s, threadIdx.x); \
  int nvfuser_zero = nvfuser_zero_s;

#define NVFUSER_UPDATE_MAGIC_ZERO \
  do {                            \
    nvfuser_zero <<= 1;           \
  } while (0);

#ifdef __NVCC__
#include <assert.h>
#endif // __NVCC__

__device__ constexpr int ceilDiv(int a, int b) {
  return (a + b - 1) / b;
}

__device__ constexpr int64_t ceilDiv(int64_t a, int64_t b) {
  return (a + b - 1) / b;
}

__device__ constexpr int64_t ceilDiv(int64_t a, int b) {
  return ceilDiv(a, (int64_t)b);
}

__device__ constexpr int64_t ceilDiv(int a, int64_t b) {
  return ceilDiv((int64_t)a, b);
}

__device__ constexpr double ceilDiv(double a, double b) {
  return std::ceil(a / b);
}

__device__ constexpr double ceilDiv(double a, int64_t b) {
  return std::ceil(a / b);
}

__device__ constexpr double ceilDiv(int64_t a, double b) {
  return std::ceil(a / b);
}

// Monotonic and precise lerp is described here:
// https://math.stackexchange.com/a/1798323
__device__ double lerp(double start, double end, double weight) {
  if (weight < 0.5) {
    return start + weight * (end - start);
  } else {
    return end - (end - start) * (1.0 - weight);
  }
}

__device__ float lerp(float start, float end, float weight) {
  if (weight < 0.5f) {
    return start + weight * (end - start);
  } else {
    return end - (end - start) * (1.0f - weight);
  }
}

__device__ float lerp(float start, float end, double weight) {
  return lerp(start, end, static_cast<float>(weight));
}

__device__ constexpr int max(int a, int b) {
  return a > b ? a : b;
}

__device__ constexpr int64_t max(int64_t a, int b) {
  return a > (int64_t)b ? a : (int64_t)b;
}

__device__ constexpr int64_t max(int a, int64_t b) {
  return (int64_t)a > b ? (int64_t)a : b;
}

__device__ constexpr int64_t max(int64_t a, int64_t b) {
  return a > b ? a : b;
}

__device__ double fmax(double a, double b) {
  // check and propagate NaN
  if (a != a) {
    return a;
  } else { // If b is nan, it will be returned in the next line
    return a > b ? a : b;
  }
}

__device__ float fmax(float a, float b) {
  // check and propagate NaN
  if (a != a) {
    return a;
  } else { // If b is nan, it will be returned in the next line
    return a > b ? a : b;
  }
}

__device__ constexpr int min(int a, int b) {
  return a > b ? b : a;
}

__device__ constexpr int64_t min(int64_t a, int b) {
  return (int64_t)a > b ? b : (int64_t)a;
}

__device__ constexpr int64_t min(int a, int64_t b) {
  return a > (int64_t)b ? (int64_t)b : a;
}

__device__ constexpr int64_t min(int64_t a, int64_t b) {
  return a > b ? b : a;
}

__device__ double fmin(double a, double b) {
  // check and propagate NaN
  if (b != b) {
    return b;
  } else { // If a is nan, it will be returned in the next line
    return a > b ? b : a;
  }
}

__device__ float fmin(float a, float b) {
  // check and propagate NaN
  if (b != b) {
    return b;
  } else { // If a is nan, it will be returned in the next line
    return a > b ? b : a;
  }
}

__device__ constexpr int alignBufferSize(int buffer, int size) {
  return (buffer + (size - 1)) & ~(size - 1);
}

__device__ double clamp(double x, double minv, double maxv) {
  return fmin(fmax(x, minv), maxv);
}

__device__ float clamp(float x, double minv, double maxv) {
  return fmin(fmax((double)x, minv), maxv);
}

__device__ int clamp(int x, int64_t minv, int64_t maxv) {
  return min(max((int64_t)x, minv), maxv);
}

__device__ int64_t clamp(int64_t x, int64_t minv, int64_t maxv) {
  return min(max(x, minv), maxv);
}

__device__ double frac(double x) {
  return x - trunc(x);
}

__device__ float frac(float x) {
  return x - trunc(x);
}

__device__ double reciprocal(double x) {
  return 1 / x;
}

__device__ float reciprocal(float x) {
  return 1 / x;
}

__device__ double relu(double x) {
  return x <= 0 ? 0 : x;
}

__device__ float relu(float x) {
  return x <= 0 ? 0 : x;
}

__device__ float relu(int64_t x) {
  return x <= 0 ? 0 : x;
}

__device__ float relu(int x) {
  return x <= 0 ? 0 : x;
}

__device__ double remainder(double a, double b) {
  auto mod = ::fmod(a, b);
  if ((mod != 0) && ((b < 0) != (mod < 0)))
    mod += b;
  return mod;
}

__device__ float remainder(float a, float b) {
  auto mod = ::fmod(a, b);
  if ((mod != 0) && ((b < 0) != (mod < 0)))
    mod += b;
  return mod;
}

__device__ double sigmoid(double x) {
  return 1.0 / (1.0 + exp(-x));
}

__device__ float sigmoid(float x) {
  return 1.0f / (1.0f + exp(-x));
}

__device__ double silu(double x) {
  return x * sigmoid(x);
}

__device__ float silu(float x) {
  return x * sigmoid(x);
}

__device__ double threshold(double x, double t, double v) {
  return x <= t ? v : x;
}

__device__ float threshold(float x, double t, double v) {
  return x <= t ? v : x;
}

__device__ int threshold(int x, int64_t t, int64_t v) {
  return x <= t ? v : x;
}

__device__ int64_t threshold(int64_t x, int64_t t, int64_t v) {
  return x <= t ? v : x;
}

__device__ constexpr int64_t remainder(int64_t a, int64_t b) {
  auto mod = a % b;
  if ((mod != 0) && ((b < 0) != (mod < 0)))
    mod += b;
  return mod;
}

__device__ constexpr int remainder(int a, int b) {
  auto mod = a % b;
  if ((mod != 0) && ((b < 0) != (mod < 0)))
    mod += b;
  return mod;
}

__device__ constexpr int64_t fmod(int64_t a, int64_t b) {
  return a % b;
}

__device__ constexpr int fmod(int a, int b) {
  return a % b;
}

__device__ constexpr double fmod(double a, double b) {
  return ::fmod(a, b);
}

__device__ constexpr float fmod(float a, float b) {
  return ::fmod(a, b);
}

__device__ constexpr double nextafter(double a, double b) {
  return ::nextafter(a, b);
}

__device__ constexpr float nextafter(float a, float b) {
  return ::nextafterf(a, b);
}

template <typename T>
__device__ T pow(T a, T b) {
  if (b < 0) {
    if (a == 1) {
      return 1;
    } else if (a == -1) {
      auto negative = (-b) % static_cast<T>(2);
      return negative ? -1 : 1;
    } else {
      return 0;
    }
  } else {
    T result = 1;
    while (b) {
      if (b & 1) {
        result *= a;
      }
      b /= 2;
      a *= a;
    }
    return result;
  }
}

template __device__ int pow<int>(int a, int b);
template __device__ int64_t pow<int64_t>(int64_t a, int64_t b);

template <>
__device__ float pow<float>(float a, float b) {
  return ::pow(a, b);
}

template <>
__device__ double pow<double>(double a, double b) {
  return ::pow(a, b);
}

__device__ float pow(float a, int b) {
  return pow(a, (float)b);
}

__device__ double pow(double a, int b) {
  return pow(a, (double)b);
}

__device__ float pow(float a, int64_t b) {
  return pow(a, (float)b);
}

__device__ double pow(double a, int64_t b) {
  return pow(a, (double)b);
}

__device__ int64_t pow(int64_t a, int b) {
  return pow(a, (int64_t)b);
}

__device__ int64_t pow(int a, int64_t b) {
  return pow((int64_t)a, b);
}

__device__ double rsqrt(double z) {
  return ::rsqrt(z);
}

__device__ float rsqrt(float z) {
  return ::rsqrtf(z);
}

__device__ int rsqrt(int z) {
  return ::rsqrtf((float)z);
}

__device__ int64_t rsqrt(int64_t z) {
  return ::rsqrt((double)z);
}

__device__ double signbit(double a) {
  return ::signbit(a);
}

__device__ float signbit(float a) {
  return ::signbit(a);
}

__device__ int signbit(int a) {
  return a < 0;
}

__device__ int64_t signbit(int64_t a) {
  return a < 0;
}

// Reference:
// https://en.wikipedia.org/wiki/Euclidean_algorithm#Implementations
// https://github.com/pytorch/pytorch/blob/c9f4f01981fd73fcc7c27676cc50230cd1b5bc22/aten/src/ATen/native/Math.h#L1232
template <typename T>
__device__ T gcd(T a, T b) {
  a = abs(a);
  b = abs(b);
  while (b != 0) {
    auto t = b;
    b = a % b;
    a = t;
  }
  return a;
}

template <int size, int align = size>
struct alignas(align) TypelessData {
  int8_t data[size];

  template <typename T, std::enable_if_t<sizeof(T) == size, int> _ = 0>
  TypelessData(T x) {
    *reinterpret_cast<T*>(data) = x;
  }

  template <typename T, std::enable_if_t<sizeof(T) == size, int> _ = 0>
  operator T() {
    return *reinterpret_cast<T*>(data);
  }
};

template <typename T>
TypelessData<sizeof(T), alignof(T)> erase_type(T x) {
  return x;
}

template <typename T>
bool isfinite(T x) {
  return ::isfinite(x);
}

// ref:
// https://github.com/NVIDIA/cutlass/blob/6fbc0d33800008d3180d3fefed4e1a653e5f72a0/include/cutlass/bfloat16.h#L213
template <>
bool isfinite<__bfloat>(__bfloat x) {
  const auto exponent_biased = int((x.raw() >> 7) & 0x0ff);
  return exponent_biased != 0x0ff;
}

// ref:
// https://github.com/NVIDIA/cutlass/blob/6fbc0d33800008d3180d3fefed4e1a653e5f72a0/include/cutlass/half.h#L511
template <>
bool isfinite<__half>(__half x) {
  const auto exponent_biased = int((x.raw() >> 10) & 0x1f);
  return exponent_biased != 0x1f;
}

template <typename T>
bool isinf(T x) {
  return ::isinf(x);
}

////////////////////////////////////////////////////////////
// TODO: the following overloads are only needed for CUDA //
// 10.2 Please remove when CUDA 10.2 support is dropped   //
////////////////////////////////////////////////////////////

bool isinf(int64_t x) {
  return false;
}

bool isinf(int x) {
  return false;
}

bool isinf(short x) {
  return false;
}

bool isinf(char x) {
  return false;
}

bool isinf(unsigned char x) {
  return false;
}

bool isinf(bool x) {
  return false;
}

bool isfinite(int64_t x) {
  return true;
}

bool isfinite(int x) {
  return true;
}

bool isfinite(short x) {
  return true;
}

bool isfinite(char x) {
  return true;
}

bool isfinite(unsigned char x) {
  return true;
}

bool isfinite(bool x) {
  return true;
}

////////////////////////////////////////////////////////////
//                        End TODO                        //
////////////////////////////////////////////////////////////

template <typename T>
bool isnan(T x) {
  return x != x;
}

template <typename T>
bool isneginf(T x) {
  return x < 0 && isinf(x);
}

template <typename T>
bool isposinf(T x) {
  return x > 0 && isinf(x);
}

template <typename T>
bool isreal(T x) {
  return true;
}

// Return the current value of the cycle counter
__device__ inline int64_t readCycleCounter() {
  // Ensures preceding memory operations are completed. Doing this
  // would make sense for measuring elapsed times enclosed with this
  // function.
  __threadfence();
  return clock64();
}

__device__ float print_impl(const char* name, float value) {
  printf(
      "%s = %f @ threadIdx=(%d,%d,%d), blockIdx=(%d,%d,%d)\n",
      name,
      value,
      (int)threadIdx.x,
      (int)threadIdx.y,
      (int)threadIdx.z,
      (int)blockIdx.x,
      (int)blockIdx.y,
      (int)blockIdx.z);
  return value;
}

__device__ double print_impl(const char* name, double value) {
  printf(
      "%s = %lf @ threadIdx=(%d,%d,%d), blockIdx=(%d,%d,%d)\n",
      name,
      value,
      (int)threadIdx.x,
      (int)threadIdx.y,
      (int)threadIdx.z,
      (int)blockIdx.x,
      (int)blockIdx.y,
      (int)blockIdx.z);
  return value;
}

__device__ int print_impl(const char* name, int value) {
  printf(
      "%s = %d @ threadIdx=(%d,%d,%d), blockIdx=(%d,%d,%d)\n",
      name,
      value,
      (int)threadIdx.x,
      (int)threadIdx.y,
      (int)threadIdx.z,
      (int)blockIdx.x,
      (int)blockIdx.y,
      (int)blockIdx.z);
  return value;
}

__device__ int64_t print_impl(const char* name, int64_t value) {
  printf(
      "%s = %ld @ threadIdx=(%d,%d,%d), blockIdx=(%d,%d,%d)\n",
      name,
      value,
      (int)threadIdx.x,
      (int)threadIdx.y,
      (int)threadIdx.z,
      (int)blockIdx.x,
      (int)blockIdx.y,
      (int)blockIdx.z);
  return value;
}

__device__ bool print_impl(const char* name, bool value) {
  printf(
      "%s = %s @ threadIdx=(%d,%d,%d), blockIdx=(%d,%d,%d)\n",
      name,
      value ? "true" : "false",
      (int)threadIdx.x,
      (int)threadIdx.y,
      (int)threadIdx.z,
      (int)blockIdx.x,
      (int)blockIdx.y,
      (int)blockIdx.z);
  return value;
}

__device__ __half print_impl(const char* name, __half value) {
  printf(
      "%s = %f @ threadIdx=(%d,%d,%d), blockIdx=(%d,%d,%d)\n",
      name,
      __half2float(value),
      (int)threadIdx.x,
      (int)threadIdx.y,
      (int)threadIdx.z,
      (int)blockIdx.x,
      (int)blockIdx.y,
      (int)blockIdx.z);
  return value;
}

#if __CUDACC_VER_MAJOR__ >= 11
__device__ __bfloat print_impl(const char* name, __bfloat value) {
  printf(
      "%s = %f @ threadIdx=(%d,%d,%d), blockIdx=(%d,%d,%d)\n",
      name,
      __bfloat2float(value),
      (int)threadIdx.x,
      (int)threadIdx.y,
      (int)threadIdx.z,
      (int)blockIdx.x,
      (int)blockIdx.y,
      (int)blockIdx.z);
  return value;
}
#endif

#define print(...) print_impl(#__VA_ARGS__, (__VA_ARGS__))
