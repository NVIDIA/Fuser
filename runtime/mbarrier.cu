
#include <hip/hip_runtime.h>
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on

// Reference:
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#asynchronous-barrier
// https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#parallel-synchronization-and-communication-instructions-mbarrier
// https://github.com/NVIDIA/cutlass/blob/main/include/cute/arch/copy_sm90_desc.hpp

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))

namespace mbarrier {

__device__ inline void init(
    uint32_t smem_barrier_ptr,
    uint32_t thread_count = 1) {
  asm volatile(
      "mbarrier.init.shared.b64 [%0], %1;\n" ::"r"(smem_barrier_ptr),
      "r"(thread_count));
}

__device__ inline void inval(uint32_t smem_barrier_ptr) {
  asm volatile("mbarrier.inval.shared.b64 [%0];\n" ::"r"(smem_barrier_ptr));
}

__device__ inline uint64_t arrive(uint32_t smem_barrier_ptr) {
  volatile uint64_t state;
  asm volatile("mbarrier.arrive.shared.b64 %0, [%1];\n"
               : "=l"(state)
               : "r"(smem_barrier_ptr));
  return state;
}

__device__ inline void wait(uint32_t smem_barrier_ptr, uint64_t state) {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
  asm volatile(
      "{\n"
      ".reg .pred                complete;\n"
      "waitLoop:\n"
      "mbarrier.try_wait.shared.b64 complete, [%0], %1;\n"
      "@!complete bra waitLoop;\n"
      "}\n" ::"r"(smem_barrier_ptr),
      "l"(state));
#else
  asm volatile(
      "{\n"
      ".reg .pred                P1;\n"
      "LAB_WAIT:\n"
      "mbarrier.test_wait.shared.b64 P1, [%0], %1;\n"
      "@P1                       bra.uni DONE;\n"
      "nanosleep.u32 20;\n"
      "bra.uni                   LAB_WAIT;\n"
      "DONE:\n"
      "}\n" ::"r"(smem_barrier_ptr),
      "l"(state));
#endif
}

} // namespace mbarrier

#endif // (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))
