#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
#ifndef __NVCC__
#define POS_INFINITY __int_as_float(0x7f800000)
#define INFINITY POS_INFINITY
#define NEG_INFINITY __int_as_float(0xff800000)
#define NAN __int_as_float(0x7fffffff)
//===----------------------------------------------------------------------===//
// The following namespace std is modified from LLVM, see the following copyright
// information
//
// -*- C++ -*-
//===----------------------------------------------------------------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
// copy-pasted from the following llvm file:
// https://github.com/llvm/llvm-project/blob/main/libcxx/include/complex
namespace std {

template <class _Tp>
class complex;

template <class _Tp>
complex<_Tp> operator*(const complex<_Tp>& __z, const complex<_Tp>& __w);
template <class _Tp>
complex<_Tp> operator/(const complex<_Tp>& __x, const complex<_Tp>& __y);

template <class _Tp>
class complex {
 public:
  typedef _Tp value_type;

 private:
  value_type __re_;
  value_type __im_;

 public:
  constexpr complex(
      const value_type& __re = value_type(),
      const value_type& __im = value_type())
      : __re_(__re), __im_(__im) {}
  template <class _Xp>
  constexpr complex(const complex<_Xp>& __c)
      : __re_(__c.real()), __im_(__c.imag()) {}

  constexpr value_type real() const {
    return __re_;
  }
  constexpr value_type imag() const {
    return __im_;
  }

  void real(value_type __re) {
    __re_ = __re;
  }
  void imag(value_type __im) {
    __im_ = __im;
  }

  constexpr operator bool() const {
    return real() || imag();
  }

  complex& operator=(const value_type& __re) {
    __re_ = __re;
    __im_ = value_type();
    return *this;
  }
  complex& operator+=(const value_type& __re) {
    __re_ += __re;
    return *this;
  }
  complex& operator-=(const value_type& __re) {
    __re_ -= __re;
    return *this;
  }
  complex& operator*=(const value_type& __re) {
    __re_ *= __re;
    __im_ *= __re;
    return *this;
  }
  complex& operator/=(const value_type& __re) {
    __re_ /= __re;
    __im_ /= __re;
    return *this;
  }

  template <class _Xp>
  complex& operator=(const complex<_Xp>& __c) {
    __re_ = __c.real();
    __im_ = __c.imag();
    return *this;
  }
  template <class _Xp>
  complex& operator+=(const complex<_Xp>& __c) {
    __re_ += __c.real();
    __im_ += __c.imag();
    return *this;
  }
  template <class _Xp>
  complex& operator-=(const complex<_Xp>& __c) {
    __re_ -= __c.real();
    __im_ -= __c.imag();
    return *this;
  }
  template <class _Xp>
  complex& operator*=(const complex<_Xp>& __c) {
    *this = *this * complex(__c.real(), __c.imag());
    return *this;
  }
  template <class _Xp>
  complex& operator/=(const complex<_Xp>& __c) {
    *this = *this / complex(__c.real(), __c.imag());
    return *this;
  }
};

template <>
class complex<double>;

template <>
class complex<float> {
  float __re_;
  float __im_;

 public:
  typedef float value_type;

  constexpr complex(float __re = 0.0f, float __im = 0.0f)
      : __re_(__re), __im_(__im) {}

  explicit constexpr complex(const complex<double>& __c);

  // copy volatile to non-volatile
  constexpr complex(const volatile complex<float>& other)
      : __re_(other.__re_), __im_(other.__im_) {}

  constexpr complex(const complex<float>& other)
      : __re_(other.__re_), __im_(other.__im_) {}

  constexpr float real() const {
    return __re_;
  }
  constexpr float imag() const {
    return __im_;
  }

  void real(value_type __re) {
    __re_ = __re;
  }
  void imag(value_type __im) {
    __im_ = __im;
  }

  constexpr operator bool() const {
    return real() || imag();
  }

  complex& operator=(float __re) {
    __re_ = __re;
    __im_ = value_type();
    return *this;
  }
  complex& operator+=(float __re) {
    __re_ += __re;
    return *this;
  }
  complex& operator-=(float __re) {
    __re_ -= __re;
    return *this;
  }
  complex& operator*=(float __re) {
    __re_ *= __re;
    __im_ *= __re;
    return *this;
  }
  complex& operator/=(float __re) {
    __re_ /= __re;
    __im_ /= __re;
    return *this;
  }

  template <class _Xp>
  complex& operator=(const complex<_Xp>& __c) {
    __re_ = __c.real();
    __im_ = __c.imag();
    return *this;
  }

  // non-volatile to volatile
  template <class _Xp>
  volatile complex& operator=(const complex<_Xp>& __c) volatile {
    __re_ = __c.real();
    __im_ = __c.imag();
    return *this;
  }
  // volatile to non-volatile
  template <class _Xp>
  complex& operator=(const volatile complex<_Xp>& __c) {
    __re_ = __c.real();
    __im_ = __c.imag();
    return *this;
  }
  // volatile to volatile
  template <class _Xp>
  volatile complex& operator=(const volatile complex<_Xp>& __c) volatile {
    __re_ = __c.real();
    __im_ = __c.imag();
    return *this;
  }

  template <class _Xp>
  complex& operator+=(const complex<_Xp>& __c) {
    __re_ += __c.real();
    __im_ += __c.imag();
    return *this;
  }
  template <class _Xp>
  complex& operator-=(const complex<_Xp>& __c) {
    __re_ -= __c.real();
    __im_ -= __c.imag();
    return *this;
  }
  template <class _Xp>
  complex& operator*=(const complex<_Xp>& __c) {
    *this = *this * complex(__c.real(), __c.imag());
    return *this;
  }
  template <class _Xp>
  complex& operator/=(const complex<_Xp>& __c) {
    *this = *this / complex(__c.real(), __c.imag());
    return *this;
  }
};

template <>
class complex<double> {
  double __re_;
  double __im_;

 public:
  typedef double value_type;

  constexpr complex(double __re = 0.0, double __im = 0.0)
      : __re_(__re), __im_(__im) {}

  constexpr complex(const complex<float>& __c);

  // copy volatile to non-volatile
  constexpr complex(const volatile complex<double>& other)
      : __re_(other.__re_), __im_(other.__im_) {}

  constexpr complex(const complex<double>& other)
      : __re_(other.__re_), __im_(other.__im_) {}

  constexpr double real() const {
    return __re_;
  }
  constexpr double imag() const {
    return __im_;
  }

  void real(value_type __re) {
    __re_ = __re;
  }
  void imag(value_type __im) {
    __im_ = __im;
  }

  constexpr operator bool() const {
    return real() || imag();
  }

  complex& operator=(double __re) {
    __re_ = __re;
    __im_ = value_type();
    return *this;
  }
  complex& operator+=(double __re) {
    __re_ += __re;
    return *this;
  }
  complex& operator-=(double __re) {
    __re_ -= __re;
    return *this;
  }
  complex& operator*=(double __re) {
    __re_ *= __re;
    __im_ *= __re;
    return *this;
  }
  complex& operator/=(double __re) {
    __re_ /= __re;
    __im_ /= __re;
    return *this;
  }

  template <class _Xp>
  complex& operator=(const complex<_Xp>& __c) {
    __re_ = __c.real();
    __im_ = __c.imag();
    return *this;
  }

  // non-volatile to volatile
  template <class _Xp>
  volatile complex& operator=(const complex<_Xp>& __c) volatile {
    __re_ = __c.real();
    __im_ = __c.imag();
    return *this;
  }
  // volatile to non-volatile
  template <class _Xp>
  complex& operator=(const volatile complex<_Xp>& __c) {
    __re_ = __c.real();
    __im_ = __c.imag();
    return *this;
  }
  // volatile to volatile
  template <class _Xp>
  volatile complex& operator=(const volatile complex<_Xp>& __c) volatile {
    __re_ = __c.real();
    __im_ = __c.imag();
    return *this;
  }

  template <class _Xp>
  complex& operator+=(const complex<_Xp>& __c) {
    __re_ += __c.real();
    __im_ += __c.imag();
    return *this;
  }
  template <class _Xp>
  complex& operator-=(const complex<_Xp>& __c) {
    __re_ -= __c.real();
    __im_ -= __c.imag();
    return *this;
  }
  template <class _Xp>
  complex& operator*=(const complex<_Xp>& __c) {
    *this = *this * complex(__c.real(), __c.imag());
    return *this;
  }
  template <class _Xp>
  complex& operator/=(const complex<_Xp>& __c) {
    *this = *this / complex(__c.real(), __c.imag());
    return *this;
  }
};

inline constexpr complex<float>::complex(const complex<double>& __c)
    : __re_(__c.real()), __im_(__c.imag()) {}

inline constexpr complex<double>::complex(const complex<float>& __c)
    : __re_(__c.real()), __im_(__c.imag()) {}

// 26.3.6 operators:

template <class _Tp>
inline complex<_Tp> operator+(
    const complex<_Tp>& __x,
    const complex<_Tp>& __y) {
  complex<_Tp> __t(__x);
  __t += __y;
  return __t;
}

template <class _Tp>
inline complex<_Tp> operator+(const complex<_Tp>& __x, const _Tp& __y) {
  complex<_Tp> __t(__x);
  __t += __y;
  return __t;
}

template <class _Tp>
inline complex<_Tp> operator+(const _Tp& __x, const complex<_Tp>& __y) {
  complex<_Tp> __t(__y);
  __t += __x;
  return __t;
}

template <class _Tp>
inline complex<_Tp> operator-(
    const complex<_Tp>& __x,
    const complex<_Tp>& __y) {
  complex<_Tp> __t(__x);
  __t -= __y;
  return __t;
}

template <class _Tp>
inline complex<_Tp> operator-(const complex<_Tp>& __x, const _Tp& __y) {
  complex<_Tp> __t(__x);
  __t -= __y;
  return __t;
}

template <class _Tp>
inline complex<_Tp> operator-(const _Tp& __x, const complex<_Tp>& __y) {
  complex<_Tp> __t(-__y);
  __t += __x;
  return __t;
}

template <class _Tp>
complex<_Tp> operator*(const complex<_Tp>& __z, const complex<_Tp>& __w) {
  _Tp __a = __z.real();
  _Tp __b = __z.imag();
  _Tp __c = __w.real();
  _Tp __d = __w.imag();
  _Tp __ac = __a * __c;
  _Tp __bd = __b * __d;
  _Tp __ad = __a * __d;
  _Tp __bc = __b * __c;
  _Tp __x = __ac - __bd;
  _Tp __y = __ad + __bc;
  if (isnan(__x) && isnan(__y)) {
    bool __recalc = false;
    if (isinf(__a) || isinf(__b)) {
      __a = copysign(isinf(__a) ? _Tp(1) : _Tp(0), __a);
      __b = copysign(isinf(__b) ? _Tp(1) : _Tp(0), __b);
      if (isnan(__c))
        __c = copysign(_Tp(0), __c);
      if (isnan(__d))
        __d = copysign(_Tp(0), __d);
      __recalc = true;
    }
    if (isinf(__c) || isinf(__d)) {
      __c = copysign(isinf(__c) ? _Tp(1) : _Tp(0), __c);
      __d = copysign(isinf(__d) ? _Tp(1) : _Tp(0), __d);
      if (isnan(__a))
        __a = copysign(_Tp(0), __a);
      if (isnan(__b))
        __b = copysign(_Tp(0), __b);
      __recalc = true;
    }
    if (!__recalc &&
        (isinf(__ac) || isinf(__bd) || isinf(__ad) || isinf(__bc))) {
      if (isnan(__a))
        __a = copysign(_Tp(0), __a);
      if (isnan(__b))
        __b = copysign(_Tp(0), __b);
      if (isnan(__c))
        __c = copysign(_Tp(0), __c);
      if (isnan(__d))
        __d = copysign(_Tp(0), __d);
      __recalc = true;
    }
    if (__recalc) {
      __x = _Tp(INFINITY) * (__a * __c - __b * __d);
      __y = _Tp(INFINITY) * (__a * __d + __b * __c);
    }
  }
  return complex<_Tp>(__x, __y);
}

template <class _Tp>
inline complex<_Tp> operator*(const complex<_Tp>& __x, const _Tp& __y) {
  complex<_Tp> __t(__x);
  __t *= __y;
  return __t;
}

template <class _Tp>
inline complex<_Tp> operator*(const _Tp& __x, const complex<_Tp>& __y) {
  complex<_Tp> __t(__y);
  __t *= __x;
  return __t;
}

template <class _Tp>
complex<_Tp> operator/(const complex<_Tp>& __z, const complex<_Tp>& __w) {
  int __ilogbw = 0;
  _Tp __a = __z.real();
  _Tp __b = __z.imag();
  _Tp __c = __w.real();
  _Tp __d = __w.imag();
  _Tp __logbw = logb(fmax(fabs(__c), fabs(__d)));
  if (isfinite(__logbw)) {
    __ilogbw = static_cast<int>(__logbw);
    __c = scalbn(__c, -__ilogbw);
    __d = scalbn(__d, -__ilogbw);
  }
  _Tp __denom = __c * __c + __d * __d;
  _Tp __x = scalbn((__a * __c + __b * __d) / __denom, -__ilogbw);
  _Tp __y = scalbn((__b * __c - __a * __d) / __denom, -__ilogbw);
  if (isnan(__x) && isnan(__y)) {
    if ((__denom == _Tp(0)) && (!isnan(__a) || !isnan(__b))) {
      __x = copysign(_Tp(INFINITY), __c) * __a;
      __y = copysign(_Tp(INFINITY), __c) * __b;
    } else if ((isinf(__a) || isinf(__b)) && isfinite(__c) && isfinite(__d)) {
      __a = copysign(isinf(__a) ? _Tp(1) : _Tp(0), __a);
      __b = copysign(isinf(__b) ? _Tp(1) : _Tp(0), __b);
      __x = _Tp(INFINITY) * (__a * __c + __b * __d);
      __y = _Tp(INFINITY) * (__b * __c - __a * __d);
    } else if (
        isinf(__logbw) && __logbw > _Tp(0) && isfinite(__a) && isfinite(__b)) {
      __c = copysign(isinf(__c) ? _Tp(1) : _Tp(0), __c);
      __d = copysign(isinf(__d) ? _Tp(1) : _Tp(0), __d);
      __x = _Tp(0) * (__a * __c + __b * __d);
      __y = _Tp(0) * (__b * __c - __a * __d);
    }
  }
  return complex<_Tp>(__x, __y);
}

template <class _Tp>
inline complex<_Tp> operator/(const complex<_Tp>& __x, const _Tp& __y) {
  return complex<_Tp>(__x.real() / __y, __x.imag() / __y);
}

template <class _Tp>
inline complex<_Tp> operator/(const _Tp& __x, const complex<_Tp>& __y) {
  complex<_Tp> __t(__x);
  __t /= __y;
  return __t;
}

template <class _Tp>
inline complex<_Tp> operator+(const complex<_Tp>& __x) {
  return __x;
}

template <class _Tp>
inline complex<_Tp> operator-(const complex<_Tp>& __x) {
  return complex<_Tp>(-__x.real(), -__x.imag());
}

template <class _Tp>
inline constexpr bool operator==(
    const complex<_Tp>& __x,
    const complex<_Tp>& __y) {
  return __x.real() == __y.real() && __x.imag() == __y.imag();
}

template <class _Tp>
inline constexpr bool operator==(const complex<_Tp>& __x, const _Tp& __y) {
  return __x.real() == __y && __x.imag() == 0;
}

template <class _Tp>
inline constexpr bool operator==(const _Tp& __x, const complex<_Tp>& __y) {
  return __x == __y.real() && 0 == __y.imag();
}

template <class _Tp>
inline constexpr bool operator!=(
    const complex<_Tp>& __x,
    const complex<_Tp>& __y) {
  return !(__x == __y);
}

template <class _Tp>
inline constexpr bool operator!=(const complex<_Tp>& __x, const _Tp& __y) {
  return !(__x == __y);
}

template <class _Tp>
inline constexpr bool operator!=(const _Tp& __x, const complex<_Tp>& __y) {
  return !(__x == __y);
}

template <class _Tp>
inline constexpr bool operator&&(
    const complex<_Tp>& __x,
    const complex<_Tp>& __y) {
  return bool(__x) && bool(__y);
}

template <class _Tp>
inline constexpr bool isnan(const complex<_Tp>& __x) {
  return isnan(__x.real()) || isnan(__x.imag());
}

template <class _Tp>
inline constexpr bool operator||(
    const complex<_Tp>& __x,
    const complex<_Tp>& __y) {
  return bool(__x) || bool(__y);
}

// 26.3.7 values:

template <
    class _Tp,
    bool = is_integral<_Tp>::value,
    bool = is_floating_point<_Tp>::value>
struct __libcpp_complex_overload_traits {};

// Integral Types
template <class _Tp>
struct __libcpp_complex_overload_traits<_Tp, true, false> {
  typedef double _ValueType;
  typedef complex<double> _ComplexType;
};

// Floating point types
template <class _Tp>
struct __libcpp_complex_overload_traits<_Tp, false, true> {
  typedef _Tp _ValueType;
  typedef complex<_Tp> _ComplexType;
};

// real

template <class _Tp>
inline constexpr _Tp real(const complex<_Tp>& __c) {
  return __c.real();
}

template <class _Tp>
inline constexpr typename __libcpp_complex_overload_traits<_Tp>::_ValueType real(
    _Tp __re) {
  return __re;
}

// imag

template <class _Tp>
inline constexpr _Tp imag(const complex<_Tp>& __c) {
  return __c.imag();
}

template <class _Tp>
inline constexpr typename __libcpp_complex_overload_traits<_Tp>::_ValueType imag(
    _Tp) {
  return 0;
}

// abs

template <class _Tp>
inline _Tp abs(const complex<_Tp>& __c) {
  return hypot(__c.real(), __c.imag());
}

// arg

template <class _Tp>
inline _Tp arg(const complex<_Tp>& __c) {
  return atan2(__c.imag(), __c.real());
}

template <class _Tp>
inline typename enable_if<
    is_integral<_Tp>::value || is_same<_Tp, double>::value,
    double>::type
arg(_Tp __re) {
  return atan2(0., __re);
}

template <class _Tp>
inline typename enable_if<is_same<_Tp, float>::value, float>::type arg(
    _Tp __re) {
  return atan2f(0.F, __re);
}

} // namespace std

namespace std {

using ::isfinite;
using ::isinf;
using ::isnan;
using ::signbit;

using ::abs;

using ::acos;
using ::acosf;
using ::asin;
using ::asinf;
using ::atan;
using ::atan2;
using ::atan2f;
using ::atanf;
using ::ceil;
using ::ceilf;
using ::cos;
using ::cosf;
using ::cosh;
using ::coshf;

using ::exp;
using ::expf;

using ::fabs;
using ::fabsf;
using ::floor;
using ::floorf;

using ::fmod;
using ::fmodf;

using ::frexp;
using ::frexpf;
using ::ldexp;
using ::ldexpf;

using ::log;
using ::logf;

using ::log10;
using ::log10f;
using ::modf;
using ::modff;

using ::pow;
using ::powf;

using ::sin;
using ::sinf;
using ::sinh;
using ::sinhf;

using ::sqrt;
using ::sqrtf;
using ::tan;
using ::tanf;

using ::tanh;
using ::tanhf;

using ::acosh;
using ::acoshf;
using ::asinh;
using ::asinhf;
using ::atanh;
using ::atanhf;
using ::cbrt;
using ::cbrtf;

using ::copysign;
using ::copysignf;

using ::erf;
using ::erfc;
using ::erfcf;
using ::erff;
using ::exp2;
using ::exp2f;
using ::expm1;
using ::expm1f;
using ::fdim;
using ::fdimf;
using ::fma;
using ::fmaf;
using ::fmax;
using ::fmaxf;
using ::fmin;
using ::fminf;
using ::hypot;
using ::hypotf;
using ::ilogb;
using ::ilogbf;
using ::lgamma;
using ::lgammaf;
using ::llrint;
using ::llrintf;
using ::llround;
using ::llroundf;
using ::log1p;
using ::log1pf;
using ::log2;
using ::log2f;
using ::logb;
using ::logbf;
using ::lrint;
using ::lrintf;
using ::lround;
using ::lroundf;

using ::nan;
using ::nanf;

using ::nearbyint;
using ::nearbyintf;
using ::nextafter;
using ::nextafterf;
using ::remainder;
using ::remainderf;
using ::remquo;
using ::remquof;
using ::rint;
using ::rintf;
using ::round;
using ::roundf;
using ::scalbln;
using ::scalblnf;
using ::scalbn;
using ::scalbnf;
using ::tgamma;
using ::tgammaf;
using ::trunc;
using ::truncf;

} // namespace std

namespace std {

// norm

template <class _Tp>
inline _Tp norm(const complex<_Tp>& __c) {
  if (isinf(__c.real()))
    return abs(__c.real());
  if (isinf(__c.imag()))
    return abs(__c.imag());
  return __c.real() * __c.real() + __c.imag() * __c.imag();
}

template <class _Tp>
inline typename __libcpp_complex_overload_traits<_Tp>::_ValueType norm(
    _Tp __re) {
  typedef typename __libcpp_complex_overload_traits<_Tp>::_ValueType _ValueType;
  return static_cast<_ValueType>(__re) * __re;
}

// conj

template <class _Tp>
inline complex<_Tp> conj(const complex<_Tp>& __c) {
  return complex<_Tp>(__c.real(), -__c.imag());
}

template <class _Tp>
inline typename __libcpp_complex_overload_traits<_Tp>::_ComplexType conj(
    _Tp __re) {
  typedef
      typename __libcpp_complex_overload_traits<_Tp>::_ComplexType _ComplexType;
  return _ComplexType(__re);
}

// proj

template <class _Tp>
inline complex<_Tp> proj(const complex<_Tp>& __c) {
  complex<_Tp> __r = __c;
  if (isinf(__c.real()) || isinf(__c.imag()))
    __r = complex<_Tp>(INFINITY, copysign(_Tp(0), __c.imag()));
  return __r;
}

template <class _Tp>
inline typename enable_if<
    is_floating_point<_Tp>::value,
    typename __libcpp_complex_overload_traits<_Tp>::_ComplexType>::type
proj(_Tp __re) {
  if (isinf(__re))
    __re = abs(__re);
  return complex<_Tp>(__re);
}

template <class _Tp>
inline typename enable_if<
    is_integral<_Tp>::value,
    typename __libcpp_complex_overload_traits<_Tp>::_ComplexType>::type
proj(_Tp __re) {
  typedef
      typename __libcpp_complex_overload_traits<_Tp>::_ComplexType _ComplexType;
  return _ComplexType(__re);
}

// polar

template <class _Tp>
complex<_Tp> polar(const _Tp& __rho, const _Tp& __theta = _Tp()) {
  if (isnan(__rho) || signbit(__rho))
    return complex<_Tp>(_Tp(NAN), _Tp(NAN));
  if (isnan(__theta)) {
    if (isinf(__rho))
      return complex<_Tp>(__rho, __theta);
    return complex<_Tp>(__theta, __theta);
  }
  if (isinf(__theta)) {
    if (isinf(__rho))
      return complex<_Tp>(__rho, _Tp(NAN));
    return complex<_Tp>(_Tp(NAN), _Tp(NAN));
  }
  _Tp __x = __rho * cos(__theta);
  if (isnan(__x))
    __x = 0;
  _Tp __y = __rho * sin(__theta);
  if (isnan(__y))
    __y = 0;
  return complex<_Tp>(__x, __y);
}

// log

template <class _Tp>
inline complex<_Tp> log(const complex<_Tp>& __x) {
  return complex<_Tp>(log(abs(__x)), arg(__x));
}

// log10

template <class _Tp>
inline complex<_Tp> log10(const complex<_Tp>& __x) {
  return log(__x) / log(_Tp(10));
}

// log2

template <class _Tp>
inline complex<_Tp> log2(const complex<_Tp>& __x) {
  return log(__x) / log(_Tp(2));
}

// sqrt

template <class _Tp>
complex<_Tp> sqrt(const complex<_Tp>& __x) {
  if (isinf(__x.imag()))
    return complex<_Tp>(_Tp(INFINITY), __x.imag());
  if (isinf(__x.real())) {
    if (__x.real() > _Tp(0))
      return complex<_Tp>(
          __x.real(),
          isnan(__x.imag()) ? __x.imag() : copysign(_Tp(0), __x.imag()));
    return complex<_Tp>(
        isnan(__x.imag()) ? __x.imag() : _Tp(0),
        copysign(__x.real(), __x.imag()));
  }
  return polar(sqrt(abs(__x)), arg(__x) / _Tp(2));
}

// exp

template <class _Tp>
complex<_Tp> exp(const complex<_Tp>& __x) {
  _Tp __i = __x.imag();
  if (__i == 0) {
    return complex<_Tp>(exp(__x.real()), copysign(_Tp(0), __x.imag()));
  }
  if (isinf(__x.real())) {
    if (__x.real() < _Tp(0)) {
      if (!isfinite(__i))
        __i = _Tp(1);
    } else if (__i == 0 || !isfinite(__i)) {
      if (isinf(__i))
        __i = _Tp(NAN);
      return complex<_Tp>(__x.real(), __i);
    }
  }
  _Tp __e = exp(__x.real());
  return complex<_Tp>(__e * cos(__i), __e * sin(__i));
}

// pow

template <class _Tp>
inline complex<_Tp> pow(const complex<_Tp>& __x, const complex<_Tp>& __y) {
  return exp(__y * log(__x));
}

template <class _Tp, class _Up>
inline complex<typename __promote<_Tp, _Up>::type> pow(
    const complex<_Tp>& __x,
    const complex<_Up>& __y) {
  typedef complex<typename __promote<_Tp, _Up>::type> result_type;
  return std::pow(result_type(__x), result_type(__y));
}

template <class _Tp, class _Up>
inline typename enable_if<
    is_arithmetic<_Up>::value,
    complex<typename __promote<_Tp, _Up>::type>>::type
pow(const complex<_Tp>& __x, const _Up& __y) {
  typedef complex<typename __promote<_Tp, _Up>::type> result_type;
  return std::pow(result_type(__x), result_type(__y));
}

template <class _Tp, class _Up>
inline typename enable_if<
    is_arithmetic<_Tp>::value,
    complex<typename __promote<_Tp, _Up>::type>>::type
pow(const _Tp& __x, const complex<_Up>& __y) {
  typedef complex<typename __promote<_Tp, _Up>::type> result_type;
  return std::pow(result_type(__x), result_type(__y));
}

// __sqr, computes pow(x, 2)

template <class _Tp>
inline complex<_Tp> __sqr(const complex<_Tp>& __x) {
  return complex<_Tp>(
      (__x.real() - __x.imag()) * (__x.real() + __x.imag()),
      _Tp(2) * __x.real() * __x.imag());
}

// asinh

template <class _Tp>
complex<_Tp> asinh(const complex<_Tp>& __x) {
  const _Tp __pi(atan2(+0., -0.));
  if (isinf(__x.real())) {
    if (isnan(__x.imag()))
      return __x;
    if (isinf(__x.imag()))
      return complex<_Tp>(__x.real(), copysign(__pi * _Tp(0.25), __x.imag()));
    return complex<_Tp>(__x.real(), copysign(_Tp(0), __x.imag()));
  }
  if (isnan(__x.real())) {
    if (isinf(__x.imag()))
      return complex<_Tp>(__x.imag(), __x.real());
    if (__x.imag() == 0)
      return __x;
    return complex<_Tp>(__x.real(), __x.real());
  }
  if (isinf(__x.imag()))
    return complex<_Tp>(
        copysign(__x.imag(), __x.real()), copysign(__pi / _Tp(2), __x.imag()));
  complex<_Tp> __z = log(__x + sqrt(__sqr(__x) + _Tp(1)));
  return complex<_Tp>(
      copysign(__z.real(), __x.real()), copysign(__z.imag(), __x.imag()));
}

// acosh

template <class _Tp>
complex<_Tp> acosh(const complex<_Tp>& __x) {
  const _Tp __pi(atan2(+0., -0.));
  if (isinf(__x.real())) {
    if (isnan(__x.imag()))
      return complex<_Tp>(abs(__x.real()), __x.imag());
    if (isinf(__x.imag())) {
      if (__x.real() > 0)
        return complex<_Tp>(__x.real(), copysign(__pi * _Tp(0.25), __x.imag()));
      else
        return complex<_Tp>(
            -__x.real(), copysign(__pi * _Tp(0.75), __x.imag()));
    }
    if (__x.real() < 0)
      return complex<_Tp>(-__x.real(), copysign(__pi, __x.imag()));
    return complex<_Tp>(__x.real(), copysign(_Tp(0), __x.imag()));
  }
  if (isnan(__x.real())) {
    if (isinf(__x.imag()))
      return complex<_Tp>(abs(__x.imag()), __x.real());
    return complex<_Tp>(__x.real(), __x.real());
  }
  if (isinf(__x.imag()))
    return complex<_Tp>(abs(__x.imag()), copysign(__pi / _Tp(2), __x.imag()));
  complex<_Tp> __z = log(__x + sqrt(__sqr(__x) - _Tp(1)));
  return complex<_Tp>(
      copysign(__z.real(), _Tp(0)), copysign(__z.imag(), __x.imag()));
}

// atanh

template <class _Tp>
complex<_Tp> atanh(const complex<_Tp>& __x) {
  const _Tp __pi(atan2(+0., -0.));
  if (isinf(__x.imag())) {
    return complex<_Tp>(
        copysign(_Tp(0), __x.real()), copysign(__pi / _Tp(2), __x.imag()));
  }
  if (isnan(__x.imag())) {
    if (isinf(__x.real()) || __x.real() == 0)
      return complex<_Tp>(copysign(_Tp(0), __x.real()), __x.imag());
    return complex<_Tp>(__x.imag(), __x.imag());
  }
  if (isnan(__x.real())) {
    return complex<_Tp>(__x.real(), __x.real());
  }
  if (isinf(__x.real())) {
    return complex<_Tp>(
        copysign(_Tp(0), __x.real()), copysign(__pi / _Tp(2), __x.imag()));
  }
  if (abs(__x.real()) == _Tp(1) && __x.imag() == _Tp(0)) {
    return complex<_Tp>(
        copysign(_Tp(INFINITY), __x.real()), copysign(_Tp(0), __x.imag()));
  }
  complex<_Tp> __z = log((_Tp(1) + __x) / (_Tp(1) - __x)) / _Tp(2);
  return complex<_Tp>(
      copysign(__z.real(), __x.real()), copysign(__z.imag(), __x.imag()));
}

// sinh

template <class _Tp>
complex<_Tp> sinh(const complex<_Tp>& __x) {
  if (isinf(__x.real()) && !isfinite(__x.imag()))
    return complex<_Tp>(__x.real(), _Tp(NAN));
  if (__x.real() == 0 && !isfinite(__x.imag()))
    return complex<_Tp>(__x.real(), _Tp(NAN));
  if (__x.imag() == 0 && !isfinite(__x.real()))
    return __x;
  return complex<_Tp>(
      sinh(__x.real()) * cos(__x.imag()), cosh(__x.real()) * sin(__x.imag()));
}

// cosh

template <class _Tp>
complex<_Tp> cosh(const complex<_Tp>& __x) {
  if (isinf(__x.real()) && !isfinite(__x.imag()))
    return complex<_Tp>(abs(__x.real()), _Tp(NAN));
  if (__x.real() == 0 && !isfinite(__x.imag()))
    return complex<_Tp>(_Tp(NAN), __x.real());
  if (__x.real() == 0 && __x.imag() == 0)
    return complex<_Tp>(_Tp(1), __x.imag());
  if (__x.imag() == 0 && !isfinite(__x.real()))
    return complex<_Tp>(abs(__x.real()), __x.imag());
  return complex<_Tp>(
      cosh(__x.real()) * cos(__x.imag()), sinh(__x.real()) * sin(__x.imag()));
}

// tanh

template <class _Tp>
complex<_Tp> tanh(const complex<_Tp>& __x) {
  if (isinf(__x.real())) {
    if (!isfinite(__x.imag()))
      return complex<_Tp>(copysign(_Tp(1), __x.real()), _Tp(0));
    return complex<_Tp>(
        copysign(_Tp(1), __x.real()),
        copysign(_Tp(0), sin(_Tp(2) * __x.imag())));
  }
  if (isnan(__x.real()) && __x.imag() == 0)
    return __x;
  _Tp __2r(_Tp(2) * __x.real());
  _Tp __2i(_Tp(2) * __x.imag());
  _Tp __d(cosh(__2r) + cos(__2i));
  _Tp __2rsh(sinh(__2r));
  if (isinf(__2rsh) && isinf(__d))
    return complex<_Tp>(
        __2rsh > _Tp(0) ? _Tp(1) : _Tp(-1), __2i > _Tp(0) ? _Tp(0) : _Tp(-0.));
  return complex<_Tp>(__2rsh / __d, sin(__2i) / __d);
}

// asin

template <class _Tp>
complex<_Tp> asin(const complex<_Tp>& __x) {
  complex<_Tp> __z = asinh(complex<_Tp>(-__x.imag(), __x.real()));
  return complex<_Tp>(__z.imag(), -__z.real());
}

// acos

template <class _Tp>
complex<_Tp> acos(const complex<_Tp>& __x) {
  const _Tp __pi(atan2(+0., -0.));
  if (isinf(__x.real())) {
    if (isnan(__x.imag()))
      return complex<_Tp>(__x.imag(), __x.real());
    if (isinf(__x.imag())) {
      if (__x.real() < _Tp(0))
        return complex<_Tp>(_Tp(0.75) * __pi, -__x.imag());
      return complex<_Tp>(_Tp(0.25) * __pi, -__x.imag());
    }
    if (__x.real() < _Tp(0))
      return complex<_Tp>(__pi, signbit(__x.imag()) ? -__x.real() : __x.real());
    return complex<_Tp>(_Tp(0), signbit(__x.imag()) ? __x.real() : -__x.real());
  }
  if (isnan(__x.real())) {
    if (isinf(__x.imag()))
      return complex<_Tp>(__x.real(), -__x.imag());
    return complex<_Tp>(__x.real(), __x.real());
  }
  if (isinf(__x.imag()))
    return complex<_Tp>(__pi / _Tp(2), -__x.imag());
  if (__x.real() == 0 && (__x.imag() == 0 || isnan(__x.imag())))
    return complex<_Tp>(__pi / _Tp(2), -__x.imag());
  complex<_Tp> __z = log(__x + sqrt(__sqr(__x) - _Tp(1)));
  if (signbit(__x.imag()))
    return complex<_Tp>(abs(__z.imag()), abs(__z.real()));
  return complex<_Tp>(abs(__z.imag()), -abs(__z.real()));
}

// atan

template <class _Tp>
complex<_Tp> atan(const complex<_Tp>& __x) {
  complex<_Tp> __z = atanh(complex<_Tp>(-__x.imag(), __x.real()));
  return complex<_Tp>(__z.imag(), -__z.real());
}

// sin

template <class _Tp>
complex<_Tp> sin(const complex<_Tp>& __x) {
  complex<_Tp> __z = sinh(complex<_Tp>(-__x.imag(), __x.real()));
  return complex<_Tp>(__z.imag(), -__z.real());
}

// cos

template <class _Tp>
inline complex<_Tp> cos(const complex<_Tp>& __x) {
  return cosh(complex<_Tp>(-__x.imag(), __x.real()));
}

// tan

template <class _Tp>
complex<_Tp> tan(const complex<_Tp>& __x) {
  complex<_Tp> __z = tanh(complex<_Tp>(-__x.imag(), __x.real()));
  return complex<_Tp>(__z.imag(), -__z.real());
}

// Literal suffix for complex number literals [complex.literals]
inline namespace literals {
inline namespace complex_literals {
constexpr complex<double> operator""i(long double __im) {
  return {0.0, static_cast<double>(__im)};
}

constexpr complex<double> operator""i(unsigned long long __im) {
  return {0.0, static_cast<double>(__im)};
}

constexpr complex<float> operator""if(long double __im) {
  return {0.0f, static_cast<float>(__im)};
}

constexpr complex<float> operator""if(unsigned long long __im) {
  return {0.0f, static_cast<float>(__im)};
}
} // namespace complex_literals
} // namespace literals

} // namespace std

__device__ std::complex<double> lerp(
    std::complex<double> start,
    std::complex<double> end,
    std::complex<double> weight) {
  if (abs(weight) < 0.5) {
    return start + weight * (end - start);
  } else {
    return end - (end - start) * (1.0 - weight);
  }
}

__device__ std::complex<float> lerp(
    std::complex<float> start,
    std::complex<float> end,
    std::complex<float> weight) {
  if (abs(weight) < 0.5f) {
    return start + weight * (end - start);
  } else {
    return end - (end - start) * (1.0f - weight);
  }
}

__device__ std::complex<double> reciprocal(std::complex<double> x) {
  return 1.0 / x;
}

__device__ std::complex<float> reciprocal(std::complex<float> x) {
  return 1.0f / x;
}

__device__ std::complex<double> sigmoid(std::complex<double> x) {
  return 1.0 / (1.0 + exp(-x));
}

__device__ std::complex<float> sigmoid(std::complex<float> x) {
  return 1.0f / (1.0f + exp(-x));
}

// The reciprocal of a complex number z is
//    1/z = conj(z)/|z|^2.
// The principal square root of a complex number z can be obtained by [1]
//    sqrt(z) = sqrt(|z|) (z + |z|) / |z + |z||.
// Combining these formulas we have
//    1/sqrt(z) = (conj(z) + |z|) / (sqrt(|z|) |z + |z||).
// [1] https://math.stackexchange.com/a/44500
__device__ std::complex<float> rsqrt(std::complex<float> z) {
  auto a = std::real(z);
  auto b = std::imag(z);
  auto absa = ::fabsf(a);
  auto absb = ::fabsf(b);
  // scale to avoid precision loss due to underflow/overflow
  auto scale = fmax(absa, absb);
  a /= scale;
  b /= scale;
  auto a_sq = a * a;
  auto b_sq = b * b;
  auto modz_sq = a_sq + b_sq;
  auto modz = ::sqrtf(modz_sq);
  auto a_plus_modz = a + modz;
  auto mod_zplusmodz_sq = a_plus_modz * a_plus_modz + b_sq;
  auto fac = ::rsqrtf(scale * modz * mod_zplusmodz_sq);
  return std::complex<float>(a_plus_modz * fac, -b * fac);
}

__device__ std::complex<double> rsqrt(std::complex<double> z) {
  auto a = std::real(z);
  auto b = std::imag(z);
  auto absa = ::abs(a);
  auto absb = ::abs(b);
  // scale to avoid precision loss due to underflow/overflow
  auto scale = fmax(absa, absb);
  a /= scale;
  b /= scale;
  auto a_sq = a * a;
  auto b_sq = b * b;
  auto modz_sq = a_sq + b_sq;
  auto modz = ::sqrt(modz_sq);
  auto a_plus_modz = a + modz;
  auto mod_zplusmodz_sq = a_plus_modz * a_plus_modz + b_sq;
  auto fac = ::rsqrt(scale * modz * mod_zplusmodz_sq);
  return std::complex<double>(a_plus_modz * fac, -b * fac);
}

template <typename T>
bool isfinite(std::complex<T> x) {
  return ::isfinite(std::real(x)) && ::isfinite(std::imag(x));
}

template <typename T>
bool isinf(std::complex<T> x) {
  return ::isinf(std::real(x)) || ::isinf(std::imag(x));
}

template <typename T>
bool isreal(std::complex<T> x) {
  return std::imag(x) == 0;
}
#endif // __NVCC__
