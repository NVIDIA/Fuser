
#include <hip/hip_runtime.h>
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
__device__ unsigned int mulhilo32(
    unsigned int a,
    unsigned int b,
    unsigned int* result_high) {
  *result_high = __umulhi(a, b);
  return a * b;
}

__device__ uint4 single_round(uint4 ctr, uint2 key) {
  constexpr unsigned long kPhiloxSA = 0xD2511F53;
  constexpr unsigned long kPhiloxSB = 0xCD9E8D57;
  unsigned int hi0;
  unsigned int hi1;
  unsigned int lo0 = mulhilo32(kPhiloxSA, ctr.x, &hi0);
  unsigned int lo1 = mulhilo32(kPhiloxSB, ctr.z, &hi1);
  uint4 ret = {hi1 ^ ctr.y ^ key.x, lo1, hi0 ^ ctr.w ^ key.y, lo0};
  return ret;
}

__device__ uint4 philox(
    unsigned long long seed,
    unsigned long long subsequence,
    unsigned long long offset) {
  constexpr unsigned long kPhilox10A = 0x9E3779B9;
  constexpr unsigned long kPhilox10B = 0xBB67AE85;
  uint2 key = {};
  key.x = (unsigned int)seed;
  key.y = (unsigned int)(seed >> 32);
  uint4 counter = make_uint4(0, 0, 0, 0);
  counter.x = (unsigned int)(offset);
  counter.y = (unsigned int)(offset >> 32);
  counter.z = (unsigned int)(subsequence);
  counter.w = (unsigned int)(subsequence >> 32);

  uint4 output = {};
  uint2 key_ = key;
  uint4 counter_ = counter;
  for (int i = 0; i < 9; i++) {
    counter_ = single_round(counter_, key_);
    key_.x += (kPhilox10A);
    key_.y += (kPhilox10B);
  }
  output = single_round(counter_, key_);
  return output;
}

__device__ float uniformf(unsigned int x) {
  constexpr float kRanInvM32 = 2.3283064e-10f; // Inverse of 2^32.
  float result = x * kRanInvM32 + kRanInvM32 / 2.0f;
  return result == 1 ? 0.0f : result;
}

__device__ double uniform(unsigned int x, unsigned int y) {
  constexpr double kRan2Pow53Inv = 1.1102230246251565e-16;
  const unsigned long long z =
      (unsigned long long)x ^ ((unsigned long long)y << (53 - 32));
  double result = z * kRan2Pow53Inv + (kRan2Pow53Inv / 2.0);
  return result == 1 ? 0.0 : result;
}

__device__ double rng_uniform(const uint4& rng_result, int rng_component) {
  return uniform(
      (&rng_result.x)[rng_component * 2],
      (&rng_result.x)[rng_component * 2 + 1]);
}

__device__ float rng_uniformf(const uint4& rng_result, int rng_component) {
  return uniformf((&rng_result.x)[rng_component]);
}

__device__ double rng_uniform_range(
    const uint4& rng_result,
    int rng_component,
    double from,
    double to) {
  auto range = to - from;
  auto uniform01 = rng_uniform(rng_result, rng_component);
  return from + range * uniform01;
}

__device__ float rng_uniform_rangef(
    const uint4& rng_result,
    int rng_component,
    float from,
    float to) {
  auto range = to - from;
  auto uniform01 = rng_uniformf(rng_result, rng_component);
  return from + range * uniform01;
}

__device__ float normalf(unsigned int x, unsigned int y, int rng_component) {
  float u = uniformf(x);
  float v = uniformf(y) * 6.2831855f;

  if (rng_component % 2 == 0) {
    return sqrtf(-2.0f * logf(u)) * sinf(v);
  } else {
    return sqrtf(-2.0f * logf(u)) * cosf(v);
  }
}

__device__ double normal(
    unsigned int x0,
    unsigned int x1,
    unsigned int y0,
    unsigned int y1,
    int rng_component) {
  double u = uniform(x0, x1);
  double v = uniform(y0, y1) * 6.2831853071795860;

  if (rng_component % 2 == 0) {
    return sqrt(-2.0 * log(u)) * sin(v);
  } else {
    return sqrt(-2.0 * log(u)) * cos(v);
  }
}

__device__ double rng_normal_standard(
    const uint4& rng_result,
    int rng_component) {
  return normal(
      rng_result.x, rng_result.y, rng_result.z, rng_result.w, rng_component);
}

__device__ float rng_normal_standardf(
    const uint4& rng_result,
    int rng_component) {
  return normalf(
      (&rng_result.x)[rng_component / 2 * 2],
      (&rng_result.y)[rng_component / 2 * 2],
      rng_component);
}

__device__ double rng_normal_general(
    const uint4& rng_result,
    int rng_component,
    double mean,
    double std) {
  auto normal01 = rng_normal_standard(rng_result, rng_component);
  return normal01 * std + mean;
}

__device__ float rng_normal_generalf(
    const uint4& rng_result,
    int rng_component,
    float mean,
    float std) {
  auto normal01 = rng_normal_standardf(rng_result, rng_component);
  return normal01 * std + mean;
}
