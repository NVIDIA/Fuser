// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
template <typename T, int Dims, int AllocDims = Dims>
struct Tensor {
  __device__ T& operator[](nvfuser_index_t ind) {
    return data[ind];
  };

  T* data;
  nvfuser_index_t size[Dims];
  nvfuser_index_t stride[AllocDims];
};

// Specialization for 0-dim case as it does not need size and stride arrays.
// They will be an error as well since zero-length arrays are not allowed.
template <typename T>
struct Tensor<T, 0> {
  __device__ T& operator[](nvfuser_index_t i) {
    return *data;
  };

  T* data;
};

// Specialization for 0-dim case that's easy to pass in a CPU based tensor.
template <typename T>
struct CpuScalarTensor {
  __device__ T& operator[](int i) {
    return data;
  };

  T data;
};
