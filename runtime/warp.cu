#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
namespace warp {

template <typename T>
__device__ __forceinline__ T shfl_xor(T var, int laneMask, int width = 32) {
  return __shfl_xor_sync(0xffffffff, var, laneMask, width);
}
template <typename T>
__device__ __forceinline__ std::complex<T> shfl_xor(
    std::complex<T> var,
    int laneMask,
    int width = 32) {
  T real = __shfl_xor_sync(0xffffffff, var.real(), laneMask, width);
  T imag = __shfl_xor_sync(0xffffffff, var.imag(), laneMask, width);
  return std::complex<T>(real, imag);
}

template <bool SINGLE_WARP, bool Aligned, typename T, typename Func>
__device__ void warpReduceTIDX(
    T& out,
    const T& inp_val,
    Func reduction_op,
    T* shared_mem,
    bool read_write_pred,
    T init_val) {
  constexpr int WARP_SIZE = 32;

  // Assume input padded to multiples of a warp
  T reduce_val = init_val;

  // Do warp reduction
  if (read_write_pred) {
    reduce_val = inp_val;
  }

  // Reduce within each warp
  for (int i = 16; i >= 1; i /= 2) {
    reduction_op(reduce_val, shfl_xor(reduce_val, i, WARP_SIZE));
  }

  // Reduce across warp if needed
  // Load value to shared mem
  if (!SINGLE_WARP) {
    unsigned int warp_idx = threadIdx.x / WARP_SIZE;
    unsigned int lane_idx = threadIdx.x % WARP_SIZE;
    unsigned int reduce_group_id = threadIdx.z * blockDim.y + threadIdx.y;
    bool is_warp_head = lane_idx == 0;
    unsigned int reduction_size = blockDim.x;
    unsigned int num_of_warps = reduction_size / WARP_SIZE;
    unsigned int smem_offset = reduce_group_id * num_of_warps;

    block_sync::sync<Aligned>();

    if (is_warp_head) {
      shared_mem[smem_offset + warp_idx] = reduce_val;
    }

    block_sync::sync<Aligned>();

    if (warp_idx == 0) {
      // This assumes num_of_warps will be < 32, meaning < 1024 threads.
      //  Should be true for long enough.
      assert(num_of_warps <= 32);

      reduce_val = lane_idx < num_of_warps ? shared_mem[smem_offset + lane_idx]
                                           : init_val;

      // Reduce within warp 0
      for (int i = 16; i >= 1; i /= 2) {
        reduction_op(reduce_val, shfl_xor(reduce_val, i, 32));
      }
    }

    if (is_warp_head) {
      reduction_op(out, reduce_val);
    }
  } else {
    reduction_op(out, reduce_val);
  }
}

} // namespace warp
