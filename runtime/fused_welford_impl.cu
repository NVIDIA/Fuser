#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
namespace fused_reduction {

namespace impl {

//! Implementation helper for welfordEach.
template <int ValIdx, typename Triplet0, typename Triplet1>
struct WelfordForEach {
  static __inline__ __device__ void call(
      Triplet0& triplet0,
      nvfuser_index_t offset0,
      const Triplet1& triplet1,
      nvfuser_index_t offset1) {
    static_assert(
        Triplet0::num_vals == Triplet1::num_vals, "Invalid Triplet types");
    static_assert(
        IsSameType<typename Triplet0::DataType, typename Triplet1::DataType>::
            value,
        "Invalid Triplet types");
    static_assert(
        IsSameType<typename Triplet0::IndexType, typename Triplet1::IndexType>::
            value,
        "Invalid Triplet types");

    using DataType = typename Triplet0::DataType;
    using IndexType = typename Triplet0::IndexType;

    WelfordForEach<ValIdx - 1, Triplet0, Triplet1>::call(
        triplet0, offset0, triplet1, offset1);
    welfordCombine<DataType, IndexType>(
        triplet0.avg.val<ValIdx>(offset0),
        triplet0.var.val<ValIdx>(offset0),
        triplet0.N.val<ValIdx>(offset0),
        triplet1.avg.val<ValIdx>(offset1),
        triplet1.var.val<ValIdx>(offset1),
        triplet1.N.val<ValIdx>(offset1));
  }
};

template <typename Triplet0, typename Triplet1>
struct WelfordForEach<-1, Triplet0, Triplet1> {
  __inline__ __device__ static void call(
      Triplet0& triplet0,
      nvfuser_index_t offset0,
      const Triplet1& triplet1,
      nvfuser_index_t offset1) {}
};

//! Call welfordCombine with each of the triplet tuples. This is a
//! welford version of reduceEach.
template <typename Triplet0, typename Triplet1>
__inline__ __device__ static void welfordEach(
    Triplet0& triplet0,
    nvfuser_index_t offset0,
    const Triplet1& triplet1,
    nvfuser_index_t offset1) {
  WelfordForEach<Triplet0::num_vals - 1, Triplet0, Triplet1>::call(
      triplet0, offset0, triplet1, offset1);
}

// Welford version of BlockReduceEach
template <
    int idx,
    bool BROADCAST,
    bool FORWARD_PROTECT_SMEM,
    int NumVals,
    typename DataType,
    typename IndexType>
struct BlockWelfordEach {
  __inline__ __device__ static void reduce(
      LocalWelfordTripletTuple<NumVals, DataType, IndexType>& block_result,
      const LocalWelfordTripletTuple<NumVals, DataType, IndexType>&
          partial_result,
      PtrTuple<DataType, DataType, IndexType> shared_buf,
      bool has_block_result,
      int tid_in_reduction,
      int num_threads_per_reduction,
      int num_elements_per_reduction,
      int reduction_idx) {
    // Finish the reduction of each tuple value with a smaller offset
    BlockWelfordEach<idx - 1, BROADCAST, true, NumVals, DataType, IndexType>::
        reduce(
            block_result,
            partial_result,
            shared_buf,
            has_block_result,
            tid_in_reduction,
            num_threads_per_reduction,
            num_elements_per_reduction,
            reduction_idx);

    if (num_elements_per_reduction == 1) {
      if (has_block_result) {
        copyWelfordTripletTuple(block_result, partial_result);
      }
      return;
    }

    LocalTuple<DataType, DataType, IndexType> block_result_i(
        partial_result.avg.val<idx>(0),
        partial_result.var.val<idx>(0),
        partial_result.N.val<idx>(0));

    const auto smem_offset =
        reduction_idx * num_threads_per_reduction + tid_in_reduction;

    const int np2 = 1 << (31 - __clz(num_elements_per_reduction));

    // Threads values are initialized, so all can participate here
    if (tid_in_reduction >= np2) {
      copyTuple(shared_buf, smem_offset, block_result_i);
    }

    block_sync::sync();
    if (tid_in_reduction < np2 &&
        tid_in_reduction + np2 < num_elements_per_reduction) {
      impl::reduceTuple(
          block_result_i,
          0,
          shared_buf,
          smem_offset + np2,
          welfordCombine<DataType, IndexType>);
    }

    if (tid_in_reduction < np2) {
      copyTuple(shared_buf, smem_offset, block_result_i);
    }

    // Always sync when communicating across smem
    block_sync::sync();

    // Reduce down to 2 values, last thread will do the final reduction and
    // can save a syncthreads this way
    for (int factor = np2 / 2; factor > 1; factor >>= 1) {
      if (tid_in_reduction < factor) {
        impl::reduceTuple(
            shared_buf,
            smem_offset,
            shared_buf,
            smem_offset + factor,
            welfordCombine<DataType, IndexType>);
      }
      block_sync::sync();
    }

    copyTuple(block_result_i, shared_buf, smem_offset);

    // Do the last reduction
    if (has_block_result) {
      impl::reduceTuple(
          block_result_i,
          0,
          shared_buf,
          smem_offset + 1,
          welfordCombine<DataType, IndexType>);
    }

    if (BROADCAST) {
      if (has_block_result) {
        // Put result back in shared memory, put in the first entry of the
        // reduction segment's buffer
        copyTuple(
            shared_buf,
            reduction_idx * num_threads_per_reduction,
            block_result_i);
      }

      // Sync threads to make sure result is in smem
      block_sync::sync();

      copyTuple(
          block_result_i,
          shared_buf,
          reduction_idx * num_threads_per_reduction);
    }

    block_result.avg.val<idx>(0) = block_result_i.val<0>(0);
    block_result.var.val<idx>(0) = block_result_i.val<1>(0);
    block_result.N.val<idx>(0) = block_result_i.val<2>(0);

    if (FORWARD_PROTECT_SMEM) {
      block_sync::sync();
    }
  }
};

// Specialization for idx == -1, i.e., no value to reduce.
template <
    bool BROADCAST,
    bool FORWARD_PROTECT_SMEM,
    int NumVals,
    typename DataType,
    typename IndexType>
struct BlockWelfordEach<
    -1,
    BROADCAST,
    FORWARD_PROTECT_SMEM,
    NumVals,
    DataType,
    IndexType> {
  __inline__ __device__ static void reduce(
      LocalWelfordTripletTuple<NumVals, DataType, IndexType>& block_result,
      const LocalWelfordTripletTuple<NumVals, DataType, IndexType>&
          partial_result,
      PtrTuple<DataType, DataType, IndexType> shared_buf,
      bool has_block_result,
      int tid_in_reduction,
      int num_threads_per_reduction,
      int num_elements_per_reduction,
      int reduction_idx) {}
};

//! Welford version of blockReduceEach. Perform block-parallel Welford
//! reduction of each Welford triplet.
template <
    bool BROADCAST,
    bool FORWARD_PROTECT_SMEM,
    int NumVals,
    typename DataType,
    typename IndexType>
__inline__ __device__ void blockWelfordEach(
    LocalWelfordTripletTuple<NumVals, DataType, IndexType>& block_result,
    const LocalWelfordTripletTuple<NumVals, DataType, IndexType>&
        partial_result,
    PtrTuple<DataType, DataType, IndexType> shared_buf,
    bool has_block_result,
    int tid_in_reduction,
    int num_threads_per_reduction,
    int num_elements_per_reduction,
    int reduction_idx) {
  BlockWelfordEach<
      NumVals - 1,
      BROADCAST,
      FORWARD_PROTECT_SMEM,
      NumVals,
      DataType,
      IndexType>::
      reduce(
          block_result,
          partial_result,
          shared_buf,
          has_block_result,
          tid_in_reduction,
          num_threads_per_reduction,
          num_elements_per_reduction,
          reduction_idx);
}

} // namespace impl

template <
    int X_BLOCK,
    int Y_BLOCK,
    int Z_BLOCK,
    int X_THREAD,
    int Y_THREAD,
    int Z_THREAD,
    bool PERSISTENT_REDUCTION,
    bool BROADCAST>
template <int NumArgs, typename DataType, typename IndexType>
__device__ __inline__ void ParallelReduce<
    X_BLOCK,
    Y_BLOCK,
    Z_BLOCK,
    X_THREAD,
    Y_THREAD,
    Z_THREAD,
    PERSISTENT_REDUCTION,
    BROADCAST>::
    welfordGroup(
        typename MakeRefTuple<NumArgs, DataType>::type out_avg,
        typename MakeRefTuple<NumArgs, DataType>::type out_var,
        typename MakeRefTuple<NumArgs, IndexType>::type out_N,
        const typename MakeConstRefTuple<NumArgs, DataType>::type& inp_avg,
        const typename MakeConstRefTuple<NumArgs, DataType>::type& inp_var,
        const typename MakeConstRefTuple<NumArgs, IndexType>::type& inp_N,
        const typename MakeLocalTuple<NumArgs, DataType>::type& init_avg,
        const typename MakeLocalTuple<NumArgs, DataType>::type& init_var,
        const typename MakeLocalTuple<NumArgs, IndexType>::type& init_N,
        typename MakeVolatilePtrTuple<NumArgs, DataType>::type
            global_work_buffer_avg,
        typename MakeVolatilePtrTuple<NumArgs, DataType>::type
            global_work_buffer_var,
        typename MakeVolatilePtrTuple<NumArgs, IndexType>::type
            global_work_buffer_N,
        int64_t* global_sync_buffer,
        PtrTuple<DataType, DataType, IndexType> shared_buf,
        const typename MakeLocalTuple<NumArgs, bool>::type& read_preds,
        const typename MakeLocalTuple<NumArgs, bool>::type& write_preds) {
  const ConstRefWelfordTripletTuple<NumArgs, DataType, IndexType> inp(
      inp_avg, inp_var, inp_N);
  RefWelfordTripletTuple<NumArgs, DataType, IndexType> out(
      out_avg, out_var, out_N);

  // If no reduction needed, just return input
  if (!BLOCK_REDUCE && !GRID_REDUCE) {
    copyWelfordTripletTupleIf(out, inp, read_preds && write_preds);
    return;
  }

  // Don't read/write in temporary buffers if in a predicated dimension
  const bool block_reduce_participate = index_utils::
      maskedIsZero<isPred(X_THREAD), isPred(Y_THREAD), isPred(Z_THREAD)>(
          threadIdx);

  // Only threads that with id == 0 in the dimensions being reduced will
  // have a valid result
  const bool has_block_result = index_utils::
      maskedIsZero<isReduce(X_THREAD), isReduce(Y_THREAD), isReduce(Z_THREAD)>(
          threadIdx);

  LocalWelfordTripletTuple<NumArgs, DataType, IndexType> block_result(
      init_avg, init_var, init_N);

  // Initial per-block reduction. Result is broadcast if specified
  // and this call is block reduction only.
  welfordGroupBlock<!GRID_REDUCE && BROADCAST, NumArgs, DataType, IndexType>(
      block_result, inp, shared_buf, read_preds, block_reduce_participate);

  // If block reduction only, save to out and exit
  if (!GRID_REDUCE) {
    copyWelfordTripletTupleIf(
        out,
        block_result,
        write_preds &&
            (block_reduce_participate && (BROADCAST || has_block_result)));

    // Need a block sync here as reduceGroupBlock does not
    // forward-protect the smem buffer. This block sync is not
    // necessary when a grid reduction follows since a block sync is
    // done just before the grid sync.
    block_sync::sync();
    return;
  }

  // -- START GRID REDUCTION -- //
  // Grid reductions are more challenging for two reasons, (1) the reduction
  // itself is 3D instead of 2D because we now have an iter domain space in
  // the grid dimension. (2) a tree reduction isn't performed, instead all
  // blocks will populate GMEM and one  block will finish the grid reduction.

  // What is the grid reduction size, block reduction already performed so
  // that doesn't have to be taken into consideration
  const auto grid_red_size = index_utils::
      maskedSize<isReduce(X_BLOCK), isReduce(Y_BLOCK), isReduce(Z_BLOCK)>(
          gridDim);

  // Which ID in the reduction is this block. Threads can participate in
  // multiple grid reductions, but the block will have the same relative index
  // in those reductions
  const auto idx_in_grid_red = index_utils::
      maskedOffset<isReduce(X_BLOCK), isReduce(Y_BLOCK), isReduce(Z_BLOCK)>(
          blockIdx, gridDim);

  // How many grid reductions have to be performed, in the grid dimension
  const auto num_block_iters = index_utils::
      maskedSize<isIter(X_BLOCK), isIter(Y_BLOCK), isIter(Z_BLOCK)>(gridDim);

  // Which grid reduction does this block participate in, in the grid
  // dimension
  const auto block_red_idx_offset = index_utils::
      maskedOffset<isIter(X_BLOCK), isIter(Y_BLOCK), isIter(Z_BLOCK)>(
          blockIdx, gridDim);

  // How many grid reductions have to be performed, in the block dimension
  const auto num_thread_iters = index_utils::
      maskedSize<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
          blockDim);

  // Which grid reduction does this thread participate in, in the block
  // dimension
  const auto thread_red_idx_offset = index_utils::
      maskedOffset<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
          threadIdx, blockDim);

  // 3D buffer of reductions:
  //    [reduction_offset(grid), iter_offset(grid), iter_offset(block)]
  // Offset into the work buffer
  auto work_buf_offset =
      (idx_in_grid_red * num_block_iters + block_red_idx_offset) *
          num_thread_iters +
      thread_red_idx_offset;

  // Don't read/write in temporary buffers if in a predicated dimension
  bool grid_reduce_participate = index_utils::
      maskedIsZero<isPred(X_BLOCK), isPred(Y_BLOCK), isPred(Z_BLOCK)>(blockIdx);

  VolatilePtrWelfordTripletTuple<NumArgs, DataType, IndexType>
      global_work_buffer(
          global_work_buffer_avg, global_work_buffer_var, global_work_buffer_N);

  if (PERSISTENT_REDUCTION && flip) {
    auto global_buffer_size =
        index_utils::
            maskedSize<isIter(X_BLOCK), isIter(Y_BLOCK), isIter(Z_BLOCK)>(
                gridDim) *
        index_utils::
            maskedSize<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
                blockDim) *
        grid_red_size;
    global_work_buffer += global_buffer_size;
  }
  flip = !flip;

  // Per-block partial reduction to global work buffer
  if (grid_reduce_participate && block_reduce_participate && has_block_result) {
    copyWelfordTripletTuple(global_work_buffer, work_buf_offset, block_result);
  }

  // -- GLOBAL BUFFER FILLED -- //

  bool last_block = index_utils::
      maskedIsLast<isReduce(X_BLOCK), isReduce(Y_BLOCK), isReduce(Z_BLOCK)>(
          blockIdx, gridDim);

  if (grid_reduce_participate) {
    // Don't need to sync up blocks that are not participating in this
    // reduction
    grid_sync::sync<
        isReduce(X_BLOCK),
        isReduce(Y_BLOCK),
        isReduce(Z_BLOCK),
        PERSISTENT_REDUCTION>(
        global_sync_buffer[block_red_idx_offset], grid_red_size, last_block);
  }

  // -- START BLOCK CLEANUP -- //
  welfordGroupLastBlock<NumArgs, DataType, IndexType>(
      out,
      global_work_buffer,
      LocalWelfordTripletTuple<NumArgs, DataType, IndexType>(
          init_avg, init_var, init_N),
      shared_buf,
      block_red_idx_offset,
      num_thread_iters,
      num_block_iters,
      thread_red_idx_offset,
      grid_red_size,
      write_preds,
      block_reduce_participate,
      grid_reduce_participate);

  // Forward protect the smem buffer
  block_sync::sync();
}

template <
    int X_BLOCK,
    int Y_BLOCK,
    int Z_BLOCK,
    int X_THREAD,
    int Y_THREAD,
    int Z_THREAD,
    bool PERSISTENT_REDUCTION,
    bool BROADCAST>
template <int NumArgs, typename DataType, typename IndexType>
__device__ __inline__ void ParallelReduce<
    X_BLOCK,
    Y_BLOCK,
    Z_BLOCK,
    X_THREAD,
    Y_THREAD,
    Z_THREAD,
    PERSISTENT_REDUCTION,
    BROADCAST>::
    welfordGroup(
        typename MakeRefTuple<NumArgs, DataType>::type out_avg,
        typename MakeRefTuple<NumArgs, DataType>::type out_var,
        typename MakeRefTuple<NumArgs, IndexType>::type out_N,
        const typename MakeConstRefTuple<NumArgs, DataType>::type& inp_avg,
        const typename MakeConstRefTuple<NumArgs, DataType>::type& inp_var,
        const typename MakeConstRefTuple<NumArgs, IndexType>::type& inp_N,
        const typename MakeLocalTuple<NumArgs, DataType>::type& init_avg,
        const typename MakeLocalTuple<NumArgs, DataType>::type& init_var,
        const typename MakeLocalTuple<NumArgs, IndexType>::type& init_N,
        typename MakeVolatilePtrTuple<NumArgs, DataType>::type
            global_work_buffer_avg,
        typename MakeVolatilePtrTuple<NumArgs, DataType>::type
            global_work_buffer_var,
        typename MakeVolatilePtrTuple<NumArgs, IndexType>::type
            global_work_buffer_N,
        int64_t* global_sync_buffer,
        PtrTuple<DataType, DataType, IndexType> shared_buf,
        const typename MakeLocalTuple<NumArgs, bool>::type& read_preds,
        const typename MakeLocalTuple<NumArgs, bool>::type& write_preds,
        int64_t& cycles,
        int64_t& count) {
  int64_t start_counter = 0;

  if (isLastBlockInGrid() &&
      index_utils::maskedIsZero<true, true, true>(threadIdx)) {
    start_counter = readCycleCounter();
  }

  welfordGroup<NumArgs, DataType, IndexType>(
      out_avg,
      out_var,
      out_N,
      inp_avg,
      inp_var,
      inp_N,
      init_avg,
      init_var,
      init_N,
      global_work_buffer_avg,
      global_work_buffer_var,
      global_work_buffer_N,
      global_sync_buffer,
      shared_buf,
      read_preds,
      write_preds);

  if (isLastBlockInGrid() &&
      index_utils::maskedIsZero<true, true, true>(threadIdx)) {
    cycles += readCycleCounter() - start_counter;
    ++count;
  }
}

template <
    int X_BLOCK,
    int Y_BLOCK,
    int Z_BLOCK,
    int X_THREAD,
    int Y_THREAD,
    int Z_THREAD,
    bool PERSISTENT_REDUCTION,
    bool BROADCAST>
template <
    bool BLOCK_BROADCAST,
    int NumVals,
    typename DataType,
    typename IndexType>
__device__ __inline__ void ParallelReduce<
    X_BLOCK,
    Y_BLOCK,
    Z_BLOCK,
    X_THREAD,
    Y_THREAD,
    Z_THREAD,
    PERSISTENT_REDUCTION,
    BROADCAST>::
    welfordGroupBlock(
        LocalWelfordTripletTuple<NumVals, DataType, IndexType>& block_result,
        const ConstRefWelfordTripletTuple<NumVals, DataType, IndexType>& inp,
        PtrTuple<DataType, DataType, IndexType> shared_buf,
        const typename MakeLocalTuple<NumVals, bool>::type& read_preds,
        bool block_reduce_participate) {
  const bool has_block_result = index_utils::
      maskedIsZero<isReduce(X_THREAD), isReduce(Y_THREAD), isReduce(Z_THREAD)>(
          threadIdx);

  copyWelfordTripletTupleIf(
      block_result, inp, block_reduce_participate && read_preds);

  // Size of the block reduction segment, can be an int since it's limited
  // to number of threads
  const int block_reduction_size = index_utils::
      maskedSize<isReduce(X_THREAD), isReduce(Y_THREAD), isReduce(Z_THREAD)>(
          blockDim);

  // Index in the reduction segment, can be an int since it's limited to
  // number of threads
  const int tid_in_block_reduction = index_utils::
      maskedOffset<isReduce(X_THREAD), isReduce(Y_THREAD), isReduce(Z_THREAD)>(
          threadIdx, blockDim);

  // ID of the block reduction this thread is participating in
  //
  // If any of the parallel dimensions are predicated out, that means
  // they've already been reduced, so we only care about the first thread in
  // that dimension. Therefore don't expand the reduction_idx by that
  // dimension
  const int block_reduction_idx = index_utils::
      maskedOffset<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
          threadIdx, blockDim);

  // Do not protect the smem buffer as it's not always necessary.
  impl::blockWelfordEach<BLOCK_BROADCAST, false, NumVals, DataType, IndexType>(
      block_result,
      block_result,
      shared_buf,
      has_block_result,
      tid_in_block_reduction,
      block_reduction_size,
      block_reduction_size,
      block_reduction_idx);
}

template <
    int X_BLOCK,
    int Y_BLOCK,
    int Z_BLOCK,
    int X_THREAD,
    int Y_THREAD,
    int Z_THREAD,
    bool PERSISTENT_REDUCTION,
    bool BROADCAST>
template <int NumVals, typename DataType, typename IndexType>
__device__ __inline__ void ParallelReduce<
    X_BLOCK,
    Y_BLOCK,
    Z_BLOCK,
    X_THREAD,
    Y_THREAD,
    Z_THREAD,
    PERSISTENT_REDUCTION,
    BROADCAST>::
    welfordGroupLastBlock(
        RefWelfordTripletTuple<NumVals, DataType, IndexType>& out,
        const VolatilePtrWelfordTripletTuple<NumVals, DataType, IndexType>&
            global_work_buffer,
        const LocalWelfordTripletTuple<NumVals, DataType, IndexType>& init_val,
        PtrTuple<DataType, DataType, IndexType> shared_buf,
        nvfuser_index_t block_red_idx_offset,
        nvfuser_index_t num_thread_iters,
        nvfuser_index_t num_block_iters,
        nvfuser_index_t thread_red_idx_offset,
        nvfuser_index_t grid_red_size,
        const typename MakeLocalTuple<NumVals, bool>::type& write_preds,
        bool block_reduce_participate,
        bool grid_reduce_participate) {
  // Initialize block result
  auto last_block_result = init_val;

  const bool last_block = index_utils::
      maskedIsLast<isReduce(X_BLOCK), isReduce(Y_BLOCK), isReduce(Z_BLOCK)>(
          blockIdx, gridDim);

  if ((PERSISTENT_REDUCTION || last_block) && grid_reduce_participate) {
    // Can use the last block to reduce all the values the blocks filled in.
    // Can use any thread that has been predicated, or has been reduced to do
    // this reduction, cannot use any block that's associated with an
    // iteration domain

    // Start with non-block reduction

    // Index in the reduction segment
    int tid_in_block_reduction = index_utils::maskedOffset<
        activeNotIter(X_THREAD),
        activeNotIter(Y_THREAD),
        activeNotIter(Z_THREAD)>(threadIdx, blockDim);

    int block_reduction_size = index_utils::maskedSize<
        activeNotIter(X_THREAD),
        activeNotIter(Y_THREAD),
        activeNotIter(Z_THREAD)>(blockDim);

    bool has_block_result = index_utils::maskedIsZero<
        activeNotIter(X_THREAD),
        activeNotIter(Y_THREAD),
        activeNotIter(Z_THREAD)>(threadIdx);

    // 3D buffer of reductions:
    //    [reduction_offset(grid), iter_offset(grid), iter_offset(block)]
    // Change the offset, we want to keep the last two dimensions, but the
    // first dimension is what we will reduce over
    const auto work_buf_offset =
        block_red_idx_offset * num_thread_iters + thread_red_idx_offset;
    for (auto reduction_i = tid_in_block_reduction; reduction_i < grid_red_size;
         reduction_i += block_reduction_size) {
      impl::welfordEach(
          last_block_result,
          0,
          global_work_buffer,
          work_buf_offset + reduction_i * num_block_iters * num_thread_iters);
    }

    // Which block reduction this thread is participating in
    int block_reduction_idx = index_utils::
        maskedOffset<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
            threadIdx, blockDim);

    impl::blockWelfordEach<BROADCAST, false, NumVals, DataType, IndexType>(
        last_block_result,
        last_block_result,
        shared_buf,
        has_block_result,
        tid_in_block_reduction,
        block_reduction_size,
        min(grid_red_size, block_reduction_size),
        block_reduction_idx);

    copyWelfordTripletTupleIf(
        out,
        last_block_result,
        write_preds &&
            (block_reduce_participate && (BROADCAST || has_block_result)));
  }
}

} // namespace fused_reduction
