#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on

namespace nvf {
namespace bq {

constexpr float F4_E2M1_MAX = 6.0;
constexpr float E4M3_EPS = 0.015625;
constexpr float F8E4M3_MAX = 448.0;

__device__ __inline__ void quadMaxReduction(float& local_max) {
  // The mask 0xffffffff indicates all 32 threads in the warp are participating.
  unsigned int mask = 0xffffffff;

  // --- Reduction Step 1 ---
  // Exchange and compare with thread 2 lanes away within the quad.
  // e.g., thread 0 exchanges with 2; thread 1 with 3.
  // The XOR pattern naturally keeps the operation within each quad.
  local_max = fmax(local_max, __shfl_xor_sync(mask, local_max, 2));

  // --- Reduction Step 2 ---
  // Exchange and compare with thread 1 lane away.
  // e.g., thread 0 exchanges with 1; thread 2 with 3.
  local_max = fmax(local_max, __shfl_xor_sync(mask, local_max, 1));

  // At this point, all threads in a quad hold the maximum value for that quad.
}

// only 2 threads compute the max as each thread computes a local
// max of 8 values
__device__ __inline__ void quadMaxReduction(__bfloat& local_max) {
  // The mask 0xffffffff indicates all 32 threads in the warp are participating.
  unsigned int mask = 0xffffffff;

  // --- Reduction Step 1 ---
  // Exchange and compare with thread 2 lanes away within the quad.
  // e.g., thread 0 exchanges with 2; thread 1 with 3.
  // The XOR pattern naturally keeps the operation within each quad.
  float local_max_f = __bfloat2float(local_max);
  local_max_f = fmax(local_max_f, __shfl_xor_sync(mask, local_max_f, 1));
  local_max = __float2bfloat(local_max_f);

  // At this point, all threads in a quad hold the maximum value for that quad.
}

__device__ __inline__ void quadMaxReductionStage1(float& local_max) {
  // The mask 0xffffffff indicates all 32 threads in the warp are participating.
  unsigned int mask = 0xffffffff;

  // --- Reduction Step 1 ---
  // Exchange and compare with thread 2 lanes away within the quad.
  // e.g., thread 0 exchanges with 2; thread 1 with 3.
  // The XOR pattern naturally keeps the operation within each quad.

  local_max = fmax(local_max, __shfl_xor_sync(mask, local_max, 1));

  // At this point, all threads in a quad hold the maximum value for that quad.
}

// TODO: Add a template parameter for input type.
// For now we just work on float.
// This also assumes a block of 16. That should be a
// template parameter.

// This assumes that ITEMS_PER_THREAD is 4.
// This assumes for block quantization, the block size is 16.
// This works for float but will extended to work with bfloat.
template <int ITEMS_PER_THREAD, int ALIGNMENT = 1>
__device__ void block_quantize_to_nvfp4(
    Array<float, ITEMS_PER_THREAD, 1>& input,
    Array<__e2m1, ITEMS_PER_THREAD, ALIGNMENT>& output,
    __e4m3& fp8_output,
    Tensor<float, 0, 0>& global_scale,
    bool use_global_scale = true) {
  assert(blockDim.x % 4 == 0);
  assert(blockDim.z == 1 && gridDim.z == 1);
  static_assert(
      ITEMS_PER_THREAD % 4 == 0, "ITEMS_PER_THREAD must be multiple of 4");

  Array<float, 4, 4> vec4;
  vec4.set(0.0f); // Initialize to zero like nvfuser does

  for (auto i = 0; i < ITEMS_PER_THREAD; i++) {
    vec4[i] = input[i];
  }

  float local_max = NEG_INFINITY;
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    local_max = fmax(local_max, fabsf(vec4[i]));
  }

  // Perform block(16 elements)-wide reduction (max)
  // across 4- threads
  float block_max = NEG_INFINITY;
  quadMaxReduction(local_max);
  block_max = local_max;

  // This division should be replaced with a multiplication
  // by a reciprocal for better performance.
  float scaled_max = block_max / 6.000000000e+00f;
  if (use_global_scale) {
    scaled_max = scaled_max / global_scale[0];
  }

  float clamped_max = clamp(
      scaled_max, 1.562500000e-02f, 4.480000000e+02f); // Clamp between 0 and 1

  __e4m3 clamped_max_fp8 = __float2e4m3(clamped_max);

  float clamped_max_converted = __e4m32float(clamped_max_fp8);
  if (use_global_scale) {
    clamped_max_converted = clamped_max_converted * global_scale[0];
  }

  // Convert back from FP8 to float using __e4m32float
  if (threadIdx.x % 4 == 0) // Only one thread per quad writes
  {
    fp8_output = clamped_max_fp8; // Broadcast to all threads
  }

  Array<float, 4, 4> clamped_vals;
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    float scaled_val = vec4[i] / clamped_max_converted;
    clamped_vals[i] = clamp(scaled_val, -6.000000000e+00f, 6.000000000e+00f);
  }

  Array<__e2m1, 4, 1> fp4_vals;
  *reinterpret_cast<Array<__e2m1, 4, 4>*>(&fp4_vals[0]) =
      __float2e2m1(*reinterpret_cast<Array<float, 4, 4>*>(&clamped_vals[0]));

  // Array<__e2m1, 4, 4> fp4_vals_aligned;
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    output[i] = fp4_vals[i];
  }
}

template <int ITEMS_PER_THREAD, int ALIGNMENT>
__device__ void block_quantize_to_nvfp4(
    Array<float, ITEMS_PER_THREAD, 1>& input,
    Array<__e2m1, ITEMS_PER_THREAD, ALIGNMENT>& output,
    __e4m3& fp8_output) {
  Tensor<float, 0, 0> scale;
  scale[0] = 1.0f;
  block_quantize_to_nvfp4<ITEMS_PER_THREAD>(
      input, output, fp8_output, scale, false);
}

template <int ITEMS_PER_THREAD>
__device__ void block_quantize_bf16_to_nvfp4(
    Array<__bfloat, ITEMS_PER_THREAD, 1>& input,
    Array<__e2m1, ITEMS_PER_THREAD, ITEMS_PER_THREAD>& output,
    __e4m3& fp8_output,
    Tensor<float, 0, 0>& global_scale,
    bool use_global_scale = true) {
  assert(blockDim.x % 2 == 0);
  assert(blockDim.z == 1 && gridDim.z == 1);
  static_assert(
      ITEMS_PER_THREAD % 8 == 0, "ITEMS_PER_THREAD must be multiple of 4");

  // Array<__bfloat, 8, 1> vec4;
  Array<float, 8, 1> vec4;
  vec4.set(0.0f); // Initialize to zero like nvfuser does
  // vec4.set(__bfloat(0)); // Initialize to zero like nvfuser does

  for (auto i = 0; i < ITEMS_PER_THREAD; i++) {
    vec4[i] = __bfloat2float(input[i]);
  }

  float local_max = NEG_INFINITY;
#pragma unroll
  for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
    local_max = fmax(local_max, vec4[i]);
  }

  // Perform block(16 elements)-wide reduction (max)
  // across 4- threads
  float block_max = NEG_INFINITY;
  quadMaxReductionStage1(local_max);
  block_max = local_max;

  // This division should be replaced with a multiplication
  // by a reciprocal for better performance.
  float scaled_max = float(block_max / F4_E2M1_MAX);
  if (use_global_scale) {
    scaled_max = scaled_max * global_scale[0];
  }
  float clamped_max =
      clamp(scaled_max, E4M3_EPS, F8E4M3_MAX); // Clamp between 0 and 1

  __e4m3 clamped_max_fp8 = __float2e4m3(clamped_max);

  float clamped_max_converted = __e4m32float(clamped_max_fp8);

  if (use_global_scale) {
    clamped_max_converted = clamped_max_converted / global_scale[0];
  }

  // Convert back from FP8 to float using __e4m32float
  if (threadIdx.x % 2 == 0) // Only one thread per quad writes
  {
    fp8_output = clamped_max_fp8; // Broadcast to all threads
  }

  Array<float, 8, 1> clamped_vals;
#pragma unroll
  for (int i = 0; i < 8; ++i) {
    float scaled_val = vec4[i] / clamped_max_converted;
    clamped_vals[i] = clamp(scaled_val, -6.000000000e+00f, 6.000000000e+00f);
  }

  Array<__e2m1, 8, 1> fp4_vals;
  *reinterpret_cast<Array<__e2m1, 8, 8>*>(&fp4_vals[0]) =
      __float2e2m1(*reinterpret_cast<Array<float, 8, 8>*>(&clamped_vals[0]));

  // Array<__e2m1, 4, 4> fp4_vals_aligned;
#pragma unroll
  for (int i = 0; i < 8; ++i) {
    output[i] = fp4_vals[i];
  }
}

template <int ITEMS_PER_THREAD, int ALIGNMENT>
__device__ void block_quantize_bf16_to_nvfp4(
    Array<__bfloat, ITEMS_PER_THREAD, 1>& input,
    Array<__e2m1, ITEMS_PER_THREAD, ALIGNMENT>& output,
    __e4m3& fp8_output) {
  Tensor<float, 0, 0> scale;
  scale[0] = 1.0f;
  block_quantize_bf16_to_nvfp4<ITEMS_PER_THREAD>(
      input, output, fp8_output, scale, false);
}

} // namespace bq
} // namespace nvf
