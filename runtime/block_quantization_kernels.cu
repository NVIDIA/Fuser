#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on

namespace nvf {
namespace bq {

template <typename T>
__device__ __inline__ void localMaxReduction(float& local_max) {
  // The mask 0xffffffff indicates all 32 threads in the warp are participating.
  unsigned int mask = 0xffffffff;

  // --- Reduction Step 1 ---
  // Exchange and compare with thread 2 lanes away within the quad.
  // e.g., thread 0 exchanges with 2; thread 1 with 3.
  // The XOR pattern naturally keeps the operation within each quad.
  if (std::is_same<T, float>::value) {
    local_max = fmax(local_max, __shfl_xor_sync(mask, local_max, 2));
  }

  // --- Reduction Step 2 ---
  // Exchange and compare with thread 1 lane away.
  // e.g., thread 0 exchanges with 1; thread 2 with 3.
  local_max = fmax(local_max, __shfl_xor_sync(mask, local_max, 1));

  // At this point, all threads in a quad hold the maximum value for that quad.
}

// TODO: Add a template parameter for input type.
// For now we just work on float.
// This also assumes a block of 16. That should be a
// template parameter.

// This assumes that ITEMS_PER_THREAD is 4.
// This assumes for block quantization, the block size is 16.
// This works for float but will extended to work with bfloat.
template <
    int ITEMS_PER_THREAD,
    typename T,
    int ALIGNMENT_1,
    int ALIGNMENT_2,
    int BLOCK_SCALE_DIM,
    int BLOCK_SCALE_ALLOC>
__device__ void block_quantize_to_nvfp4(
    const Array<T, ITEMS_PER_THREAD, ALIGNMENT_1>& input,
    Array<__e2m1, ITEMS_PER_THREAD, ALIGNMENT_2>& output,
    Tensor<__e4m3, BLOCK_SCALE_DIM, BLOCK_SCALE_ALLOC>& block_scales,
    nvfuser_index_t logical_index,
    int input_logical_inner_dim_size) {
  constexpr bool is_half_or_bfloat =
      std::is_same<T, __bfloat>::value || std::is_same<T, __half>::value;
  constexpr bool is_float = std::is_same<T, float>::value;
  static_assert(
      is_float || is_half_or_bfloat,
      "Input type must be float, __half or __bfloat");

  if constexpr (is_float) {
    assert(blockDim.x % 4 == 0);
  } else if constexpr (is_half_or_bfloat) {
    assert(blockDim.x % 2 == 0);
  }

  static_assert(
      (is_float && ITEMS_PER_THREAD == 4) ||
          (is_half_or_bfloat && ITEMS_PER_THREAD == 8),
      "ITEMS_PER_THREAD must be 4 for float type or 8 for __bfloat or __half "
      "type");

  assert(input_logical_inner_dim_size % 16 == 0);

  int THREADS_PER_SCALING_FACTOR = 16 / ITEMS_PER_THREAD;

  Array<float, ITEMS_PER_THREAD, ITEMS_PER_THREAD> vec_in;
  vec_in.set(0.0f); // Initialize to zero like nvfuser does

  for (auto i = 0; i < ITEMS_PER_THREAD; i++) {
    if constexpr (std::is_same<T, float>::value) {
      vec_in[i] = input[i];
    } else if constexpr (std::is_same<T, __bfloat>::value) {
      vec_in[i] = __bfloat2float(input[i]);
    } else if constexpr (std::is_same<T, __half>::value) {
      vec_in[i] = __half2float(input[i]);
    }
  }

  float local_max = NEG_INFINITY;
#pragma unroll
  for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
    local_max = fmax(local_max, fabsf(vec_in[i]));
  }

  // Perform block(16 elements)-wide reduction (max)
  // across 4- threads
  float block_max = NEG_INFINITY;
  localMaxReduction<T>(local_max);
  block_max = local_max;

  // This division should be replaced with a multiplication
  // by a reciprocal for better performance.
  float scaled_max = block_max / 6.000000000e+00f;
  float clamped_max = clamp(
      scaled_max, 1.562500000e-02f, 4.480000000e+02f); // Clamp between 0 and 1

  __e4m3 clamped_max_fp8 = __float2e4m3(clamped_max);

  float clamped_max_converted = __e4m32float(clamped_max_fp8);

  int offset_y_blocks = blockIdx.y * blockDim.y * blockDim.x * gridDim.x;
  int offset_dim_y = threadIdx.y * blockDim.x * gridDim.x;
  int offset_into_block = blockIdx.x * blockDim.x + threadIdx.x;

  int offset = logical_index / 16;

  // Convert back from FP8 to float using __e4m32float
  if (threadIdx.x % THREADS_PER_SCALING_FACTOR == 0) {
    fp8_output[offset] = clamped_max_fp8; // Broadcast to all threads
  }

  Array<float, ITEMS_PER_THREAD, ITEMS_PER_THREAD> clamped_vals;
#pragma unroll
  for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
    float scaled_val = vec_in[i] / clamped_max_converted;
    clamped_vals[i] = clamp(scaled_val, -6.000000000e+00f, 6.000000000e+00f);
  }

  Array<__e2m1, ITEMS_PER_THREAD, 1> fp4_vals;
  *reinterpret_cast<Array<__e2m1, ITEMS_PER_THREAD, ITEMS_PER_THREAD>*>(
      &fp4_vals[0]) =
      __float2e2m1(
          *reinterpret_cast<Array<float, ITEMS_PER_THREAD, ITEMS_PER_THREAD>*>(
              &clamped_vals[0]));

#pragma unroll
  for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
    output[i] = fp4_vals[i];
  }
}

} // namespace bq
} // namespace nvf
