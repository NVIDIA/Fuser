#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on

#define __NVFUSER_HALF_TO_US(var) *(reinterpret_cast<unsigned short*>(&(var)))
#define __NVFUSER_HALF_TO_CUS(var) \
  *(reinterpret_cast<const unsigned short*>(&(var)))

struct __half;
__device__ __inline__ __half __float2half(const float);

struct __align__(2) __half {
  __half() = default;

  __half(const __half& other) {
    __x = other.__x;
  }

  __half(const __half&& other) {
    __x = other.__x;
  }

  __half(const volatile __half& other) {
    __x = other.__x;
  }

  __half(const volatile __half&& other) {
    __x = other.__x;
  }

  // Note: not returning reference for `__half::operator=`
  // Doing so would requires us to return `volatile __half&` for the volatile
  // variants, which would trigger a gcc warning `implicit dereference will not
  // access object of type ‘volatile S’ in statement`
  __device__ void operator=(const __half& other) {
    __x = other.__x;
  }

  __device__ void operator=(const __half&& other) {
    __x = other.__x;
  }

  __device__ void operator=(const volatile __half& other) {
    __x = other.__x;
  }

  __device__ void operator=(const volatile __half&& other) {
    __x = other.__x;
  }

  __device__ void operator=(const __half& other) volatile {
    __x = other.__x;
  }

  __device__ void operator=(const __half&& other) volatile {
    __x = other.__x;
  }

  __device__ void operator=(const volatile __half& other) volatile {
    __x = other.__x;
  }

  __device__ void operator=(const volatile __half&& other) volatile {
    __x = other.__x;
  }

  __device__ __half(const float f) {
    __x = __float2half(f).__x;
  }

  __device__ uint16_t raw() const {
    return __x;
  }

 protected:
  unsigned short __x;
};

__device__ __inline__ __half __float2half(const float f) {
  __half val;
  asm("{  cvt.rn.f16.f32 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "f"(f));
  return val;
}

__device__ __inline__ __half __double2half(const double d) {
  __half val;
  asm("{  cvt.rn.f16.f64 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "d"(d));
  return val;
}

__device__ __inline__ __half __int2half(const int i) {
  __half val;
  asm("{  cvt.rn.f16.s32 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "r"(i));
  return val;
}

__device__ __inline__ __half __int2half(const int64_t i64) {
  __half val;
  asm("{  cvt.rn.f16.s64 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "l"(i64));
  return val;
}

__device__ __inline__ __half __int2half(const uint32_t i) {
  __half val;
  asm("{  cvt.rn.f16.u32 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "r"(i));
  return val;
}

__device__ __inline__ __half __int2half(const uint64_t i64) {
  __half val;
  asm("{  cvt.rn.f16.u64 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "l"(i64));
  return val;
}

__device__ __inline__ __half __bool2half(const bool b) {
  return __int2half((int)b);
}

__device__ __inline__ float __half2float(const __half h) {
  float val;
  asm("{  cvt.f32.f16 %0, %1;}\n" : "=f"(val) : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
}

__device__ __inline__ double __half2double(const __half h) {
  double val;
  asm("{  cvt.f64.f16 %0, %1;}\n" : "=d"(val) : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
}

__device__ int __half2int32(const __half h) {
  int val;
  asm("{  cvt.rzi.s32.f16 %0, %1;}\n"
      : "=r"(val)
      : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
}

__device__ __inline__ int64_t __half2int(const __half h) {
  int64_t val;
  asm("{  cvt.rzi.s64.f16 %0, %1;}\n"
      : "=l"(val)
      : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
}

__device__ int __half2uint32(const __half h) {
  int val;
  asm("{  cvt.rzi.u32.f16 %0, %1;}\n"
      : "=r"(val)
      : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
}

__device__ __inline__ int64_t __half2uint(const __half h) {
  int64_t val;
  asm("{  cvt.rzi.u64.f16 %0, %1;}\n"
      : "=l"(val)
      : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
}

__device__ __inline__ void __half2int(const __half h, int& output) {
  output = __half2int32(h);
}

__device__ __inline__ void __half2int(const __half h, int64_t& output) {
  output = __half2int(h);
}

__device__ __inline__ void __half2int(const __half h, uint32_t& output) {
  output = __half2uint32(h);
}

__device__ __inline__ void __half2int(const __half h, uint64_t& output) {
  output = __half2uint(h);
}

__device__ __inline__ nvfuser_index_t __half2index(const __half h) {
  nvfuser_index_t result;
  __half2int(h, result);
  return result;
}

__device__ __inline__ bool __half2bool(const __half h) {
  return (bool)__half2float(h) != 0;
}

__device__ __inline__ __half __real_then_2half(const std::complex<float> c) {
  return __float2half(std::real(c));
}

__device__ __inline__ __half __real_then_2half(const std::complex<double> c) {
  return __double2half(std::real(c));
}

__device__ __inline__ bool __heq(const __half a, const __half b) {
  // From cuda_fp16.hpp
  unsigned short val;
  asm("{ .reg .pred __$temp3;\n"
      "  setp.eq.f16  __$temp3, %1, %2;\n"
      "  selp.u16 %0, 1, 0, __$temp3;}"
      : "=h"(val)
      : "h"(__NVFUSER_HALF_TO_CUS(a)), "h"(__NVFUSER_HALF_TO_CUS(b)));
  return (val != 0U) ? true : false;
}

__device__ __inline__ __half operator|(const __half x, const __half y) {
  __half val;
  asm("{  or.b16 %0, %1, %2;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "h"(__NVFUSER_HALF_TO_CUS(x)), "h"(__NVFUSER_HALF_TO_CUS(y)));
  return val;
}
