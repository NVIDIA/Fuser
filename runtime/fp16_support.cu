#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on

#define __NVFUSER_HALF_TO_US(var) *(reinterpret_cast<unsigned short*>(&(var)))
#define __NVFUSER_HALF_TO_CUS(var) \
  *(reinterpret_cast<const unsigned short*>(&(var)))

struct __half;
__device__ __inline__ __half __float2half(const float);

struct __align__(2) __half {
  __half() = default;

  __device__ __half(const float f) {
    __x = __float2half(f).__x;
  }

  __device__ uint16_t raw() const {
    return __x;
  }

 protected:
  unsigned short __x;
};

__device__ __inline__ __half __float2half(const float f) {
  __half val;
  asm("{  cvt.rn.f16.f32 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "f"(f));
  return val;
}

__device__ __inline__ __half __double2half(const double d) {
  __half val;
  asm("{  cvt.rn.f16.f64 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "d"(d));
  return val;
}

__device__ __inline__ __half __int322half(const int i) {
  __half val;
  asm("{  cvt.rn.f16.s32 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "r"(i));
  return val;
}

__device__ __inline__ __half __int2half(const int64_t i64) {
  __half val;
  asm("{  cvt.rn.f16.s64 %0, %1;}\n"
      : "=h"(__NVFUSER_HALF_TO_US(val))
      : "l"(i64));
  return val;
}

__device__ __inline__ __half __bool2half(const bool b) {
  return __int2half((int)b);
}

__device__ __inline__ float __half2float(const __half h) {
  float val;
  asm("{  cvt.f32.f16 %0, %1;}\n" : "=f"(val) : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
}

__device__ __inline__ double __half2double(const __half h) {
  double val;
  asm("{  cvt.f64.f16 %0, %1;}\n" : "=d"(val) : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
}

__device__ int __half2int32(const __half h) {
  int val;
  asm("{  cvt.rzi.s32.f16 %0, %1;}\n"
      : "=r"(val)
      : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
}

__device__ __inline__ int64_t __half2int(const __half h) {
  int64_t val;
  asm("{  cvt.rzi.s64.f16 %0, %1;}\n"
      : "=l"(val)
      : "h"(__NVFUSER_HALF_TO_CUS(h)));
  return val;
}

__device__ __inline__ bool __half2bool(const __half h) {
  return (bool)__half2float(h) != 0;
}

__device__ __inline__ __half __real_then_2half(const std::complex<float> c) {
  return __float2half(std::real(c));
}

__device__ __inline__ __half __real_then_2half(const std::complex<double> c) {
  return __double2half(std::real(c));
}
