// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on

namespace broadcast {
// Broadcasts within partitioned groups of threads.
//
// X_THREAD: Broadcast from threadIdx.x == 0 if true
// Y_THREAD: Broadcast from threadIdx.y == 0 if true
// Z_THREAD: Broadcast from threadIdx.z == 0 if true
// inp_val: Per-thread source value. Only valid when the thread is a source.
// out: Per-thread output location
//
template <bool X_THREAD, bool Y_THREAD, bool Z_THREAD, typename T>
__device__ void blockBroadcast(
    T& out,
    const T& inp_val,
    T* shared_mem,
    bool read_write_pred) {
  const bool has_valid_data = (!X_THREAD || threadIdx.x == 0) &&
      (!Y_THREAD || threadIdx.y == 0) && (!Z_THREAD || threadIdx.z == 0);

  const auto shared_offset =
      index_utils::maskedOffset<!X_THREAD, !Y_THREAD, !Z_THREAD>(
          threadIdx, blockDim);

  if (has_valid_data && read_write_pred) {
    shared_mem[shared_offset] = inp_val;
  }

  block_sync::sync();

  if (read_write_pred) {
    out = shared_mem[shared_offset];
  }

  block_sync::sync();
}

} // namespace broadcast
