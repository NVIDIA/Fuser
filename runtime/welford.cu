#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
template <typename DataType>
struct WelfordTriplet {
  DataType avg;
  DataType var;
  nvfuser_index_t N;
};

template <typename DataType>
__inline__ __device__ void copyTriplet(
    DataType* dst_avg,
    DataType* dst_var,
    nvfuser_index_t* dst_N,
    const WelfordTriplet<DataType>& src) {
  *dst_avg = src.avg;
  *dst_var = src.var;
  *dst_N = src.N;
}

template <typename DataType>
__inline__ __device__ void copyTriplet(
    WelfordTriplet<DataType>& dst,
    const DataType* src_avg,
    const DataType* src_var,
    const nvfuser_index_t* src_N) {
  dst.avg = *src_avg;
  dst.var = *src_var;
  dst.N = *src_N;
}

template <typename DataType>
__inline__ __device__ void copyTriplet(
    WelfordTriplet<DataType>& dst,
    const WelfordTriplet<DataType>& src) {
  dst.avg = src.avg;
  dst.var = src.var;
  dst.N = src.N;
}

// -----------------------------------------------------------------------------------------------
//  Block Welford Primitives
// -----------------------------------------------------------------------------------------------
// Basic utility for welford update. Can be used to scan one value, or two merge
// two welford results
template <typename T, typename TN>
__inline__ __device__ void welfordCombine(
    T& a_avg,
    T& a_M2,
    TN& a_N,
    const T b_avg,
    const T b_M2,
    TN b_N) {
  if (b_N == 0) {
    return;
  }
  TN ab_N = a_N + b_N;
  T b_N_div_ab_N = ((T)(nvfuser_index_t)(b_N)) / ((T)(nvfuser_index_t)(ab_N));
  T delta = b_avg - a_avg;
  a_avg += delta * b_N_div_ab_N;
  a_M2 += b_M2 + delta * delta * ((T)(nvfuser_index_t)(a_N)) * b_N_div_ab_N;
  a_N = ab_N;
}

template <typename T, bool OutputGmem>
__inline__ __device__ void welfordVectorized(
    T& a_avg,
    T& a_M2,
    nvfuser_index_t& a_N,
    const T b_avg,
    const T b_N_div_ab_N,
    const nvfuser_index_t ab_N,
    const bool pred) {
  // Want only predicated statements and don't want to have
  // "if", but for gmem output writes can be illegal, so needs to
  // bail out here.
  if (OutputGmem && !pred) {
    return;
  }
  T predicated_b_avg = pred ? b_avg : a_avg;
  T delta0 = predicated_b_avg - a_avg;
  a_avg += delta0 * b_N_div_ab_N;
  T delta1 = predicated_b_avg - a_avg;
  a_M2 += delta0 * delta1;
  a_N = ab_N;
}

// Non predicated version
template <typename T>
__inline__ __device__ void welfordVectorized(
    T& a_avg,
    T& a_M2,
    nvfuser_index_t& a_N,
    const T b_avg,
    const T b_N_div_ab_N,
    const nvfuser_index_t ab_N) {
  T delta0 = b_avg - a_avg;
  a_avg += delta0 * b_N_div_ab_N;
  T delta1 = b_avg - a_avg;
  a_M2 += delta0 * delta1;
  a_N = ab_N;
}

// [Z,Y,X]_THREADS is the number of participating threads in the z, y, x
// dimension of the block.
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    typename T,
    typename TN,
    typename _dim3,
    typename _dim3_2>
__inline__ __device__ void blockWelford(
    T& out_avg,
    T& out_M2,
    TN& out_N,
    const T& in_avg,
    const T& in_M2,
    const TN& in_N,
    const _dim3& thread_idx,
    const _dim3_2& block_dim,
    T* shared_mem_avg,
    T* shared_mem_M2,
    TN* shared_mem_N,
    bool read_pred,
    bool write_pred,
    T init_val) {
  // If this thread will output a final result
  bool should_write =
      index_utils::maskedIsZero<X_REDUCE, Y_REDUCE, Z_REDUCE>(thread_idx);

  // Size of the reduction segments
  unsigned int reduction_size =
      index_utils::maskedSize<X_REDUCE, Y_REDUCE, Z_REDUCE>(block_dim);

  // Index into the reduction segment
  unsigned int reduction_tid =
      index_utils::maskedOffset<X_REDUCE, Y_REDUCE, Z_REDUCE>(
          thread_idx, block_dim);

  // Index of the reduction segment
  unsigned int reduction_idx =
      index_utils::maskedOffset<!X_REDUCE, !Y_REDUCE, !Z_REDUCE>(
          thread_idx, block_dim);

  // Offset into smem for the current thread
  unsigned int smem_offset = reduction_idx * reduction_size + reduction_tid;

  if (read_pred) {
    shared_mem_avg[smem_offset] = in_avg;
    shared_mem_M2[smem_offset] = in_M2;
    shared_mem_N[smem_offset] = in_N;
  } else {
    shared_mem_avg[smem_offset] = init_val;
    shared_mem_M2[smem_offset] = init_val;
    shared_mem_N[smem_offset] = 0;
  }

  block_sync::sync();
  // Reduce down to nearest power of 2:
  int np2 = 1 << (31 - __clz(reduction_size));

  if (reduction_tid < np2 && reduction_tid + np2 < reduction_size) {
    welfordCombine(
        shared_mem_avg[smem_offset],
        shared_mem_M2[smem_offset],
        shared_mem_N[smem_offset],
        shared_mem_avg[smem_offset + np2],
        shared_mem_M2[smem_offset + np2],
        shared_mem_N[smem_offset + np2]);
  }
  block_sync::sync();

  // loop peel the final iteration to save one syncthread for the end
  for (int factor = np2 / 2; factor > 1; factor >>= 1) {
    if (reduction_tid < factor) {
      welfordCombine(
          shared_mem_avg[smem_offset],
          shared_mem_M2[smem_offset],
          shared_mem_N[smem_offset],
          shared_mem_avg[smem_offset + factor],
          shared_mem_M2[smem_offset + factor],
          shared_mem_N[smem_offset + factor]);
    }
    block_sync::sync();
  }

  if (should_write && write_pred) {
    T res_avg = out_avg;
    T res_M2 = out_M2;
    TN res_N = out_N;
    welfordCombine(
        res_avg,
        res_M2,
        res_N,
        shared_mem_avg[smem_offset],
        shared_mem_M2[smem_offset],
        shared_mem_N[smem_offset]);
    if (reduction_size > 1) {
      welfordCombine(
          res_avg,
          res_M2,
          res_N,
          shared_mem_avg[smem_offset + 1],
          shared_mem_M2[smem_offset + 1],
          shared_mem_N[smem_offset + 1]);
    }
    out_avg = res_avg;
    out_M2 = res_M2;
    out_N = res_N;
  }
  block_sync::sync();
}

// Use the same pred for both reads and writes
template <
    bool X_REDUCE,
    bool Y_REDUCE,
    bool Z_REDUCE,
    typename T,
    typename TN,
    typename _dim3,
    typename _dim3_2>
__inline__ __device__ void blockWelford(
    T& out_avg,
    T& out_M2,
    TN& out_N,
    const T& in_avg,
    const T& in_M2,
    const TN& in_N,
    const _dim3& thread_idx,
    const _dim3_2& block_dim,
    T* shared_mem_avg,
    T* shared_mem_M2,
    TN* shared_mem_N,
    bool read_write_pred,
    T init_val) {
  blockWelford<X_REDUCE, Y_REDUCE, Z_REDUCE, T, TN, _dim3, _dim3_2>(
      out_avg,
      out_M2,
      out_N,
      in_avg,
      in_M2,
      in_N,
      thread_idx,
      block_dim,
      shared_mem_avg,
      shared_mem_M2,
      shared_mem_N,
      read_write_pred,
      read_write_pred,
      init_val);
}
// -----------------------------------------------------------------------------------------------
//  Grid Welford Prototype
// -----------------------------------------------------------------------------------------------
namespace welford {

template <bool X_THREAD, bool Y_THREAD, bool Z_THREAD, typename T, typename TN>
__device__ void gridWelfordLastBlock(
    T& out_avg,
    T& out_M2,
    TN& out_N,
    const volatile T* in_avg,
    const volatile T* in_M2,
    const volatile TN* in_N,
    const nvfuser_index_t
        grid_reduction_segment_size, // Number of reductions across
                                     // grid reduce dimensions
    const nvfuser_index_t
        block_reduction_segment_size, // Number of reductions across the block
    T* shared_buf_avg,
    T* shared_buf_M2,
    TN* shared_buf_N,
    bool write_pred,
    T init_val) {
  // We have to do num_reductions across reduction_size. The reductions are
  // contiguous, but offset by reduction_size. There is an entry in "in" for
  // every block, and every thread marked as true. Threads in dimensions marked
  // as false can be used to parallelize the reduction.

  // Find the reduction id of the participating threads
  const auto block_reduction_segment_idx =
      index_utils::maskedOffset<X_THREAD, Y_THREAD, Z_THREAD>(
          threadIdx, blockDim);

  // Find an id associated within a reduction segment for all
  // "non-participating" threads, which will parallelize the reductions for the
  // "participating" threads
  const auto id_in_block_segment =
      index_utils::maskedOffset<!X_THREAD, !Y_THREAD, !Z_THREAD>(
          threadIdx, blockDim);

  // Stride by the "non-participating" threads
  const auto input_stride_for_thread_in_segment =
      index_utils::maskedSize<!X_THREAD, !Y_THREAD, !Z_THREAD>(blockDim);

  T inp_avg = init_val;
  T inp_M2 = init_val;
  TN inp_N = 0;

  // Block stride across the reduction until we only have one value per thread
  for (nvfuser_index_t reduction_i = id_in_block_segment;
       reduction_i < grid_reduction_segment_size;
       reduction_i += input_stride_for_thread_in_segment) {
    auto work_buf_offset = reduction_i * block_reduction_segment_size +
        block_reduction_segment_idx;
    welfordCombine(
        inp_avg,
        inp_M2,
        inp_N,
        in_avg[work_buf_offset],
        in_M2[work_buf_offset],
        in_N[work_buf_offset]);
  }

  // Block reduce the per thread values into per "participating" thread values
  T inp_avg_tmp = init_val;
  T inp_M2_tmp = init_val;
  TN inp_N_tmp = 0;
  blockWelford<!X_THREAD, !Y_THREAD, !Z_THREAD>(
      inp_avg_tmp,
      inp_M2_tmp,
      inp_N_tmp,
      inp_avg,
      inp_M2,
      inp_N,
      threadIdx,
      blockDim,
      shared_buf_avg,
      shared_buf_M2,
      shared_buf_N,
      true,
      init_val);
  const bool should_write = (X_THREAD || threadIdx.x == 0) &&
      (Y_THREAD || threadIdx.y == 0) && (Z_THREAD || threadIdx.z == 0);
  if (should_write && write_pred) {
    welfordCombine(out_avg, out_M2, out_N, inp_avg_tmp, inp_M2_tmp, inp_N_tmp);
  }
}

// Grid welford combine. See GridReduction for more information
template <
    bool X_BLOCK,
    bool Y_BLOCK,
    bool Z_BLOCK,
    bool X_THREAD,
    bool Y_THREAD,
    bool Z_THREAD,
    bool PERSISTENT_REDUCTION,
    typename T,
    typename TN>
__device__ void gridWelford(
    T& out_avg,
    T& out_M2,
    TN& out_N,
    const T& inp_avg,
    const T& inp_M2,
    const TN& inp_N,
    volatile T* work_buf_avg,
    volatile T* work_buf_M2,
    volatile TN* work_buf_N,
    Tensor<int64_t, 1> sync_flags,
    T* shared_buf_avg,
    T* shared_buf_M2,
    TN* shared_buf_N,
    bool read_pred,
    bool write_pred,
    T init_val,
    const nvfuser_index_t entrance_ind,
    const nvfuser_index_t n_entrances) {
  // entrance index only matters for non-persistent re-entrant grid reductions.
  const nvfuser_index_t entrance_ind_ = PERSISTENT_REDUCTION ? 0 : entrance_ind;
  const nvfuser_index_t n_entrances_ = PERSISTENT_REDUCTION ? 1 : n_entrances;

  // Number of values to reduce in the reduction segment
  const auto grid_reduction_segment_size =
      index_utils::maskedSize<X_BLOCK, Y_BLOCK, Z_BLOCK>(gridDim);

  // Index of the reduction we're performing out of the
  // grid_reduction_segment_size
  const auto idx_in_grid_segment =
      index_utils::maskedOffset<!X_BLOCK, !Y_BLOCK, !Z_BLOCK>(
          blockIdx, gridDim);

  // Number of threads we can use in final reduction, Seems to assume all
  // threads in the block participate
  const auto block_reduction_segment_size =
      index_utils::maskedSize<X_THREAD, Y_THREAD, Z_THREAD>(blockDim);

  // Number of reductions in the grid
  const nvfuser_index_t grid_segment_size = PERSISTENT_REDUCTION
      ? 1
      : index_utils::maskedSize<!X_BLOCK, !Y_BLOCK, !Z_BLOCK>(gridDim);

  // advance to the offset for this segment
  // index of reduction * size of the reduction * size of threads
  work_buf_avg += (entrance_ind_ * grid_segment_size + idx_in_grid_segment) *
      grid_reduction_segment_size * block_reduction_segment_size;
  work_buf_M2 += (entrance_ind_ * grid_segment_size + idx_in_grid_segment) *
      grid_reduction_segment_size * block_reduction_segment_size;
  work_buf_N += (entrance_ind_ * grid_segment_size + idx_in_grid_segment) *
      grid_reduction_segment_size * block_reduction_segment_size;

  if ((X_THREAD || threadIdx.x == 0) && (Y_THREAD || threadIdx.y == 0) &&
      (Z_THREAD || threadIdx.z == 0)) {
    auto block_offset =
        index_utils::maskedOffset<X_BLOCK, Y_BLOCK, Z_BLOCK>(blockIdx, gridDim);
    auto thread_offset =
        index_utils::maskedOffset<X_THREAD, Y_THREAD, Z_THREAD>(
            threadIdx, blockDim);
    auto work_buf_offset =
        block_offset * block_reduction_segment_size + thread_offset;
    if (read_pred) {
      work_buf_avg[work_buf_offset] = inp_avg;
      work_buf_M2[work_buf_offset] = inp_M2;
      work_buf_N[work_buf_offset] = inp_N;
    } else {
      work_buf_avg[work_buf_offset] = init_val;
      work_buf_M2[work_buf_offset] = init_val;
      work_buf_N[work_buf_offset] = 0;
    }
  }

  if (PERSISTENT_REDUCTION) {
    grid_sync::sync<X_BLOCK, Y_BLOCK, Z_BLOCK, PERSISTENT_REDUCTION>(
        sync_flags[idx_in_grid_segment], grid_reduction_segment_size);
  } else {
    // Use a different sync flag for each call
    grid_sync::sync<X_BLOCK, Y_BLOCK, Z_BLOCK, PERSISTENT_REDUCTION>(
        sync_flags[entrance_ind_ * grid_segment_size + idx_in_grid_segment],
        grid_reduction_segment_size);
  }

  bool last_block =
      index_utils::maskedIsLast<X_BLOCK, Y_BLOCK, Z_BLOCK>(blockIdx, gridDim);

  if (last_block) {
    // final reduction
    gridWelfordLastBlock<X_THREAD, Y_THREAD, Z_THREAD>(
        out_avg,
        out_M2,
        out_N,
        work_buf_avg,
        work_buf_M2,
        work_buf_N,
        grid_reduction_segment_size,
        block_reduction_segment_size,
        shared_buf_avg,
        shared_buf_M2,
        shared_buf_N,
        write_pred,
        init_val);
  }

  if (PERSISTENT_REDUCTION) {
    // Make sure we're done with global memory before we allow the kernel to
    // continue
    grid_sync::sync<X_BLOCK, Y_BLOCK, Z_BLOCK, PERSISTENT_REDUCTION>(
        sync_flags[idx_in_grid_segment], grid_reduction_segment_size);
  }
}

} // namespace welford
