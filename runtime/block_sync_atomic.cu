
#include <hip/hip_runtime.h>
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on

// Counter-based block synchronization. Only meant to be used for
// debugging and validating synchronization. This should be replaced
// with cuda::barrier::arrive_and_wait as that should be more robust.

namespace block_sync {

using CounterType = unsigned int;
static constexpr CounterType COUNTER_TYPE_MAX = ~(CounterType)0;
__shared__ CounterType sync_counter;

__device__ void init() {
  const unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x +
      threadIdx.z * blockDim.x * blockDim.y;
  if (tid == 0) {
    sync_counter = 0;
  }
  __syncthreads();
}

// Emulate __syncthreads() with a synchronization counter
__device__ void sync() {
  unsigned int backoff = 8;
  const unsigned int backoff_max = 256;
  const unsigned int num_threads = blockDim.x * blockDim.y * blockDim.z;

  __threadfence_block();

  // Use counter range only up to a limit so that the next val won't
  // overflow.

  const auto counter_max = (COUNTER_TYPE_MAX / num_threads) * num_threads;
  const auto old = atomicInc(&sync_counter, counter_max - 1);

  const auto next = (old / num_threads) * num_threads + num_threads;

  auto local_sync_counter = *(volatile CounterType*)(&sync_counter);

  // sync_counter may wrap around, which means local_sync_counter
  // becomes smaller than old. In that case, it's guaranteed that all
  // threads have incremented the counter.
  while (local_sync_counter < next && old < local_sync_counter) {
#if __CUDA_ARCH__ >= 700
    // __nanosleep only available on compute capability 7.0 or higher
    __nanosleep(backoff); // avoids busy waiting
#endif
    if (backoff < backoff_max) {
      backoff *= 2;
    }
    local_sync_counter = *(volatile CounterType*)(&sync_counter);
  }
}

} // namespace block_sync
