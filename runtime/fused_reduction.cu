#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
namespace fused_reduction {

namespace impl {

//! Suppose f_i be the i-th function of the binary function
//! parameters. Call the function as: f_i(x, y)
template <int i, typename DataType, typename Func, typename... Funcs>
struct FuncSelector {
  static __device__ void call(
      DataType& x,
      const DataType y,
      Func f,
      Funcs... funcs) {
    // Here, i is guaranteed to be larger than 0 as there's a
    // specialization for i == 0 below. Recursively call FuncSelector
    // by dropping f and decrementing i.
    FuncSelector<i - 1, DataType, Funcs...>::call(x, y, funcs...);
  }
};

//! Specialization of FuncSelector when i == 0, so f_i is f.
template <typename DataType, typename Func, typename... Funcs>
struct FuncSelector<0, DataType, Func, Funcs...> {
  static __device__ void call(
      DataType& x,
      const DataType y,
      Func f,
      Funcs... funcs) {
    f(x, y);
  }
};

//! Call each of the first i+1 functions with the first i+1 values of
//! tuples. Here, i is guaranteed to be larger than -1 as there's a
//! specialization for i == -1.
template <int i, typename TupleType0, typename TupleType1, typename... Funcs>
struct FuncForEach {
  static __device__ void call(
      TupleType0& val0,
      nvfuser_index_t offset0,
      const TupleType1& val1,
      nvfuser_index_t offset1,
      Funcs... funcs) {
    static_assert(
        IsSameType<
            typename TupleType0::template ValType<i>,
            typename TupleType1::template ValType<i>>::value,
        "Invalid tuple types");
    // Process the first i functions first.
    FuncForEach<i - 1, TupleType0, TupleType1, Funcs...>::call(
        val0, offset0, val1, offset1, funcs...);
    // Call the i+1-th function
    FuncSelector<i, typename TupleType0::template ValType<i>, Funcs...>::call(
        val0.val<i>(offset0), val1.val<i>(offset1), funcs...);
  }
};

//! Specialization of FuncForEach when i == -1, which means no
//! function to call. Just for stopping the recursive pattern here.
template <typename TupleType0, typename TupleType1, typename... Funcs>
struct FuncForEach<-1, TupleType0, TupleType1, Funcs...> {
  static __device__ void call(
      TupleType0& val0,
      nvfuser_index_t offset0,
      const TupleType1& val1,
      nvfuser_index_t offset1,
      Funcs... funcs) {}
};

//! Reduce one value of a tuple using one of the reduction ops. The
//! value at val_idx is reduced by the function at func_idx.
template <
    int func_idx,
    int val_idx,
    typename TupleType0,
    typename TupleType1,
    typename... Funcs>
__inline__ __device__ static void reduceVal(
    TupleType0& val0,
    nvfuser_index_t offset0,
    const TupleType1& val1,
    nvfuser_index_t offset1,
    Funcs... reduction_ops) {
  static_assert(
      IsSameType<
          typename TupleType0::template ValType<val_idx>,
          typename TupleType1::template ValType<val_idx>>::value,
      "Invalid tuple types");
  FuncSelector<
      func_idx,
      typename TupleType0::template ValType<val_idx>,
      Funcs...>::
      call(
          val0.val<val_idx>(offset0),
          val1.val<val_idx>(offset1),
          reduction_ops...);
}

//! Accumulate each value of a given pair of tuples using its corresponding
//! function. Suppose f_i be the i-th reduciton function. Call f_i as:
//! f_i(val0.val<i>(offset0), val1.val<i>(offset1)).
template <typename TupleType0, typename TupleType1, typename... Funcs>
__inline__ __device__ static void reduceEach(
    TupleType0& val0,
    nvfuser_index_t offset0,
    const TupleType1& val1,
    nvfuser_index_t offset1,
    Funcs... reduction_ops) {
  constexpr int num_funcs = sizeof...(reduction_ops);
  FuncForEach<num_funcs - 1, TupleType0, TupleType1, Funcs...>::call(
      val0, offset0, val1, offset1, reduction_ops...);
}

template <typename TupleType0, typename TupleType1, typename Func, int num_vals>
struct TupleReduce {};

template <typename TupleType0, typename TupleType1, typename Func>
struct TupleReduce<TupleType0, TupleType1, Func, 1> {
  __inline__ __device__ static void reduce(
      TupleType0& val0,
      nvfuser_index_t offset0,
      const TupleType1& val1,
      nvfuser_index_t offset1,
      Func reduction_op) {
    static_assert(
        IsSameType<
            typename TupleType0::ValTypes,
            typename TupleType1::ValTypes>::value,
        "Invalid value types");
    reduction_op(val0.val<0>(offset0), val1.val<0>(offset1));
  }
};

template <typename TupleType0, typename TupleType1, typename Func>
struct TupleReduce<TupleType0, TupleType1, Func, 2> {
  __inline__ __device__ static void reduce(
      TupleType0& val0,
      nvfuser_index_t offset0,
      const TupleType1& val1,
      nvfuser_index_t offset1,
      Func reduction_op) {
    static_assert(
        IsSameType<
            typename TupleType0::ValTypes,
            typename TupleType1::ValTypes>::value,
        "Invalid value types");
    reduction_op(
        val0.val<0>(offset0),
        val0.val<1>(offset0),
        val1.val<0>(offset1),
        val1.val<1>(offset1));
  }
};

template <typename TupleType0, typename TupleType1, typename Func>
struct TupleReduce<TupleType0, TupleType1, Func, 3> {
  __inline__ __device__ static void reduce(
      TupleType0& val0,
      nvfuser_index_t offset0,
      const TupleType1& val1,
      nvfuser_index_t offset1,
      Func reduction_op) {
    static_assert(
        IsSameType<
            typename TupleType0::ValTypes,
            typename TupleType1::ValTypes>::value,
        "Invalid value types");
    reduction_op(
        val0.val<0>(offset0),
        val0.val<1>(offset0),
        val0.val<2>(offset0),
        val1.val<0>(offset1),
        val1.val<1>(offset1),
        val1.val<2>(offset1));
  }
};

//! Reduce all values of a tuple together. The reduction function must
//! have the same number of inputs as the number of values of each tuple.
template <typename TupleType0, typename TupleType1, typename Func>
__inline__ __device__ void reduceTuple(
    TupleType0& val0,
    nvfuser_index_t offset0,
    const TupleType1& val1,
    nvfuser_index_t offset1,
    Func reduction_op) {
  static_assert(
      TupleType0::num_vals == TupleType1::num_vals, "Invalid number of values");
  TupleReduce<TupleType0, TupleType1, Func, TupleType0::num_vals>::reduce(
      val0, offset0, val1, offset1, reduction_op);
}

// Reduces all of the first (idx+1) values by a thread block
template <
    int idx,
    bool BROADCAST,
    bool FORWARD_PROTECT_SMEM,
    typename LocalTupleT,
    typename... Funcs>
struct BlockReduceEach {
  __inline__ __device__ static void reduce(
      LocalTupleT& block_result,
      const LocalTupleT& partial_result,
      void* shared_mem,
      bool has_block_result,
      int tid_in_reduction,
      int num_threads_per_reduction,
      int num_elements_per_reduction,
      int reduction_idx,
      Funcs... funcs) {
    // Finish the reduction of each tuple value with a smaller offset
    BlockReduceEach<idx - 1, BROADCAST, true, LocalTupleT, Funcs...>::reduce(
        block_result,
        partial_result,
        shared_mem,
        has_block_result,
        tid_in_reduction,
        num_threads_per_reduction,
        num_elements_per_reduction,
        reduction_idx,
        funcs...);

    if (num_elements_per_reduction == 1) {
      if (has_block_result) {
        block_result.val<idx>(0) = partial_result.val<idx>(0);
      }
      return;
    }

    using DataType = typename LocalTupleT::template ValType<idx>;

    PtrTuple<DataType> shared_buf(static_cast<DataType*>(shared_mem));

    LocalTuple<DataType> block_result_i(partial_result.val<idx>(0));

    const auto smem_offset =
        reduction_idx * num_threads_per_reduction + tid_in_reduction;

    const int np2 = 1 << (31 - __clz(num_elements_per_reduction));

    // Threads values are initialized, so all can participate here
    if (tid_in_reduction >= np2) {
      copyTuple(shared_buf, smem_offset, block_result_i);
    }

    block_sync::sync();

    if (tid_in_reduction < np2 &&
        tid_in_reduction + np2 < num_elements_per_reduction) {
      impl::reduceVal<idx, 0>(
          block_result_i, 0, shared_buf, smem_offset + np2, funcs...);
    }

    if (tid_in_reduction < np2) {
      copyTuple(shared_buf, smem_offset, block_result_i);
    }

    // Always sync when communicating across smem
    block_sync::sync();

    // Reduce down to 2 values, last thread will do the final reduction and
    // can save a syncthreads this way
    for (int factor = np2 / 2; factor > 1; factor >>= 1) {
      if (tid_in_reduction < factor) {
        impl::reduceVal<idx, 0>(
            shared_buf,
            smem_offset,
            shared_buf,
            smem_offset + factor,
            funcs...);
      }
      block_sync::sync();
    }

    copyTuple(block_result_i, shared_buf, smem_offset);

    // Do the last reduction
    if (has_block_result) {
      impl::reduceVal<idx, 0>(
          block_result_i, 0, shared_buf, smem_offset + 1, funcs...);
    }

    if (BROADCAST) {
      if (has_block_result) {
        // Put result back in shared memory, put in the first entry of the
        // reduction segment's buffer
        copyTuple(
            shared_buf,
            reduction_idx * num_threads_per_reduction,
            block_result_i);
      }

      // Sync threads to make sure result is in smem
      block_sync::sync();

      copyTuple(
          block_result_i,
          shared_buf,
          reduction_idx * num_threads_per_reduction);
    }

    block_result.val<idx>(0) = block_result_i.val<0>(0);

    if (FORWARD_PROTECT_SMEM) {
      block_sync::sync();
    }
  }
};

// Specialization for idx == -1, i.e., no value to reduce.
template <
    bool BROADCAST,
    bool FORWARD_PROTECT_SMEM,
    typename LocalTupleT,
    typename... Funcs>
struct BlockReduceEach<
    -1,
    BROADCAST,
    FORWARD_PROTECT_SMEM,
    LocalTupleT,
    Funcs...> {
  __inline__ __device__ static void reduce(
      LocalTupleT& block_result,
      const LocalTupleT& partial_result,
      void* shared_mem,
      bool has_block_result,
      int tid_in_reduction,
      int num_threads_per_reduction,
      int num_elements_per_reduction,
      int reduction_idx,
      Funcs... funcs) {}
};

//! Reduce each value of a tuple by a thread block.
//!
//! The final result is broadcast when BROADCAST is true.
//!
//! \param block_result result of the block reduction
//! \param partial_result Per-thread input tuple
//! \param shared_mem
//! \param has_block_result
//! \param tid_in_reduction
//! \param num_threads_per_reduction
//! \param num_elements_per_reduction
//! \param reduction_idx
//! \param reduction_ops
template <
    bool BROADCAST,
    bool FORWARD_PROTECT_SMEM,
    typename LocalTupleT,
    typename... Funcs>
__inline__ __device__ void blockReduceEach(
    LocalTupleT& block_result,
    const LocalTupleT& partial_result,
    void* shared_mem,
    bool has_block_result,
    int tid_in_reduction,
    int num_threads_per_reduction,
    int num_elements_per_reduction,
    int reduction_idx,
    Funcs... reduction_ops) {
  BlockReduceEach<
      LocalTupleT::num_vals - 1,
      BROADCAST,
      FORWARD_PROTECT_SMEM,
      LocalTupleT,
      Funcs...>::
      reduce(
          block_result,
          partial_result,
          shared_mem,
          has_block_result,
          tid_in_reduction,
          num_threads_per_reduction,
          num_elements_per_reduction,
          reduction_idx,
          reduction_ops...);
}

} // namespace impl

// We have 6 dimensions, 3 in the grid, 3 in the block
// They can be 1 of 3 states,
// Reduction Domain - TEMPLATE STATE 0
//   - Participating in the reduction, has values coming in, one value coming
//     out across the dimension
// Iteration Domain - TEMPLATE STATE 1
//   - Not participating in the reduction, has values across the dimension after
//     the reduction
// Collapsed Domain - TEMPLATE STATE 2
//   - Previously reduced, doesn't need to be reduced on that dimension, doesn't
//     have values across that dimension
constexpr __device__ bool isReduce(int STATE) {
  return STATE == 0;
}

constexpr __device__ bool isIter(int STATE) {
  return STATE == 1;
}

constexpr __device__ bool isPred(int STATE) {
  return STATE == 2;
}

constexpr __device__ bool inactive(int STATE) {
  return STATE == 3;
}

constexpr __device__ bool activeNotIter(int STATE) {
  return STATE != 3 && STATE != 1;
}

constexpr __device__ bool isReduceOrIter(int STATE) {
  return isReduce(STATE) || isIter(STATE);
}

// When generating an index into the reduction, we have to stride by iteration
// domains and reduction domains. Collapsed domains we can ignore, but we need
// to make sure they never read or write (need to be predicated to correct
// participation).

// All inclusive reduction with option to re-broadcast. This reduction class
// does not use predication of parallelization in the read or write predicates.
// Instead there are 3 states each dimension of parallelization can have,
// described above. Predication, indexing, and reduction will be done based on
// this information.
template <
    int X_BLOCK,
    int Y_BLOCK,
    int Z_BLOCK,
    int X_THREAD,
    int Y_THREAD,
    int Z_THREAD,
    bool PERSISTENT_REDUCTION,
    bool BROADCAST>
class ParallelReduce {
  static_assert(
      !BROADCAST || PERSISTENT_REDUCTION,
      "Broadcast requires persistent reduction");

  static constexpr bool BLOCK_REDUCE =
      isReduce(X_THREAD) || isReduce(Y_THREAD) || isReduce(Z_THREAD);

  static constexpr bool GRID_REDUCE =
      isReduce(X_BLOCK) || isReduce(Y_BLOCK) || isReduce(Z_BLOCK);

  // ping-pong between global buffers to avoid a second sync
  bool flip = false;

 public:
  __device__ ParallelReduce() {}

  // reduceGroup does not support Welford-style reductions that reduce
  // all values of a tuple together, so this is the only entry point
  // for Welford for now.
  template <typename Func, typename... Types>
  __device__ __inline__ void reduce(
      RefTuple<Types...> out,
      const ConstRefTuple<Types...>& inp,
      VolatilePtrTuple<Types...> global_work_buffer,
      int64_t* global_sync_buffer, // Allocated as product of all
                                   // non-participating Grid dimension
      PtrTuple<Types...> shared_buf,
      bool read_pred, // Prevent reading from out of bounds memory
      bool write_pred, // Prevent from writing out of bounds
      const LocalTuple<Types...>& init_val,
      Func reduction_op);

  //! Profiled version
  template <typename Func, typename... Types>
  __device__ __inline__ void reduce(
      RefTuple<Types...> out,
      const ConstRefTuple<Types...>& inp,
      VolatilePtrTuple<Types...> global_work_buffer,
      int64_t* global_sync_buffer, // Allocated as product of all
                                   // non-participating Grid dimension
      PtrTuple<Types...> shared_buf,
      bool read_pred, // Prevent reading from out of bounds memory
      bool write_pred, // Prevent from writing out of bounds
      const LocalTuple<Types...>& init_val,
      Func reduction_op,
      int64_t& cycles,
      int64_t& count);

  //! Each value of a tuple is independently reduced by the
  //! corresponding reduction op. Thus, Welford-like reductions are
  //! not supported by this interface.
  //!
  //! Note that out is purely used as the output parameter, and its
  //! initial value is not used but just overwritten. Since grid
  //! reductions do not allow serial reduction IterDomains, there is
  //! no need to accumulate into the out parameter.
  template <typename... DataTypes, typename... Funcs, typename... BoolTypes>
  __device__ __inline__ void reduceGroup(
      RefTuple<DataTypes...> out,
      const ConstRefTuple<DataTypes...>& inp,
      VolatilePtrTuple<DataTypes...> global_work_buffer,
      const LocalTuple<DataTypes...>& init_val,
      int64_t* global_sync_buffer,
      void* shared_mem,
      const LocalTuple<BoolTypes...>& read_preds,
      const LocalTuple<BoolTypes...>& write_preds,
      Funcs... funcs);

  //! Profiled version
  template <typename... DataTypes, typename... Funcs, typename... BoolTypes>
  __device__ __inline__ void reduceGroup(
      RefTuple<DataTypes...> out,
      const ConstRefTuple<DataTypes...>& inp,
      VolatilePtrTuple<DataTypes...> global_work_buffer,
      const LocalTuple<DataTypes...>& init_val,
      int64_t* global_sync_buffer,
      void* shared_mem,
      const LocalTuple<BoolTypes...>& read_preds,
      const LocalTuple<BoolTypes...>& write_preds,
      int64_t& cycles,
      int64_t& count,
      Funcs... funcs);

  // User-visible entry point of grouped grid welford +
  // broadcast. Mostly the same as reduceGroup, and it would be
  // possible to combine this to reduceGroup, but it might make the
  // templated data structures even more complicated and difficult to
  // understand. For now, keep it as a separate function.
  //
  // Unlike reduceGroup, though, the data types of welford ops must be
  // the same. For example, reduceGroup can be used to reduce half and
  // float values by passing a tuple of, e.g., LocalTuple<half,
  // float>, but that's not supported here for implementation
  // simplicity. In practice, it should be really uncommon to group
  // welford ops with different data types, so this restriction
  // shouldn't be an issue.
  template <int NumArgs, typename DataType, typename IndexType>
  __device__ __inline__ void welfordGroup(
      typename MakeRefTuple<NumArgs, DataType>::type out_avg,
      typename MakeRefTuple<NumArgs, DataType>::type out_var,
      typename MakeRefTuple<NumArgs, IndexType>::type out_N,
      const typename MakeConstRefTuple<NumArgs, DataType>::type& inp_avg,
      const typename MakeConstRefTuple<NumArgs, DataType>::type& inp_var,
      const typename MakeConstRefTuple<NumArgs, IndexType>::type& inp_N,
      const typename MakeLocalTuple<NumArgs, DataType>::type& init_avg,
      const typename MakeLocalTuple<NumArgs, DataType>::type& init_var,
      const typename MakeLocalTuple<NumArgs, IndexType>::type& init_N,
      typename MakeVolatilePtrTuple<NumArgs, DataType>::type
          global_work_buffer_avg,
      typename MakeVolatilePtrTuple<NumArgs, DataType>::type
          global_work_buffer_var,
      typename MakeVolatilePtrTuple<NumArgs, IndexType>::type
          global_work_buffer_N,
      int64_t* global_sync_buffer,
      PtrTuple<DataType, DataType, IndexType> shared_buf,
      const typename MakeLocalTuple<NumArgs, bool>::type& read_preds,
      const typename MakeLocalTuple<NumArgs, bool>::type& write_preds);

  //! Profiled version
  template <int NumArgs, typename DataType, typename IndexType>
  __device__ __inline__ void welfordGroup(
      typename MakeRefTuple<NumArgs, DataType>::type out_avg,
      typename MakeRefTuple<NumArgs, DataType>::type out_var,
      typename MakeRefTuple<NumArgs, IndexType>::type out_N,
      const typename MakeConstRefTuple<NumArgs, DataType>::type& inp_avg,
      const typename MakeConstRefTuple<NumArgs, DataType>::type& inp_var,
      const typename MakeConstRefTuple<NumArgs, IndexType>::type& inp_N,
      const typename MakeLocalTuple<NumArgs, DataType>::type& init_avg,
      const typename MakeLocalTuple<NumArgs, DataType>::type& init_var,
      const typename MakeLocalTuple<NumArgs, IndexType>::type& init_N,
      typename MakeVolatilePtrTuple<NumArgs, DataType>::type
          global_work_buffer_avg,
      typename MakeVolatilePtrTuple<NumArgs, DataType>::type
          global_work_buffer_var,
      typename MakeVolatilePtrTuple<NumArgs, IndexType>::type
          global_work_buffer_N,
      int64_t* global_sync_buffer,
      PtrTuple<DataType, DataType, IndexType> shared_buf,
      const typename MakeLocalTuple<NumArgs, bool>::type& read_preds,
      const typename MakeLocalTuple<NumArgs, bool>::type& write_preds,
      int64_t& cycles,
      int64_t& count);

  // This is highly specific to the outer-reduction pattern. All the
  // assumptions should be asserted with static_assert at the begging of
  // the fuction.
  template <int NumVals, typename DataType, int BDIMX, int BDIMY>
  __device__ __inline__ void welfordGroupOuter(
      DataType out_avg[NumVals],
      DataType out_var[NumVals],
      nvfuser_index_t out_N[NumVals],
      const DataType in_avg[NumVals],
      const DataType in_var[NumVals],
      nvfuser_index_t in_N,
      DataType* global_buf_avg,
      DataType* global_buf_var,
      nvfuser_index_t* global_buf_N,
      DataType* shared_buf,
      int64_t* global_sync_buffer);

  // Profiled version
  template <int NumVals, typename DataType, int BDIMX, int BDIMY>
  __device__ __inline__ void welfordGroupOuter(
      DataType out_avg[NumVals],
      DataType out_var[NumVals],
      nvfuser_index_t out_N[NumVals],
      const DataType in_avg[NumVals],
      const DataType in_var[NumVals],
      nvfuser_index_t in_N,
      DataType* global_buf_avg,
      DataType* global_buf_var,
      nvfuser_index_t* global_buf_N,
      DataType* shared_buf,
      int64_t* global_sync_buffer,
      int64_t& cycles,
      int64_t& count);

 private:
  __device__ static bool isLastBlockInGrid() {
    return index_utils::maskedIsLast<
               isReduceOrIter(X_BLOCK),
               isReduceOrIter(Y_BLOCK),
               isReduceOrIter(Z_BLOCK)>(blockIdx, gridDim) &&
        index_utils::maskedIsZero<
               !isReduceOrIter(X_BLOCK),
               !isReduceOrIter(Y_BLOCK),
               !isReduceOrIter(Z_BLOCK)>(blockIdx);
  }

  //! Initial per-CTA reduction of each value of a tuple. Each value
  //! is reduced individually, so the shared memory buffer just needs
  //! to be large enough for each value. NOTE that the smem buffer is
  //! not forward protected.
  template <
      bool BLOCK_BROADCAST,
      typename... DataTypes,
      typename... Funcs,
      typename... BoolTypes>
  __device__ __inline__ static LocalTuple<DataTypes...> reduceGroupBlock(
      const ConstRefTuple<DataTypes...>& inp,
      const LocalTuple<DataTypes...>& init_val,
      void* shared_mem,
      const LocalTuple<BoolTypes...>& read_preds,
      bool block_reduce_participate,
      Funcs... funcs);

  //! Final reduction of partial results. Done by all blocks
  //! redundantly when BROADCAST is true, or just one block otherwise.
  //! The smem buffer is assumed synchronized when it is passed in,
  //! but it isn't synchronized when returning from this function.
  template <typename... DataTypes, typename... Funcs, typename... BoolTypes>
  __device__ __inline__ static void reduceGroupLastBlock(
      RefTuple<DataTypes...>& out,
      const VolatilePtrTuple<DataTypes...>& global_work_buffer,
      const LocalTuple<DataTypes...>& init_val,
      void* shared_mem,
      nvfuser_index_t block_red_idx_offset,
      nvfuser_index_t num_thread_iters,
      nvfuser_index_t num_block_iters,
      nvfuser_index_t thread_red_idx_offset,
      nvfuser_index_t grid_red_size,
      const LocalTuple<BoolTypes...>& write_preds,
      bool block_reduce_participate,
      bool grid_reduce_participate,
      Funcs... reduction_ops);

  //! Welford version of reduceGroupBlock
  template <
      bool BLOCK_BROADCAST,
      int NumVals,
      typename DataType,
      typename IndexType>
  __device__ __inline__ static void welfordGroupBlock(
      LocalWelfordTripletTuple<NumVals, DataType, IndexType>& block_result,
      const ConstRefWelfordTripletTuple<NumVals, DataType, IndexType>& inp,
      PtrTuple<DataType, DataType, IndexType> shared_buf,
      const typename MakeLocalTuple<NumVals, bool>::type& read_preds,
      bool block_reduce_participate);

  //! Welford version of reduceGrouplLastBlock
  template <int NumVals, typename DataType, typename IndexType>
  __device__ __inline__ static void welfordGroupLastBlock(
      RefWelfordTripletTuple<NumVals, DataType, IndexType>& out,
      const VolatilePtrWelfordTripletTuple<NumVals, DataType, IndexType>&
          global_work_buffer,
      const LocalWelfordTripletTuple<NumVals, DataType, IndexType>& init_val,
      PtrTuple<DataType, DataType, IndexType> shared_buf,
      nvfuser_index_t block_red_idx_offset,
      nvfuser_index_t num_thread_iters,
      nvfuser_index_t num_block_iters,
      nvfuser_index_t thread_red_idx_offset,
      nvfuser_index_t grid_red_size,
      const typename MakeLocalTuple<NumVals, bool>::type& write_preds,
      bool block_reduce_participate,
      bool grid_reduce_participate);

  // End Parallel reduce class
};

template <
    int X_BLOCK,
    int Y_BLOCK,
    int Z_BLOCK,
    int X_THREAD,
    int Y_THREAD,
    int Z_THREAD,
    bool PERSISTENT_REDUCTION,
    bool BROADCAST>
template <typename Func, typename... Types>
__device__ __inline__ void ParallelReduce<
    X_BLOCK,
    Y_BLOCK,
    Z_BLOCK,
    X_THREAD,
    Y_THREAD,
    Z_THREAD,
    PERSISTENT_REDUCTION,
    BROADCAST>::
    reduce(
        RefTuple<Types...> out,
        const ConstRefTuple<Types...>& inp,
        VolatilePtrTuple<Types...> global_work_buffer,
        int64_t* global_sync_buffer, // Allocated as product of all
        // non-participating Grid dimension
        PtrTuple<Types...> shared_buf,
        bool read_pred, // Prevent reading from out of bounds memory
        bool write_pred, // Prevent from writing out of bounds
        const LocalTuple<Types...>& init_val,
        Func reduction_op) {
  // If no reduction needed, just return input
  if (!BLOCK_REDUCE && !GRID_REDUCE) {
    if (read_pred && write_pred) {
      out = inp;
    }
    return;
  }

  // Don't read/write in temporary buffers if in a predicated dimension
  bool block_reduce_participate = index_utils::
      maskedIsZero<isPred(X_THREAD), isPred(Y_THREAD), isPred(Z_THREAD)>(
          threadIdx);

  // Initialize block result
  LocalTuple<Types...> block_result = init_val;

  // Grab input data if participating in the reduction, set to block_result in
  // the case there is no block reduction
  if (block_reduce_participate && read_pred) {
    block_result = inp;
  }

  // Only threads that with id == 0 in the dimensions being reduced will
  // have a valid result
  bool has_block_result = index_utils::
      maskedIsZero<isReduce(X_THREAD), isReduce(Y_THREAD), isReduce(Z_THREAD)>(
          threadIdx);

  if (BLOCK_REDUCE) {
    // -- START BLOCK REDUCTION -- //

    // Size of the block reduction segment, can be an int since it's limited
    // to number of threads
    int block_reduction_size = index_utils::
        maskedSize<isReduce(X_THREAD), isReduce(Y_THREAD), isReduce(Z_THREAD)>(
            blockDim);

    // Index in the reduction segment, can be an int since it's limited to
    // number of threads
    int tid_in_block_reduction = index_utils::maskedOffset<
        isReduce(X_THREAD),
        isReduce(Y_THREAD),
        isReduce(Z_THREAD)>(threadIdx, blockDim);

    // ID of the block reduction this thread is participating in
    //
    // If any of the parallel dimensions are predicated out, that means
    // they've already been reduced, so we only care about the first thread in
    // that dimension. Therefore don't expand the reduction_idx by that
    // dimension
    int block_reduction_idx = index_utils::
        maskedOffset<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
            threadIdx, blockDim);

    // Shared memory buffer is 2D
    // [iter dimension, reduction dimension]

    // Offset into smem for the current thread
    int block_reduce_smem_offset =
        block_reduction_idx * block_reduction_size + tid_in_block_reduction;

    // Initialize shared memory
    if (block_reduce_participate) {
      copyTuple(shared_buf, block_reduce_smem_offset, block_result);
    }

    // Sync to make sure smem is completely initialized
    block_sync::sync();

    // Round reduction size down to nearest power of 2
    int np2 = 1 << (31 - __clz(block_reduction_size));

    // Perform an initial reduction leaving np2 elements
    if (block_reduce_participate && tid_in_block_reduction < np2 &&
        tid_in_block_reduction + np2 < block_reduction_size) {
      impl::reduceTuple(
          shared_buf,
          block_reduce_smem_offset,
          shared_buf,
          block_reduce_smem_offset + np2,
          reduction_op);
    }

    // Always need to sync while operating on shared memory
    block_sync::sync();

    // Reduce down until 2 values, leaving 2 values allows us to manually
    // perform the last reduction and avoid a syncthreads
    for (int factor = np2 / 2; factor > 1; factor >>= 1) {
      if (tid_in_block_reduction < factor && block_reduce_participate) {
        impl::reduceTuple(
            shared_buf,
            block_reduce_smem_offset,
            shared_buf,
            block_reduce_smem_offset + factor,
            reduction_op);
      }
      block_sync::sync();
    }

    // Accumulate that last valid result
    if (has_block_result) {
      copyTuple(block_result, shared_buf, block_reduce_smem_offset);
      if (block_reduction_size > 1) {
        impl::reduceTuple(
            block_result,
            0,
            shared_buf,
            block_reduce_smem_offset + 1,
            reduction_op);
      }
    }

    // ===== BLOCK REDUCTION CLEANUP =======
    if (!GRID_REDUCE) {
      // If no grid reduction, we don't have to continue. Either broadcast
      // back across the block or return the correct reduction
      if (has_block_result && write_pred) {
        impl::reduceTuple(block_result, 0, out, 0, reduction_op);
        out = block_result;
      }
      if (BROADCAST) {
        // No grid reduce, but need to broadcast, perform block broadcast
        if (has_block_result && write_pred) {
          // Put result back in shared memory, put in the first entry of the
          // reduction segment's buffer
          copyTuple(
              shared_buf,
              block_reduction_idx * block_reduction_size,
              block_result);
        }

        // Sync threads to make sure result is in smem
        block_sync::sync();
        // If the thread is participating, and is not attempting to write out
        // of bounds, return the broadcasted value.
        if (block_reduce_participate && write_pred) {
          copyTuple(
              out, shared_buf, block_reduction_idx * block_reduction_size);
        }
      }

      // Forward protect shared memory, don't want threads to continue to
      // another reduction/broadcast and pollute shared memory before the
      // reduction is completely finished.
      //
      // This could be avoided in some cases if we added thread syncs from
      // block reductions in the syncthread insertion pass.
      block_sync::sync();
      return;
    }
  }

  // -- START GRID REDUCTION -- //
  // Grid reductions are more challenging for two reasons, (1) the reduction
  // itself is 3D instead of 2D because we now have an iter domain space in
  // the grid dimension. (2) a tree reduction isn't performed, instead all
  // blocks will populate GMEM and one  block will finish the grid reduction.

  // What is the grid reduction size, block reduction already performed so
  // that doesn't have to be taken into consideration
  const auto grid_red_size = index_utils::
      maskedSize<isReduce(X_BLOCK), isReduce(Y_BLOCK), isReduce(Z_BLOCK)>(
          gridDim);

  // Which ID in the reduction is this block. Threads can participate in
  // multiple grid reductions, but the block will have the same relative index
  // in those reductions
  const auto idx_in_grid_red = index_utils::
      maskedOffset<isReduce(X_BLOCK), isReduce(Y_BLOCK), isReduce(Z_BLOCK)>(
          blockIdx, gridDim);

  if (PERSISTENT_REDUCTION && flip) {
    auto global_buffer_size =
        index_utils::
            maskedSize<isIter(X_BLOCK), isIter(Y_BLOCK), isIter(Z_BLOCK)>(
                gridDim) *
        index_utils::
            maskedSize<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
                blockDim) *
        grid_red_size;
    global_work_buffer += global_buffer_size;
  }
  flip = !flip;

  // How many grid reductions have to be performed, in the grid dimension
  const auto num_block_iters = index_utils::
      maskedSize<isIter(X_BLOCK), isIter(Y_BLOCK), isIter(Z_BLOCK)>(gridDim);

  // Which grid reduction does this block participate in, in the grid
  // dimension
  const auto block_red_idx_offset = index_utils::
      maskedOffset<isIter(X_BLOCK), isIter(Y_BLOCK), isIter(Z_BLOCK)>(
          blockIdx, gridDim);

  // How many grid reductions have to be performed, in the block dimension
  const auto num_thread_iters = index_utils::
      maskedSize<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
          blockDim);

  // Which grid reduction does this thread participate in, in the block
  // dimension
  const auto thread_red_idx_offset = index_utils::
      maskedOffset<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
          threadIdx, blockDim);

  // 3D buffer of reductions:
  //    [reduction_offset(grid), iter_offset(grid), iter_offset(block)]
  // Offset into the work buffer
  const auto work_buf_offset =
      (idx_in_grid_red * num_block_iters + block_red_idx_offset) *
          num_thread_iters +
      thread_red_idx_offset;

  // Don't read/write in temporary buffers if in a predicated dimension
  bool grid_reduce_participate = index_utils::
      maskedIsZero<isPred(X_BLOCK), isPred(Y_BLOCK), isPred(Z_BLOCK)>(blockIdx);

  if (grid_reduce_participate && block_reduce_participate) {
    if (has_block_result) {
      copyTuple(global_work_buffer, work_buf_offset, block_result);
    }
  }

  // -- GLOBAL BUFFER FILLED -- //

  bool last_block = index_utils::
      maskedIsLast<isReduce(X_BLOCK), isReduce(Y_BLOCK), isReduce(Z_BLOCK)>(
          blockIdx, gridDim);

  if (grid_reduce_participate) {
    // Don't need to sync up blocks that are not participating in this
    // reduction
    grid_sync::sync<
        isReduce(X_BLOCK),
        isReduce(Y_BLOCK),
        isReduce(Z_BLOCK),
        PERSISTENT_REDUCTION>(
        global_sync_buffer[block_red_idx_offset], grid_red_size, last_block);
  }

  // -- START BLOCK CLEANUP -- //
  // All blocks perform the last cleanup, so every block, and every thread
  // will have the final result

  // Initialize block result
  LocalTuple<Types...> last_block_result(init_val);

  if ((PERSISTENT_REDUCTION || last_block) && grid_reduce_participate) {
    // Can use the last block to reduce all the values the blocks filled in.
    // Can use any thread that has been predicated, or has been reduced to do
    // this reduction, cannot use any block that's associated with an
    // iteration domain

    // Start with non-block reduction

    // Index in the reduction segment
    int tid_in_block_reduction_2 = index_utils::maskedOffset<
        activeNotIter(X_THREAD),
        activeNotIter(Y_THREAD),
        activeNotIter(Z_THREAD)>(threadIdx, blockDim);

    int block_reduction_size_2 = index_utils::maskedSize<
        activeNotIter(X_THREAD),
        activeNotIter(Y_THREAD),
        activeNotIter(Z_THREAD)>(blockDim);

    // 3D buffer of reductions:
    //    [reduction_offset(grid), iter_offset(grid), iter_offset(block)]
    // Change the offset, we want to keep the last two dimensions, but the
    // first dimension is what we will reduce over
    const auto work_buf_offset_2 =
        block_red_idx_offset * num_thread_iters + thread_red_idx_offset;
    for (auto reduction_i = tid_in_block_reduction_2;
         reduction_i < grid_red_size;
         reduction_i += block_reduction_size_2) {
      impl::reduceTuple(
          last_block_result,
          0,
          global_work_buffer,
          work_buf_offset_2 +
              reduction_i * num_block_iters *
                  num_thread_iters, // Iterating over the outer most
          // dimension, so need to stride by the
          // total number of grid reductions. Could
          // come back and change it so this is the
          // contiguous dimension
          reduction_op);
    }

    // -- START LAST BLOCK - BLOCK REDUCTION -- //

    // Reduced so we have one value per thread, we need to further reduce any
    // dimension that is not an iter dimension

    // Which block reduction this thread is participating in
    int block_reduction_idx = index_utils::
        maskedOffset<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
            threadIdx, blockDim);

    // Offset in smem for this thread's result
    auto smem_offset =
        block_reduction_idx * block_reduction_size_2 + tid_in_block_reduction_2;

    // Similar as before, reduce down to nearest power of 2 so we can do a
    // tree reduction
    int np2 = 1 << (31 - __clz(min(block_reduction_size_2, grid_red_size)));

    // Threads values are initialized, so all can participate here
    if (tid_in_block_reduction_2 >= np2) {
      copyTuple(shared_buf, smem_offset, last_block_result);
    }

    block_sync::sync();

    if (tid_in_block_reduction_2 < np2 &&
        tid_in_block_reduction_2 + np2 <
            min(block_reduction_size_2, grid_red_size)) {
      impl::reduceTuple(
          last_block_result, 0, shared_buf, smem_offset + np2, reduction_op);
    }

    if (tid_in_block_reduction_2 < np2) {
      copyTuple(shared_buf, smem_offset, last_block_result);
    }

    // Always sync when communicating across smem
    block_sync::sync();

    // Reduce down to 2 values, last thread will do the final reduction and
    // can save a syncthreads this way
    for (int factor = np2 / 2; factor > 1; factor >>= 1) {
      if (tid_in_block_reduction_2 < factor) {
        impl::reduceTuple(
            shared_buf,
            smem_offset,
            shared_buf,
            smem_offset + factor,
            reduction_op);
      }
      block_sync::sync();
    }

    // If this thread in each block has the final result before broadcasting
    // to all other threads in block
    bool has_block_result_2 = index_utils::maskedIsZero<
        activeNotIter(X_THREAD),
        activeNotIter(Y_THREAD),
        activeNotIter(Z_THREAD)>(threadIdx);
    // Do the last reduction, protected by the write predicate
    copyTuple(last_block_result, shared_buf, smem_offset);
    if (has_block_result && grid_reduce_participate) {
      impl::reduceTuple(last_block_result, 0, out, 0, reduction_op);
      if (min(block_reduction_size_2, grid_red_size) > 1) {
        impl::reduceTuple(
            last_block_result, 0, shared_buf, smem_offset + 1, reduction_op);
      }
    }

    if (grid_reduce_participate && PERSISTENT_REDUCTION) {
      // If persistent reduction, always broadcast reduced values
      copyTuple(shared_buf, smem_offset, last_block_result);
      block_sync::sync();
      if (write_pred && block_reduce_participate) {
        copyTuple(
            out, shared_buf, block_reduction_idx * block_reduction_size_2);
      }
      // For persistent kernels we double the global buffer allocation so we
      // don't need to protect those buffers every iteration preventing the
      // need of an additional grid_sync. Since we flip back and forth between
      // sections of the buffer, the one grid sync protects the other part of
      // the buffer.
    } else {
      if (grid_reduce_participate) {
        if (last_block && has_block_result && block_reduce_participate &&
            write_pred) {
          copyTuple(
              out, shared_buf, block_reduction_idx * block_reduction_size_2);
        }
      }
    }
    // Forward protect the smem used in this reduction
    block_sync::sync();
  }
}

//! Profiled version
template <
    int X_BLOCK,
    int Y_BLOCK,
    int Z_BLOCK,
    int X_THREAD,
    int Y_THREAD,
    int Z_THREAD,
    bool PERSISTENT_REDUCTION,
    bool BROADCAST>
template <typename Func, typename... Types>
__device__ __inline__ void ParallelReduce<
    X_BLOCK,
    Y_BLOCK,
    Z_BLOCK,
    X_THREAD,
    Y_THREAD,
    Z_THREAD,
    PERSISTENT_REDUCTION,
    BROADCAST>::
    reduce(
        RefTuple<Types...> out,
        const ConstRefTuple<Types...>& inp,
        VolatilePtrTuple<Types...> global_work_buffer,
        int64_t* global_sync_buffer, // Allocated as product of all
        // non-participating Grid dimension
        PtrTuple<Types...> shared_buf,
        bool read_pred, // Prevent reading from out of bounds memory
        bool write_pred, // Prevent from writing out of bounds
        const LocalTuple<Types...>& init_val,
        Func reduction_op,
        int64_t& cycles,
        int64_t& count) {
  int64_t start_counter = 0;

  if (isLastBlockInGrid() &&
      index_utils::maskedIsZero<true, true, true>(threadIdx)) {
    start_counter = readCycleCounter();
  }

  reduce(
      out,
      inp,
      global_work_buffer,
      global_sync_buffer,
      shared_buf,
      read_pred,
      write_pred,
      init_val,
      reduction_op);

  if (isLastBlockInGrid() &&
      index_utils::maskedIsZero<true, true, true>(threadIdx)) {
    cycles += readCycleCounter() - start_counter;
    ++count;
  }
}

template <
    int X_BLOCK,
    int Y_BLOCK,
    int Z_BLOCK,
    int X_THREAD,
    int Y_THREAD,
    int Z_THREAD,
    bool PERSISTENT_REDUCTION,
    bool BROADCAST>
template <typename... DataTypes, typename... Funcs, typename... BoolTypes>
__device__ __inline__ void ParallelReduce<
    X_BLOCK,
    Y_BLOCK,
    Z_BLOCK,
    X_THREAD,
    Y_THREAD,
    Z_THREAD,
    PERSISTENT_REDUCTION,
    BROADCAST>::
    reduceGroup(
        RefTuple<DataTypes...> out,
        const ConstRefTuple<DataTypes...>& inp,
        VolatilePtrTuple<DataTypes...> global_work_buffer,
        const LocalTuple<DataTypes...>& init_val,
        int64_t* global_sync_buffer,
        void* shared_mem,
        const LocalTuple<BoolTypes...>& read_preds,
        const LocalTuple<BoolTypes...>& write_preds,
        Funcs... funcs) {
  static_assert(
      sizeof...(DataTypes) == sizeof...(Funcs),
      "Mismatched number of Tuple values and functions");
  static_assert(
      sizeof...(DataTypes) == sizeof...(BoolTypes),
      "Mismatched number of Tuple values and predicate values");

  // If no reduction needed, just return input
  if (!BLOCK_REDUCE && !GRID_REDUCE) {
    copyTupleIf(out, inp, read_preds && write_preds);
    return;
  }

  // Don't read/write in temporary buffers if in a predicated dimension
  const bool block_reduce_participate = index_utils::
      maskedIsZero<isPred(X_THREAD), isPred(Y_THREAD), isPred(Z_THREAD)>(
          threadIdx);

  // Only threads that with id == 0 in the dimensions being reduced will
  // have a valid result
  const bool has_block_result = index_utils::
      maskedIsZero<isReduce(X_THREAD), isReduce(Y_THREAD), isReduce(Z_THREAD)>(
          threadIdx);

  // Initial per-block reduction. Result is broadcast if specified
  // and this call is block reduction only.
  const auto block_result = reduceGroupBlock < !GRID_REDUCE &&
      BROADCAST > (inp,
                   init_val,
                   shared_mem,
                   read_preds,
                   block_reduce_participate,
                   funcs...);
  // If block reduction only, save to out and exit
  if (!GRID_REDUCE) {
    copyTupleIf(
        out,
        block_result,
        write_preds &&
            (block_reduce_participate && (BROADCAST || has_block_result)));

    // Need a block sync here as reduceGroupBlock does not
    // forward-protect the smem buffer. This block sync is not
    // necessary when a grid reduction follows since a block sync is
    // done just before the grid sync.
    block_sync::sync();
    return;
  }

  // -- START GRID REDUCTION -- //
  // Grid reductions are more challenging for two reasons, (1) the reduction
  // itself is 3D instead of 2D because we now have an iter domain space in
  // the grid dimension. (2) a tree reduction isn't performed, instead all
  // blocks will populate GMEM and one  block will finish the grid reduction.

  // What is the grid reduction size, block reduction already performed so
  // that doesn't have to be taken into consideration
  const auto grid_red_size = index_utils::
      maskedSize<isReduce(X_BLOCK), isReduce(Y_BLOCK), isReduce(Z_BLOCK)>(
          gridDim);

  // Which ID in the reduction is this block. Threads can participate in
  // multiple grid reductions, but the block will have the same relative index
  // in those reductions
  const auto idx_in_grid_red = index_utils::
      maskedOffset<isReduce(X_BLOCK), isReduce(Y_BLOCK), isReduce(Z_BLOCK)>(
          blockIdx, gridDim);

  // How many grid reductions have to be performed, in the grid dimension
  const auto num_block_iters = index_utils::
      maskedSize<isIter(X_BLOCK), isIter(Y_BLOCK), isIter(Z_BLOCK)>(gridDim);

  // Which grid reduction does this block participate in, in the grid
  // dimension
  const auto block_red_idx_offset = index_utils::
      maskedOffset<isIter(X_BLOCK), isIter(Y_BLOCK), isIter(Z_BLOCK)>(
          blockIdx, gridDim);

  // How many grid reductions have to be performed, in the block dimension
  const auto num_thread_iters = index_utils::
      maskedSize<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
          blockDim);

  // Which grid reduction does this thread participate in, in the block
  // dimension
  const auto thread_red_idx_offset = index_utils::
      maskedOffset<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
          threadIdx, blockDim);

  // 3D buffer of reductions:
  //    [reduction_offset(grid), iter_offset(grid), iter_offset(block)]
  // Offset into the work buffer
  const auto work_buf_offset =
      (idx_in_grid_red * num_block_iters + block_red_idx_offset) *
          num_thread_iters +
      thread_red_idx_offset;

  // Don't read/write in temporary buffers if in a predicated dimension
  bool grid_reduce_participate = index_utils::
      maskedIsZero<isPred(X_BLOCK), isPred(Y_BLOCK), isPred(Z_BLOCK)>(blockIdx);

  if (PERSISTENT_REDUCTION && flip) {
    auto global_buffer_size =
        index_utils::
            maskedSize<isIter(X_BLOCK), isIter(Y_BLOCK), isIter(Z_BLOCK)>(
                gridDim) *
        index_utils::
            maskedSize<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
                blockDim) *
        grid_red_size;
    global_work_buffer += global_buffer_size;
  }
  flip = !flip;

  // Per-block partial reduction to global work buffer
  if (grid_reduce_participate && block_reduce_participate && has_block_result) {
    copyTuple(global_work_buffer, work_buf_offset, block_result);
  }

  // -- GLOBAL BUFFER FILLED -- //

  bool last_block = index_utils::
      maskedIsLast<isReduce(X_BLOCK), isReduce(Y_BLOCK), isReduce(Z_BLOCK)>(
          blockIdx, gridDim);

  if (grid_reduce_participate) {
    // Don't need to sync up blocks that are not participating in this
    // reduction
    grid_sync::sync<
        isReduce(X_BLOCK),
        isReduce(Y_BLOCK),
        isReduce(Z_BLOCK),
        PERSISTENT_REDUCTION>(
        global_sync_buffer[block_red_idx_offset], grid_red_size, last_block);
  }

  // -- START BLOCK CLEANUP -- //
  reduceGroupLastBlock(
      out,
      global_work_buffer,
      init_val,
      shared_mem,
      block_red_idx_offset,
      num_thread_iters,
      num_block_iters,
      thread_red_idx_offset,
      grid_red_size,
      write_preds,
      block_reduce_participate,
      grid_reduce_participate,
      funcs...);

  // Forward protect the smem buffer
  block_sync::sync();
}

template <
    int X_BLOCK,
    int Y_BLOCK,
    int Z_BLOCK,
    int X_THREAD,
    int Y_THREAD,
    int Z_THREAD,
    bool PERSISTENT_REDUCTION,
    bool BROADCAST>
template <typename... DataTypes, typename... Funcs, typename... BoolTypes>
__device__ __inline__ void ParallelReduce<
    X_BLOCK,
    Y_BLOCK,
    Z_BLOCK,
    X_THREAD,
    Y_THREAD,
    Z_THREAD,
    PERSISTENT_REDUCTION,
    BROADCAST>::
    reduceGroup(
        RefTuple<DataTypes...> out,
        const ConstRefTuple<DataTypes...>& inp,
        VolatilePtrTuple<DataTypes...> global_work_buffer,
        const LocalTuple<DataTypes...>& init_val,
        int64_t* global_sync_buffer,
        void* shared_mem,
        const LocalTuple<BoolTypes...>& read_preds,
        const LocalTuple<BoolTypes...>& write_preds,
        int64_t& cycles,
        int64_t& count,
        Funcs... funcs) {
  int64_t start_counter = 0;

  if (isLastBlockInGrid() &&
      index_utils::maskedIsZero<true, true, true>(threadIdx)) {
    start_counter = readCycleCounter();
  }

  reduceGroup(
      out,
      inp,
      global_work_buffer,
      init_val,
      global_sync_buffer,
      shared_mem,
      read_preds,
      write_preds,
      funcs...);

  if (isLastBlockInGrid() &&
      index_utils::maskedIsZero<true, true, true>(threadIdx)) {
    cycles += readCycleCounter() - start_counter;
    ++count;
  }
}

template <
    int X_BLOCK,
    int Y_BLOCK,
    int Z_BLOCK,
    int X_THREAD,
    int Y_THREAD,
    int Z_THREAD,
    bool PERSISTENT_REDUCTION,
    bool BROADCAST>
template <
    bool BLOCK_BROADCAST,
    typename... DataTypes,
    typename... Funcs,
    typename... BoolTypes>
__device__ __inline__ LocalTuple<DataTypes...> ParallelReduce<
    X_BLOCK,
    Y_BLOCK,
    Z_BLOCK,
    X_THREAD,
    Y_THREAD,
    Z_THREAD,
    PERSISTENT_REDUCTION,
    BROADCAST>::
    reduceGroupBlock(
        const ConstRefTuple<DataTypes...>& inp,
        const LocalTuple<DataTypes...>& init_val,
        void* shared_mem,
        const LocalTuple<BoolTypes...>& read_preds,
        bool block_reduce_participate,
        Funcs... funcs) {
  const bool has_block_result = index_utils::
      maskedIsZero<isReduce(X_THREAD), isReduce(Y_THREAD), isReduce(Z_THREAD)>(
          threadIdx);

  // Initialize block result
  LocalTuple<DataTypes...> block_result = init_val;

  copyTupleIf(block_result, inp, block_reduce_participate && read_preds);

  // Size of the block reduction segment, can be an int since it's limited
  // to number of threads
  const int block_reduction_size = index_utils::
      maskedSize<isReduce(X_THREAD), isReduce(Y_THREAD), isReduce(Z_THREAD)>(
          blockDim);

  // Index in the reduction segment, can be an int since it's limited to
  // number of threads
  const int tid_in_block_reduction = index_utils::
      maskedOffset<isReduce(X_THREAD), isReduce(Y_THREAD), isReduce(Z_THREAD)>(
          threadIdx, blockDim);

  // ID of the block reduction this thread is participating in
  //
  // If any of the parallel dimensions are predicated out, that means
  // they've already been reduced, so we only care about the first thread in
  // that dimension. Therefore don't expand the reduction_idx by that
  // dimension
  const int block_reduction_idx = index_utils::
      maskedOffset<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
          threadIdx, blockDim);

  // Do not protect the smem buffer as it's not always necessary.
  impl::blockReduceEach<
      BLOCK_BROADCAST,
      false,
      LocalTuple<DataTypes...>,
      Funcs...>(
      block_result,
      block_result,
      shared_mem,
      has_block_result,
      tid_in_block_reduction,
      block_reduction_size,
      block_reduction_size,
      block_reduction_idx,
      funcs...);

  return block_result;
}

template <
    int X_BLOCK,
    int Y_BLOCK,
    int Z_BLOCK,
    int X_THREAD,
    int Y_THREAD,
    int Z_THREAD,
    bool PERSISTENT_REDUCTION,
    bool BROADCAST>
template <typename... DataTypes, typename... Funcs, typename... BoolTypes>
__device__ __inline__ void ParallelReduce<
    X_BLOCK,
    Y_BLOCK,
    Z_BLOCK,
    X_THREAD,
    Y_THREAD,
    Z_THREAD,
    PERSISTENT_REDUCTION,
    BROADCAST>::
    reduceGroupLastBlock(
        RefTuple<DataTypes...>& out,
        const VolatilePtrTuple<DataTypes...>& global_work_buffer,
        const LocalTuple<DataTypes...>& init_val,
        void* shared_mem,
        nvfuser_index_t block_red_idx_offset,
        nvfuser_index_t num_thread_iters,
        nvfuser_index_t num_block_iters,
        nvfuser_index_t thread_red_idx_offset,
        nvfuser_index_t grid_red_size,
        const LocalTuple<BoolTypes...>& write_preds,
        bool block_reduce_participate,
        bool grid_reduce_participate,
        Funcs... reduction_ops) {
  // Initialize block result
  LocalTuple<DataTypes...> last_block_result(init_val);

  const bool last_block = index_utils::
      maskedIsLast<isReduce(X_BLOCK), isReduce(Y_BLOCK), isReduce(Z_BLOCK)>(
          blockIdx, gridDim);

  if ((PERSISTENT_REDUCTION || last_block) && grid_reduce_participate) {
    // Can use the last block to reduce all the values the blocks filled in.
    // Can use any thread that has been predicated, or has been reduced to do
    // this reduction, cannot use any block that's associated with an
    // iteration domain

    // Start with non-block reduction

    // Index in the reduction segment
    int tid_in_block_reduction = index_utils::maskedOffset<
        activeNotIter(X_THREAD),
        activeNotIter(Y_THREAD),
        activeNotIter(Z_THREAD)>(threadIdx, blockDim);

    int block_reduction_size = index_utils::maskedSize<
        activeNotIter(X_THREAD),
        activeNotIter(Y_THREAD),
        activeNotIter(Z_THREAD)>(blockDim);

    bool has_block_result = index_utils::maskedIsZero<
        activeNotIter(X_THREAD),
        activeNotIter(Y_THREAD),
        activeNotIter(Z_THREAD)>(threadIdx);

    // 3D buffer of reductions:
    //    [reduction_offset(grid), iter_offset(grid), iter_offset(block)]
    // Change the offset, we want to keep the last two dimensions, but the
    // first dimension is what we will reduce over
    const auto work_buf_offset =
        block_red_idx_offset * num_thread_iters + thread_red_idx_offset;
    for (auto reduction_i = tid_in_block_reduction; reduction_i < grid_red_size;
         reduction_i += block_reduction_size) {
      impl::reduceEach(
          last_block_result,
          0,
          global_work_buffer,
          work_buf_offset +
              reduction_i * num_block_iters *
                  num_thread_iters, // Iterating over the outer most
                                    // dimension, so need to stride by the
                                    // total number of grid reductions. Could
                                    // come back and change it so this is the
                                    // contiguous dimension
          reduction_ops...);
    }

    // Which block reduction this thread is participating in
    int block_reduction_idx = index_utils::
        maskedOffset<isIter(X_THREAD), isIter(Y_THREAD), isIter(Z_THREAD)>(
            threadIdx, blockDim);

    impl::blockReduceEach<BROADCAST, false, LocalTuple<DataTypes...>, Funcs...>(
        last_block_result,
        last_block_result,
        shared_mem,
        has_block_result,
        tid_in_block_reduction,
        block_reduction_size,
        min(grid_red_size, block_reduction_size),
        block_reduction_idx,
        reduction_ops...);

    copyTupleIf(
        out,
        last_block_result,
        write_preds &&
            (block_reduce_participate && (BROADCAST || has_block_result)));
  }
}

} // namespace fused_reduction
