#include "hip/hip_runtime.h"
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on
namespace grid_broadcast {

// Broadcasts per-thread values across threads and blocks.
//
// Function parameters:
// - out: Per-thread output location
// - inp_val: Per-thread input value
// - work_buf: Temporary buffer for communication across threads/blocks
// - sync_flags: A vector of integers for synchronizations
//
// Template parameters:
// - X/Y/Z_BLOCK: When true, broadcasts across thread blocks along the X/Y/Z
//   dimensions
// - X/Y/Z_THREAD: When true, broadcasts across threads along the X/Y/Z
//   dimensions
template <
    bool X_BLOCK,
    bool Y_BLOCK,
    bool Z_BLOCK,
    bool X_THREAD,
    bool Y_THREAD,
    bool Z_THREAD,
    typename T>
__device__ void broadcast(
    T& out,
    const T& inp_val,
    volatile T* work_buf,
    Tensor<int64_t, 1> sync_flags,
    bool read_write_pred) {
  // Number of values broadcasted in the grid dimensions
  const auto grid_seg_size =
      index_utils::maskedSize<X_BLOCK, Y_BLOCK, Z_BLOCK>(gridDim);

  // Index of the broadcast we're performing out of the grid_seg_size
  const auto grid_seg_idx =
      index_utils::maskedOffset<!X_BLOCK, !Y_BLOCK, !Z_BLOCK>(
          blockIdx, gridDim);

  // Number of threads not participating in a broadcast dimension, this is the
  // number of thread entries to expect in the work buffer, therefore a striding
  const auto block_stride =
      index_utils::maskedSize<!X_THREAD, !Y_THREAD, !Z_THREAD>(blockDim);

  // Which broadcast in the block this is to line up the entry with the work
  // buffer
  const auto thread_offset =
      index_utils::maskedOffset<!X_THREAD, !Y_THREAD, !Z_THREAD>(
          threadIdx, blockDim);

  const bool has_valid_data = (!X_BLOCK || blockIdx.x == gridDim.x - 1) &&
      (!Y_BLOCK || blockIdx.y == gridDim.y - 1) &&
      (!Z_BLOCK || blockIdx.z == gridDim.z - 1) &&
      (!X_THREAD || threadIdx.x == 0) && (!Y_THREAD || threadIdx.y == 0) &&
      (!Z_THREAD || threadIdx.z == 0);

  if (has_valid_data && read_write_pred) {
    work_buf[grid_seg_idx * block_stride + thread_offset] = inp_val;
    __threadfence();
  }

  grid_sync::sync<X_BLOCK, Y_BLOCK, Z_BLOCK, true>(
      sync_flags[grid_seg_idx], grid_seg_size);

  if (read_write_pred) {
    out = work_buf[grid_seg_idx * block_stride + thread_offset];
  }

  // Make sure everyone has read from the buffer before continuing the kernel
  // and potentially overwriting
  grid_sync::sync<X_BLOCK, Y_BLOCK, Z_BLOCK, true>(
      sync_flags[grid_seg_idx], grid_seg_size);
}
} // namespace grid_broadcast
