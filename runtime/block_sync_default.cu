
#include <hip/hip_runtime.h>
// clang-format off
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-present NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 */
// clang-format on

struct DefaultBlockDim {
  const uint32_t x, y, z;
  __device__ DefaultBlockDim() : x(blockDim.x), y(blockDim.y), z(blockDim.z) {}
  operator dim3() const {
    return blockDim;
  }
};

// Default block synchronization. Just use __barrier_sync
namespace block_sync {

__forceinline__ __device__ void init() {}

// Thread-block synchronization
template <bool aligned, typename BlockDimT>
__forceinline__ __device__ void sync(BlockDimT block_dim) {
  if constexpr (aligned) {
    __syncthreads();
  } else if constexpr (std::is_same_v<BlockDimT, DefaultBlockDim>) {
    __barrier_sync(0);
  } else {
    uint32_t num_threads = block_dim.x * block_dim.y * block_dim.z;
    asm volatile("bar.sync 0, %0;" : : "r"(num_threads) : "memory");
  }
}

} // namespace block_sync
